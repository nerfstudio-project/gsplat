#include "hip/hip_runtime.h"
#include <cstdlib>
#include <cstring>
#include <ctime>
#include <iostream>
#include <math.h>

#include "backward.cuh"
#include "backward_ref.cuh"
#include "forward.cuh"
#include "helpers.cuh"

float random_float() { return (float)std::rand() / RAND_MAX; }

float4 random_quat() {
    float u = random_float();
    float v = random_float();
    float w = random_float();
    return {
        sqrt(1.f - u) * sin(2.f * (float)M_PI * v),
        sqrt(1.f - u) * cos(2.f * (float)M_PI * v),
        sqrt(u) * sin(2.f * (float)M_PI * w),
        sqrt(u) * cos(2.f * (float)M_PI * w)};
}

float3 compute_conic(const float3 cov2d) {
    float det = cov2d.x * cov2d.z - cov2d.y * cov2d.y;
    if (det == 0.f)
        return {0.f, 0.f, 0.f};
    float inv_det = 1.f / det;
    float3 conic;
    conic.x = cov2d.z * inv_det;
    conic.y = -cov2d.y * inv_det;
    conic.z = cov2d.x * inv_det;
    return conic;
}

void compare_project2d_mean_backward() {
    float3 mean = {random_float(), random_float(), random_float()};
    // clang-format off
    float proj[] = {
        1.f, 0.f, 0.f, 0.f,
        0.f, 1.f, 0.f, 0.f,
        0.f, 0.f, 1.f, 0.f,
        0.f, 0.f, 0.f, 1.f
    };
    // clang-format on
    float2 dL_dmean2d = {random_float(), random_float()};

    float3 dL_dmean = project_pix_vjp(proj, mean, (dim3){1, 1, 1}, dL_dmean2d);
    float3 dL_dmean_ref = projectMean2DBackward(mean, proj, dL_dmean2d);
    printf("project2d backward\n");
    printf("dL_dmean\n");
    printf("ours %.2e %.2e %.2e\n", dL_dmean.x, dL_dmean.y, dL_dmean.z);
    printf(
        "ref %.2e %.2e %.2e\n", dL_dmean_ref.x, dL_dmean_ref.y, dL_dmean_ref.z
    );
}

void compare_conic_backward() {
    float3 cov2d = {random_float(), random_float(), random_float()};
    float3 conic = compute_conic(cov2d);
    float3 dL_dconic = {random_float(), random_float(), random_float()};
    float3 dL_dcov2d;
    cov2d_to_conic_vjp(conic, dL_dconic, dL_dcov2d);
    float3 dL_dcov2d_ref = computeConicBackward(cov2d, dL_dconic);

    printf("conic backward\n");
    printf("dL_dcov2d\n");
    printf("ours %.2e %.2e %.2e\n", dL_dcov2d.x, dL_dcov2d.y, dL_dcov2d.z);
    printf(
        "ref %.2e %.2e %.2e\n",
        dL_dcov2d_ref.x,
        dL_dcov2d_ref.y,
        dL_dcov2d_ref.z
    );
}

void compare_cov3d_backward() {
    float3 scale = {random_float(), random_float(), random_float()};
    float4 quat = random_quat();
    float4 quat_ref = {quat.w, quat.x, quat.y, quat.z};
    float dL_dcov3d[] = {
        random_float(),
        random_float(),
        random_float(),
        random_float(),
        random_float(),
        random_float()};
    float3 dL_ds = {0.f, 0.f, 0.f};
    float4 dL_dq = {0.f, 0.f, 0.f, 0.f};
    scale_rot_to_cov3d_vjp(scale, 1.f, quat, dL_dcov3d, dL_ds, dL_dq);

    float3 dL_ds_ref = {0.f, 0.f, 0.f};
    float4 dL_dq_ref = {0.f, 0.f, 0.f, 0.f};
    computeCov3DBackward(scale, 1.f, quat_ref, dL_dcov3d, dL_ds_ref, dL_dq_ref);

    printf("cov3d backward\n");
    printf("dL_dscale\n");
    printf("ours %.2e %.2e %.2e\n", dL_ds.x, dL_ds.y, dL_ds.z);
    printf("ref %.2e %.2e %.2e\n", dL_ds_ref.x, dL_ds_ref.y, dL_ds_ref.z);

    printf("dL_dquat\n");
    printf("ours %.2e %.2e %.2e %.2e\n", dL_dq.x, dL_dq.y, dL_dq.z, dL_dq.w);
    printf(
        "ref %.2e %.2e %.2e %.2e\n",
        dL_dq_ref.y,
        dL_dq_ref.z,
        dL_dq_ref.w,
        dL_dq_ref.x
    );
}

void compare_cov2d_ewa_backward() {
    float3 mean = {random_float(), random_float(), random_float()};
    float3 scale = {random_float(), random_float(), random_float()};
    float4 quat = random_quat();
    float cov3d[] = {0.f, 0.f, 0.f, 0.f, 0.f, 0.f};
    scale_rot_to_cov3d(scale, 1.f, quat, cov3d);
    float3 dL_dcov2d = {random_float(), random_float(), random_float()};
    float3 dL_dmean, dL_dmean_ref;
    float dL_dcov[] = {0.f, 0.f, 0.f, 0.f, 0.f, 0.f};
    float dL_dcov_ref[] = {0.f, 0.f, 0.f, 0.f, 0.f, 0.f};

    // functions expect different view matrix convention
    float viewmat[] = {
        1.f,
        0.f,
        0.f,
        0.f,
        0.f,
        1.f,
        0.f,
        0.f,
        0.f,
        0.f,
        1.f,
        10.f,
        0.f,
        0.f,
        0.f,
        1.f};
    float viewmat_ref[] = {
        1.f,
        0.f,
        0.f,
        0.f,
        0.f,
        1.f,
        0.f,
        0.f,
        0.f,
        0.f,
        1.f,
        0.f,
        0.f,
        0.f,
        10.f,
        1.f};
    computeCov2DBackward(
        mean,
        cov3d,
        viewmat_ref,
        1.f,
        1.f,
        1.f,
        1.f,
        dL_dcov2d,
        dL_dmean_ref,
        dL_dcov_ref
    );
    project_cov3d_ewa_vjp(
        mean, cov3d, viewmat, 1.f, 1.f, dL_dcov2d, dL_dmean, dL_dcov
    );
    printf("cov2d_ewa backward\n");
    printf("dL_dmean\n");
    printf("ours %.2e %.2e %.2e\n", dL_dmean.x, dL_dmean.y, dL_dmean.z);
    printf(
        "ref %.2e %.2e %.2e\n", dL_dmean_ref.x, dL_dmean_ref.y, dL_dmean_ref.z
    );
    printf("dL_dcov\nours ");
    for (int i = 0; i < 6; ++i) {
        printf("%.2e ", dL_dcov[i]);
    }
    printf("\ntheirs ");
    for (int i = 0; i < 6; ++i) {
        printf("%.2e ", dL_dcov[i]);
    }
    printf("\n");
}

void compare_rasterize_backward() {
    for (int x = 0; x < 100000; x++) {
        std::cout << "Step!! " << x << std::endl;
        int N = 8192 * 8;
        float2 p = {0.f, 0.f};
        float T_final = 5e-3;
        const int C = 3;
        float dL_dout[C];
        for (int i = 0; i < C; ++i) {
            dL_dout[i] = random_float();
        }

        float2 xys[N];
        float3 conics[N];
        float opacities[N];
        float4 conics_o[N];
        float rgbs[C * N];
        for (int i = 0; i < N; ++i) {
            float v = (float)i - (float)N * 0.5f;
            xys[i] = {0.1f * v, 0.1f * v};
            conics[i] = {10.f, 0.f, 10.f};
            opacities[i] = 0.5f;
            conics_o[i] = {conics[i].x, conics[i].y, conics[i].z, opacities[i]};
            for (int c = 0; c < C; ++c) {
                rgbs[C * i + c] = random_float();
            }
        }
        float dL_drgb[C * N] = {0.f};
        float dL_drgb_ref[C * N] = {0.f};
        float dL_do[N] = {0.f};
        float dL_do_ref[N] = {0.f};
        float2 dL_dm[N] = {0.f};
        float2 dL_dm_ref[N] = {0.f};
        float3 dL_dc[N] = {0.f};
        float3 dL_dc_ref[N] = {0.f};

        rasterize_vjp(
            N,
            p,
            C,
            xys,
            conics,
            opacities,
            rgbs,
            T_final,
            dL_dout,
            dL_drgb,
            dL_do,
            dL_dm,
            dL_dc
        );
        rasterizeBackward(
            N,
            2,
            2,
            p,
            xys,
            conics_o,
            rgbs,
            T_final,
            dL_dout,
            dL_drgb_ref,
            dL_do_ref,
            dL_dm_ref,
            dL_dc_ref
        );
        printf("rasterize backward\n");
        printf("dL_dout %.2e %.2e %.2e\n", dL_dout[0], dL_dout[1], dL_dout[2]);
        for (int i = 0; i < N; ++i) {
            const auto something_went_wrong = [&]() {
                printf("\n");
                printf("\n");
                printf("Something went wrong!");
                printf("\n");
                printf("\n");
                printf("dL_drgb %d\n", i);
                printf(
                    "ours %.2e %.2e %.2e\n",
                    dL_drgb[C * i + 0],
                    dL_drgb[C * i + 1],
                    dL_drgb[C * i + 2]
                );
                printf(
                    "theirs %.2e %.2e %.2e\n",
                    dL_drgb_ref[C * i + 0],
                    dL_drgb_ref[C * i + 1],
                    dL_drgb_ref[C * i + 2]
                );
                printf("\n");
                printf("dL_do %d\n", i);
                printf("ours %.2e\n", dL_do[i]);
                printf("theirs %.2e\n", dL_do_ref[i]);
                printf("\n");
                printf("dL_dm %d\n", i);
                printf("ours %.2e %.2e\n", dL_dm[i].x, dL_dm[i].y);
                printf("theirs %.2e %.2e\n", dL_dm_ref[i].x, dL_dm_ref[i].y);
                printf("\n");
                printf("dL_dc %d\n", i);
                printf(
                    "ours %.2e %.2e %.2e\n", dL_dc[i].x, dL_dc[i].y, dL_dc[i].z
                );
                printf(
                    "theirs %.2e %.2e %.2e\n",
                    dL_dc_ref[i].x,
                    dL_dc_ref[i].y,
                    dL_dc_ref[i].z
                );
                printf("\n");
            };

            if (abs(dL_drgb[C * i + 0] - dL_drgb_ref[C * i + 0]) > 1e-3) {
                something_went_wrong();
                assert(false);
            }
            if (abs(dL_drgb[C * i + 1] - dL_drgb_ref[C * i + 1]) > 1e-3) {
                something_went_wrong();
                assert(false);
            }
            if (abs(dL_drgb[C * i + 2] - dL_drgb_ref[C * i + 2]) > 1e-3) {
                something_went_wrong();
                assert(false);
            }

            if (abs(dL_do[i] - dL_do_ref[i]) > 1e-3) {
                something_went_wrong();
                assert(false);
            }
            if (abs(dL_do[i] - dL_do_ref[i]) > 1e-3) {
                something_went_wrong();
                assert(false);
            }

            if (abs(dL_dm[i].x - dL_dm_ref[i].x) > 1e-3) {
                something_went_wrong();
                assert(false);
            }
            if (abs(dL_dm[i].y - dL_dm_ref[i].y) > 1e-3) {
                something_went_wrong();
                assert(false);
            }

            if (abs(dL_dc[i].x - dL_dc_ref[i].x) > 1e-3) {
                something_went_wrong();
                assert(false);
            }
            if (abs(dL_dc[i].y - dL_dc_ref[i].y) > 1e-3) {
                something_went_wrong();
                assert(false);
            }
            if (abs(dL_dc[i].z - dL_dc_ref[i].z) > 1e-3) {
                something_went_wrong();
                assert(false);
            }
        }
    }
}

int main() {
    // compare_project2d_mean_backward();
    compare_conic_backward();
    // compare_cov3d_backward();
    // compare_cov2d_ewa_backward();
    compare_rasterize_backward();
    return 0;
}
