#include "hip/hip_runtime.h"
/*
 * Copyright (C) 2023, Inria
 * GRAPHDECO research group, https://team.inria.fr/graphdeco
 * All rights reserved.
 *
 * This software is free for non-commercial, research and evaluation use 
 * under the terms of the LICENSE.md file.
 *
 * For inquiries contact  george.drettakis@inria.fr
 */

#include "rasterizer_impl.h"
#include <iostream>
#include <fstream>
#include <algorithm>
#include <numeric>
#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include ""
#include <hipcub/hipcub.hpp>
#include <cub/device/device_radix_sort.cuh>
#define GLM_FORCE_CUDA
#include <glm/glm.hpp>

#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/reduce.h>
namespace cg = cooperative_groups;

#include "auxiliary.h"
#include "forward.h"
#include "backward.h"

// Helper function to find the next-highest bit of the MSB
// on the CPU.
uint32_t getHigherMsb(uint32_t n)
{
	uint32_t msb = sizeof(n) * 4;
	uint32_t step = msb;
	while (step > 1)
	{
		step /= 2;
		if (n >> msb)
			msb += step;
		else
			msb -= step;
	}
	if (n >> msb)
		msb++;
	return msb;
}

// Wrapper method to call auxiliary coarse frustum containment test.
// Mark all Gaussians that pass it.
__global__ void checkFrustum(int P,
	const float* orig_points,
	const float* viewmatrix,
	const float* projmatrix,
	bool* present)
{
	auto idx = cg::this_grid().thread_rank();
	if (idx >= P)
		return;

	float3 p_view;
	present[idx] = in_frustum(idx, orig_points, viewmatrix, projmatrix, false, p_view);
}

// Generates one key/value pair for all Gaussian / tile overlaps. 
// Run once per Gaussian (1:N mapping).
__global__ void duplicateWithKeys(
	int P,
	const float2* points_xy,
	const float* depths,
	const uint32_t* offsets,
	uint64_t* gaussian_keys_unsorted,
	uint32_t* gaussian_values_unsorted,
	int* radii,
	dim3 grid)
{
	auto idx = cg::this_grid().thread_rank();
	if (idx >= P)
		return;

	// Generate no key/value pair for invisible Gaussians
	if (radii[idx] > 0)
	{
		// Find this Gaussian's offset in buffer for writing keys/values.
		uint32_t off = (idx == 0) ? 0 : offsets[idx - 1];
		uint2 rect_min, rect_max;

		getRect(points_xy[idx], radii[idx], rect_min, rect_max, grid);

		// For each tile that the bounding rect overlaps, emit a 
		// key/value pair. The key is |  tile ID  |      depth      |,
		// and the value is the ID of the Gaussian. Sorting the values 
		// with this key yields Gaussian IDs in a list, such that they
		// are first sorted by tile and then by depth. 
		for (int y = rect_min.y; y < rect_max.y; y++)
		{
			for (int x = rect_min.x; x < rect_max.x; x++)
			{
				uint64_t key = y * grid.x + x;
				key <<= 32;
				key |= *((uint32_t*)&depths[idx]);
				gaussian_keys_unsorted[off] = key;
				gaussian_values_unsorted[off] = idx;
				off++;
			}
		}
	}
}

// Check keys to see if it is at the start/end of one tile's range in 
// the full sorted list. If yes, write start/end of this tile. 
// Run once per instanced (duplicated) Gaussian ID.
__global__ void identifyTileRanges(int L, uint64_t* point_list_keys, uint2* ranges)
{
	auto idx = cg::this_grid().thread_rank();
	if (idx >= L)
		return;

	// Read tile ID from key. Update start/end of tile range if at limit.
	uint64_t key = point_list_keys[idx];
	uint32_t currtile = key >> 32;
	if (idx == 0)
		ranges[currtile].x = 0;
	else
	{
		uint32_t prevtile = point_list_keys[idx - 1] >> 32;
		if (currtile != prevtile)
		{
			ranges[prevtile].y = idx;
			ranges[currtile].x = idx;
		}
	}
	if (idx == L - 1)
		ranges[currtile].y = L;
}

// Mark Gaussians as visible/invisible, based on view frustum testing
void CudaRasterizer::Rasterizer::markVisible(
	int P,
	float* means3D,
	float* viewmatrix,
	float* projmatrix,
	bool* present)
{
	checkFrustum << <(P + 255) / 256, 256 >> > (
		P,
		means3D,
		viewmatrix, projmatrix,
		present);
}

CudaRasterizer::GeometryState CudaRasterizer::GeometryState::fromChunk(char*& chunk, size_t P)
{
	GeometryState geom;
	obtain(chunk, geom.depths, P, 128);
	obtain(chunk, geom.clamped, P * 3, 128);
	obtain(chunk, geom.internal_radii, P, 128);
	obtain(chunk, geom.means2D, P, 128);
	obtain(chunk, geom.cov3D, P * 6, 128);
	obtain(chunk, geom.conic_opacity, P, 128);
	obtain(chunk, geom.rgb, P * 3, 128);
	obtain(chunk, geom.tiles_touched, P, 128);
	hipcub::DeviceScan::InclusiveSum(nullptr, geom.scan_size, geom.tiles_touched, geom.tiles_touched, P);
	obtain(chunk, geom.scanning_space, geom.scan_size, 128);
	obtain(chunk, geom.point_offsets, P, 128);
	return geom;
}

CudaRasterizer::ImageState CudaRasterizer::ImageState::fromChunk(char*& chunk, size_t N)
{
	ImageState img;
	obtain(chunk, img.accum_alpha, N, 128);
	obtain(chunk, img.n_contrib, N, 128);
	obtain(chunk, img.ranges, N, 128);
	return img;
}

CudaRasterizer::BinningState CudaRasterizer::BinningState::fromChunk(char*& chunk, size_t P)
{
	BinningState binning;
	obtain(chunk, binning.point_list, P, 128);
	obtain(chunk, binning.point_list_unsorted, P, 128);
	obtain(chunk, binning.point_list_keys, P, 128);
	obtain(chunk, binning.point_list_keys_unsorted, P, 128);
	hipcub::DeviceRadixSort::SortPairs(
		nullptr, binning.sorting_size,
		binning.point_list_keys_unsorted, binning.point_list_keys,
		binning.point_list_unsorted, binning.point_list, P);
	obtain(chunk, binning.list_sorting_space, binning.sorting_size, 128);
	return binning;
}

// Forward rendering procedure for differentiable rasterization
// of Gaussians.
int CudaRasterizer::Rasterizer::forward(
	std::function<char* (size_t)> geometryBuffer,
	std::function<char* (size_t)> binningBuffer,
	std::function<char* (size_t)> imageBuffer,
	const int P, int D, int M,
	const float* background,
	const int width, int height,
	const float* means3D,
	const float* shs,
	const float* colors_precomp,
	const float* opacities,
	const float* scales,
	const float scale_modifier,
	const float* rotations,
	const float* cov3D_precomp,
	const float* viewmatrix,
	const float* projmatrix,
	const float* cam_pos,
	const float tan_fovx, float tan_fovy,
	const bool prefiltered,
	float* out_color,
	int* radii,
	bool debug)
{
	const float focal_y = height / (2.0f * tan_fovy);
	const float focal_x = width / (2.0f * tan_fovx);

	size_t chunk_size = required<GeometryState>(P);
	char* chunkptr = geometryBuffer(chunk_size);
	GeometryState geomState = GeometryState::fromChunk(chunkptr, P);

	if (radii == nullptr)
	{
		radii = geomState.internal_radii;
	}

	dim3 tile_grid((width + BLOCK_X - 1) / BLOCK_X, (height + BLOCK_Y - 1) / BLOCK_Y, 1);
	dim3 block(BLOCK_X, BLOCK_Y, 1);

	// Dynamically resize image-based auxiliary buffers during training
	size_t img_chunk_size = required<ImageState>(width * height);
	char* img_chunkptr = imageBuffer(img_chunk_size);
	ImageState imgState = ImageState::fromChunk(img_chunkptr, width * height);

	if (NUM_CHANNELS != 3 && colors_precomp == nullptr)
	{
		throw std::runtime_error("For non-RGB, provide precomputed Gaussian colors!");
	}

	// Run preprocessing per-Gaussian (transformation, bounding, conversion of SHs to RGB)
	CHECK_CUDA(FORWARD::preprocess(
		P, D, M,
		means3D,
		(glm::vec3*)scales,
		scale_modifier,
		(glm::vec4*)rotations,
		opacities,
		shs,
		geomState.clamped,
		cov3D_precomp,
		colors_precomp,
		viewmatrix, projmatrix,
		(glm::vec3*)cam_pos,
		width, height,
		focal_x, focal_y,
		tan_fovx, tan_fovy,
		radii,
		geomState.means2D,
		geomState.depths,
		geomState.cov3D,
		geomState.rgb,
		geomState.conic_opacity,
		tile_grid,
		geomState.tiles_touched,
		prefiltered
	), debug)

	// Compute prefix sum over full list of touched tile counts by Gaussians
	// E.g., [2, 3, 0, 2, 1] -> [2, 5, 5, 7, 8]
	CHECK_CUDA(hipcub::DeviceScan::InclusiveSum(geomState.scanning_space, geomState.scan_size, geomState.tiles_touched, geomState.point_offsets, P), debug)

	// Retrieve total number of Gaussian instances to launch and resize aux buffers
	int num_rendered;
	CHECK_CUDA(hipMemcpy(&num_rendered, geomState.point_offsets + P - 1, sizeof(int), hipMemcpyDeviceToHost), debug);

	size_t binning_chunk_size = required<BinningState>(num_rendered);
	char* binning_chunkptr = binningBuffer(binning_chunk_size);
	BinningState binningState = BinningState::fromChunk(binning_chunkptr, num_rendered);

	// For each instance to be rendered, produce adequate [ tile | depth ] key 
	// and corresponding dublicated Gaussian indices to be sorted
	duplicateWithKeys << <(P + 255) / 256, 256 >> > (
		P,
		geomState.means2D,
		geomState.depths,
		geomState.point_offsets,
		binningState.point_list_keys_unsorted,
		binningState.point_list_unsorted,
		radii,
		tile_grid)
	CHECK_CUDA(, debug)

	int bit = getHigherMsb(tile_grid.x * tile_grid.y);

	// Sort complete list of (duplicated) Gaussian indices by keys
	CHECK_CUDA(hipcub::DeviceRadixSort::SortPairs(
		binningState.list_sorting_space,
		binningState.sorting_size,
		binningState.point_list_keys_unsorted, binningState.point_list_keys,
		binningState.point_list_unsorted, binningState.point_list,
		num_rendered, 0, 32 + bit), debug)

	CHECK_CUDA(hipMemset(imgState.ranges, 0, tile_grid.x * tile_grid.y * sizeof(uint2)), debug);

	// Identify start and end of per-tile workloads in sorted list
	if (num_rendered > 0)
		identifyTileRanges << <(num_rendered + 255) / 256, 256 >> > (
			num_rendered,
			binningState.point_list_keys,
			imgState.ranges);
	CHECK_CUDA(, debug)

	// Let each tile blend its range of Gaussians independently in parallel
	const float* feature_ptr = colors_precomp != nullptr ? colors_precomp : geomState.rgb;
	CHECK_CUDA(FORWARD::render(
		tile_grid, block,
		imgState.ranges,
		binningState.point_list,
		width, height,
		geomState.means2D,
		feature_ptr,
		geomState.conic_opacity,
		imgState.accum_alpha,
		imgState.n_contrib,
		background,
		out_color), debug)

	return num_rendered;
}

// Produce necessary gradients for optimization, corresponding
// to forward render pass
void CudaRasterizer::Rasterizer::backward(
	const int P, int D, int M, int R,
	const float* background,
	const int width, int height,
	const float* means3D,
	const float* shs,
	const float* colors_precomp,
	const float* scales,
	const float scale_modifier,
	const float* rotations,
	const float* cov3D_precomp,
	const float* viewmatrix,
	const float* projmatrix,
	const float* campos,
	const float tan_fovx, float tan_fovy,
	const int* radii,
	char* geom_buffer,
	char* binning_buffer,
	char* img_buffer,
	const float* dL_dpix,
	float* dL_dmean2D,
	float* dL_dconic,
	float* dL_dopacity,
	float* dL_dcolor,
	float* dL_dmean3D,
	float* dL_dcov3D,
	float* dL_dsh,
	float* dL_dscale,
	float* dL_drot,
	bool debug)
{
	GeometryState geomState = GeometryState::fromChunk(geom_buffer, P);
	BinningState binningState = BinningState::fromChunk(binning_buffer, R);
	ImageState imgState = ImageState::fromChunk(img_buffer, width * height);

	if (radii == nullptr)
	{
		radii = geomState.internal_radii;
	}

	const float focal_y = height / (2.0f * tan_fovy);
	const float focal_x = width / (2.0f * tan_fovx);

	const dim3 tile_grid((width + BLOCK_X - 1) / BLOCK_X, (height + BLOCK_Y - 1) / BLOCK_Y, 1);
	const dim3 block(BLOCK_X, BLOCK_Y, 1);

	// Compute loss gradients w.r.t. 2D mean position, conic matrix,
	// opacity and RGB of Gaussians from per-pixel loss gradients.
	// If we were given precomputed colors and not SHs, use them.
	const float* color_ptr = (colors_precomp != nullptr) ? colors_precomp : geomState.rgb;
	CHECK_CUDA(BACKWARD::render(
		tile_grid,
		block,
		imgState.ranges,
		binningState.point_list,
		width, height,
		background,
		geomState.means2D,
		geomState.conic_opacity,
		color_ptr,
		imgState.accum_alpha,
		imgState.n_contrib,
		dL_dpix,
		(float3*)dL_dmean2D,
		(float4*)dL_dconic,
		dL_dopacity,
		dL_dcolor), debug)

	// Take care of the rest of preprocessing. Was the precomputed covariance
	// given to us or a scales/rot pair? If precomputed, pass that. If not,
	// use the one we computed ourselves.
	const float* cov3D_ptr = (cov3D_precomp != nullptr) ? cov3D_precomp : geomState.cov3D;
	CHECK_CUDA(BACKWARD::preprocess(P, D, M,
		(float3*)means3D,
		radii,
		shs,
		geomState.clamped,
		(glm::vec3*)scales,
		(glm::vec4*)rotations,
		scale_modifier,
		cov3D_ptr,
		viewmatrix,
		projmatrix,
		focal_x, focal_y,
		tan_fovx, tan_fovy,
		(glm::vec3*)campos,
		(float3*)dL_dmean2D,
		dL_dconic,
		(glm::vec3*)dL_dmean3D,
		dL_dcolor,
		dL_dcov3D,
		dL_dsh,
		(glm::vec3*)dL_dscale,
		(glm::vec4*)dL_drot), debug)
}