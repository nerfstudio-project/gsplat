#include "hip/hip_runtime.h"
#include "forward.cuh"
#include "backward.cuh"
#include "helpers.cuh"
#include "rasterize.h"
#include <hipcub/hipcub.hpp>
#include <cub/device/device_radix_sort.cuh>

#define CHECK_CUDA(x)                                                          \
    AT_ASSERTM(x.type().is_cuda(), #x " must be a CUDA tensor")
#define CHECK_CONTIGUOUS(x)                                                    \
    AT_ASSERTM(x.is_contiguous(), #x " must be contiguous")
#define CHECK_INPUT(x)                                                         \
    CHECK_CUDA(x);                                                             \
    CHECK_CONTIGUOUS(x)

// function to interface torch tensors and lower level pointer operations
std::
    tuple<
        int,
        torch::Tensor, // output image
        torch::Tensor,  // output radii
        torch::Tensor, // final_Ts
        torch::Tensor, // final_idx
        torch::Tensor, // gaussian_ids_sorted 
        torch::Tensor, // tile_bins
        torch::Tensor, // xys
        torch::Tensor  // conics
        >
    rasterize_forward_tensor(
        const torch::Tensor &means3d,
        const torch::Tensor &scales,
        const float glob_scale,
        const torch::Tensor &rotations_quat,
        const torch::Tensor &colors,
        const torch::Tensor &opacity,
        const torch::Tensor &view_matrix,
        const torch::Tensor &proj_matrix,
        const int img_height,
        const int img_width,
        const float fx,
        const float fy
    ) {
    CHECK_INPUT(means3d);
    CHECK_INPUT(scales);
    CHECK_INPUT(rotations_quat);
    CHECK_INPUT(colors);
    CHECK_INPUT(opacity);
    CHECK_INPUT(view_matrix);
    CHECK_INPUT(proj_matrix);

    if (means3d.ndimension() != 2 || means3d.size(1) != 3) {
        AT_ERROR("means3d must have dimensions (num_points, 3)");
    }

    int rendered = 0;
    const int num_points = means3d.size(0);
    auto int_opts = means3d.options().dtype(torch::kInt32);
    auto float_opts = means3d.options().dtype(torch::kFloat32);


    torch::Tensor out_img = torch::zeros(
        {img_height, img_width, 3}, means3d.options().dtype(torch::kFloat32)
    );
    torch::Tensor out_radii =
        torch::zeros({num_points}, means3d.options().dtype(torch::kFloat32));

    torch::Tensor final_Ts =
        torch::zeros({img_width,img_height}, means3d.options().dtype(torch::kFloat32));

    torch::Tensor final_idx =
        torch::zeros({img_width,img_height}, means3d.options().dtype(torch::kInt32));

    torch::Tensor gaussian_ids_sorted =
        torch::zeros({img_width,img_height}, means3d.options().dtype(torch::kInt32));
    
    // TILE BINS
    const dim3 tile_bounds = {
        (img_width + BLOCK_X - 1) / BLOCK_X,
        (img_height + BLOCK_Y - 1) / BLOCK_Y,
        1};

    int num_tiles = tile_bounds.x * tile_bounds.y;

    torch::Tensor tile_bins =  torch::zeros({num_tiles, 1}, means3d.options().dtype(torch::kInt32));
    
    torch::Tensor xy =
        torch::zeros({num_points, 2}, means3d.options().dtype(torch::kFloat32));

    torch::Tensor conics =
        torch::zeros({num_points, 6}, means3d.options().dtype(torch::kFloat32));

    rasterize_forward_impl(
        num_points,
        means3d.contiguous().data_ptr<float>(),
        scales.contiguous().data_ptr<float>(),
        glob_scale,
        rotations_quat.contiguous().data_ptr<float>(),
        colors.contiguous().data_ptr<float>(),
        opacity.contiguous().data_ptr<float>(),
        view_matrix.contiguous().data_ptr<float>(),
        proj_matrix.contiguous().data_ptr<float>(),
        img_height,
        img_width,
        fx,
        fy,
        out_img.contiguous().data_ptr<float>(),
        out_radii.contiguous().data_ptr<float>(),
        final_Ts.contiguous().data_ptr<float>(),
        final_idx.contiguous().data_ptr<int>(),
        gaussian_ids_sorted.contiguous().data_ptr<int>(),
        tile_bins.contiguous().data_ptr<int>(),
        xy.contiguous().data_ptr<float>(),
        conics.contiguous().data_ptr<float>()
    );

    return std::make_tuple(rendered, out_img, out_radii, final_Ts, final_idx, gaussian_ids_sorted, tile_bins, xy, conics);
}

int rasterize_forward_impl(
    const int num_points,
    const float *means3d,
    const float *scales,
    const float glob_scale,
    const float *quats,
    const float *colors,
    const float *opacity,
    const float *view_matrix,
    const float *proj_matrix,
    const int img_height,
    const int img_width,
    const float fx,
    const float fy,
    float *out_img,
    float *out_radii,
    float *final_Ts,
    int *final_idx,
    int *gaussian_ids_sorted,
    int *tile_bins,
    float *xy,
    float *conics
) {
    const int channels = 3; // TODO: make this a var
    const int W = img_width;
    const int H = img_height;

    // launch projection of 3d gaussians into 2d
    // project_gaussians_forward_impl(...)
    float3 *scales_d, *means_d;
    float *rgbs_d;
    float4 *quats_d;
    float *viewmat_d;
    float *opacities_d;
    int num_view = 16; // 16 entries in 4x4 projection matrix
    hipMalloc((void **)&scales_d, num_points * sizeof(float3));
    hipMalloc((void **)&means_d, num_points * sizeof(float3));
    hipMalloc((void **)&quats_d, num_points * sizeof(float4));
    hipMalloc((void **)&rgbs_d, num_points * sizeof(float) * channels);
    hipMalloc((void **)&opacities_d, num_points * sizeof(float));
    hipMalloc((void **)&viewmat_d, num_view * sizeof(float));

    hipMemcpy(
        scales_d, scales, num_points * sizeof(float3), hipMemcpyHostToDevice
    );
    hipMemcpy(
        means_d, means3d, num_points * sizeof(float3), hipMemcpyHostToDevice
    );
    hipMemcpy(
        rgbs_d, colors, num_points * sizeof(float) * channels, hipMemcpyHostToDevice
    );
    hipMemcpy(
        opacities_d, opacity, num_points * sizeof(float), hipMemcpyHostToDevice
    );
    hipMemcpy(
        quats_d, quats, num_points * sizeof(float4), hipMemcpyHostToDevice
    );
    hipMemcpy(
        viewmat_d, view_matrix, num_view * sizeof(float), hipMemcpyHostToDevice
    );

    // allocate memory for outputs
    int num_cov3d = num_points * 6;
    float *covs3d = new float[num_cov3d];
    //float2 *xy = new float2[num_points]; // 2d mean locations
    float *z = new float[num_points]; // depths
    // float3 *conics = new float3[num_points]; // inverse of cov2ds
    int32_t *num_tiles_hit = new int32_t[num_points]; // num_tiles_hit[gauss_idx]=tile_area i.e. tiles that 2D gaussian projects to within 3 stds. 

    float *covs3d_d, *z_d;
    float2 *xy_d;
    float3 *conics_d;
    int *radii_d; // radius of 2D gaussians in screen space
    int32_t *num_tiles_hit_d;
    hipMalloc((void **)&covs3d_d, num_cov3d * sizeof(float));
    hipMalloc((void **)&xy_d, num_points * sizeof(float2));
    hipMalloc((void **)&z_d, num_points * sizeof(float));
    hipMalloc((void **)&radii_d, num_points * sizeof(int));
    hipMalloc((void **)&conics_d, num_points * sizeof(float3));
    hipMalloc((void **)&num_tiles_hit_d, num_points * sizeof(int32_t));

    const dim3 img_size = {img_width, img_height, 1};
    const dim3 tile_bounds = {
        (img_width + BLOCK_X - 1) / BLOCK_X,
        (img_height + BLOCK_Y - 1) / BLOCK_Y,
        1};
    project_gaussians_forward_impl(
        num_points,
        means_d,
        scales_d,
        1.f,
        quats_d,
        viewmat_d,
        viewmat_d,
        fx,
        fy,
        img_size,
        tile_bounds,
        covs3d_d,
        xy_d,
        z_d,
        radii_d,
        conics_d,
        num_tiles_hit_d
    );
    hipMemcpy(
        covs3d, covs3d_d, num_cov3d * sizeof(float), hipMemcpyDeviceToHost
    );
    hipMemcpy(xy, xy_d, num_points * sizeof(float2), hipMemcpyDeviceToHost);
    hipMemcpy(z, z_d, num_points * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(
        out_radii, radii_d, num_points * sizeof(int), hipMemcpyDeviceToHost
    );
    hipMemcpy(
        num_tiles_hit,
        num_tiles_hit_d,
        num_points * sizeof(int32_t),
        hipMemcpyDeviceToHost
    );

    int32_t num_intersects;
    int32_t *cum_tiles_hit = new int32_t[num_points];
    int32_t *cum_tiles_hit_d;
    hipMalloc((void **)&cum_tiles_hit_d, num_points * sizeof(int32_t));
    compute_cumulative_intersects(
        num_points, num_tiles_hit_d, num_intersects, cum_tiles_hit_d
    );

    int64_t *isect_ids_sorted_d;
    int32_t *gaussian_ids_sorted_d; // sorted by tile and depth
    int64_t *isect_ids_sorted = new int64_t[num_intersects];
    //int32_t *gaussian_ids_sorted = new int32_t[num_intersects];
    hipMalloc((void **)&isect_ids_sorted_d, num_intersects * sizeof(int64_t));
    hipMalloc(
        (void **)&gaussian_ids_sorted_d, num_intersects * sizeof(int32_t)
    );

    int64_t *isect_ids_unsorted_d;
    int32_t *gaussian_ids_unsorted_d; // sorted by tile and depth
    int64_t *isect_ids_unsorted = new int64_t[num_intersects];
    int32_t *gaussian_ids_unsorted = new int32_t[num_intersects];
    hipMalloc(
        (void **)&isect_ids_unsorted_d, num_intersects * sizeof(int64_t)
    );
    hipMalloc(
        (void **)&gaussian_ids_unsorted_d, num_intersects * sizeof(int32_t)
    );

    int num_tiles = tile_bounds.x * tile_bounds.y;
    uint2 *tile_bins_d; // start and end indices for each tile
    //uint2 *tile_bins = new uint2[num_tiles];
    hipMalloc((void **)&tile_bins_d, num_tiles * sizeof(uint2));

    bin_and_sort_gaussians(
        num_points,
        num_intersects,
        xy_d,
        z_d,
        radii_d,
        cum_tiles_hit_d,
        tile_bounds,
        isect_ids_unsorted_d,
        gaussian_ids_unsorted_d,
        isect_ids_sorted_d,
        gaussian_ids_sorted_d,
        tile_bins_d
    );

    hipMemcpy(
        isect_ids_unsorted,
        isect_ids_unsorted_d,
        num_intersects * sizeof(int64_t),
        hipMemcpyDeviceToHost
    );
    hipMemcpy(
        gaussian_ids_unsorted,
        gaussian_ids_unsorted_d,
        num_intersects * sizeof(int32_t),
        hipMemcpyDeviceToHost
    );
    hipMemcpy(
        isect_ids_sorted,
        isect_ids_sorted_d,
        num_intersects * sizeof(int64_t),
        hipMemcpyDeviceToHost
    );
    hipMemcpy(
        gaussian_ids_sorted,
        gaussian_ids_sorted_d,
        num_intersects * sizeof(int32_t),
        hipMemcpyDeviceToHost
    );

    // launch final rasterization method
    // rasterize_forward_impl(...)
    float *final_Ts_d;
    int *final_idx_d;
    float *out_img_d;
    hipMalloc((void **)&out_img_d, W * H * sizeof(float)*channels);
    hipMalloc((void **)&final_Ts_d, W * H * sizeof(float));
    hipMalloc((void **)&final_idx_d, W * H * sizeof(int));

    const dim3 block = {
        BLOCK_X, BLOCK_Y, 1}; // TODO: make this a user custom setting.

    

    rasterize_forward_impl( // Should this be renamed? it is overloaded with two implementations
        tile_bounds,
        block,
        img_size,
        channels,
        gaussian_ids_sorted_d,
        tile_bins_d,
        xy_d,
        conics_d,
        rgbs_d,
        opacities_d,
        final_Ts_d,
        final_idx_d,
        out_img_d
    );

    // Handle outputs
    hipMemcpy(
        out_img, out_img_d, W * H * sizeof(float3), hipMemcpyDeviceToHost
    );
    hipMemcpy(
        final_Ts, final_Ts_d, W * H * sizeof(float3), hipMemcpyDeviceToHost
    );
    hipMemcpy(
        final_idx, final_idx_d, W * H * sizeof(int), hipMemcpyDeviceToHost
    );
    hipMemcpy(
        conics, conics_d, num_points * sizeof(float3), hipMemcpyDeviceToHost
    );
    hipMemcpy(
        tile_bins, tile_bins_d, num_tiles * sizeof(uint2), hipMemcpyDeviceToHost
    );

    return 0;
}


std::
    tuple<
        torch::Tensor, // dL_dcolors
        torch::Tensor // dL_dopacity
        >
    rasterize_backward_tensor(
        const torch::Tensor &means3D,
        const torch::Tensor &colors,
        const torch::Tensor &scales,
        const torch::Tensor &v_output, //dL_dout_color
        const int img_height,
        const int img_width,
        const float fx,
        const float fy,
        const torch::Tensor gaussians_ids_sorted,
        const torch::Tensor tile_bins,
        const torch::Tensor xy,
        const torch::Tensor conics,
        const torch::Tensor opacities,
        const torch::Tensor final_Ts,
        const torch::Tensor final_idx
    ){
    
        CHECK_INPUT(means3D);
        CHECK_INPUT(scales);
        CHECK_INPUT(colors);

        const int num_points = means3D.size(0);
        const int H = v_output.size(1);
        const int W = v_output.size(2);

        torch::Tensor v_xy = torch::zeros({num_points, 3}, means3D.options());
        torch::Tensor v_colors = torch::zeros({num_points, 3}, means3D.options());
        torch::Tensor v_conic = torch::zeros({num_points, 2, 2}, means3D.options());
        torch::Tensor v_opacity = torch::zeros({num_points, 1}, means3D.options());

        const dim3 tile_bounds = {
            (img_width + BLOCK_X - 1) / BLOCK_X,
            (img_height + BLOCK_Y - 1) / BLOCK_Y,
            1};

        if (means3D.ndimension() != 2 || means3D.size(1) != 3) {
            AT_ERROR("means3d must have dimensions (num_points, 3)");
        }
        
	    const dim3 block(BLOCK_X, BLOCK_Y, 1);

        const dim3 img_size = {img_width,img_height,1};
        
        const int channels = 3; // TODO: make this a user var

        rasterize_backward_impl(
            tile_bounds,
            block,
            img_size,
            channels,
            gaussians_ids_sorted.contiguous().data_ptr<int>(),
            (uint2 *)tile_bins.contiguous().data_ptr<int>(), 
            (float2 *)xy.contiguous().data_ptr<float>(),
            (float3 *)conics.contiguous().data_ptr<float>(),
            colors.contiguous().data_ptr<float>(),
            opacities.contiguous().data_ptr<float>(),
            final_Ts.contiguous().data_ptr<float>(),
            final_idx.contiguous().data_ptr<int>(),
            v_output.contiguous().data_ptr<float>(),
            (float2 *)v_xy.contiguous().data_ptr<float>(),
            (float3 *)v_conic.contiguous().data_ptr<float>(),
            v_colors.contiguous().data_ptr<float>(),
            v_opacity.contiguous().data_ptr<float>());

        return std::make_tuple(v_colors,v_opacity);
    }