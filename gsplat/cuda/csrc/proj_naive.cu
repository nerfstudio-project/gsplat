#include "hip/hip_runtime.h"
#include "proj_naive.h"
#include "utils.cuh"

#include <hip/hip_cooperative_groups.h>

namespace cg = cooperative_groups;

/****************************************************************************
 * Projection Forward Pass
 ****************************************************************************/

__global__ void proj_naive_fwd_kernel(
    const uint32_t C,
    const uint32_t N,
    const float *__restrict__ means,  // [C, N, 3]
    const float *__restrict__ covars, // [C, N, 3, 3]
    const float *__restrict__ Ks,     // [C, 3, 3]
    const uint32_t width,
    const uint32_t height,
    const CameraModelType camera_model,
    float *__restrict__ means2d, // [C, N, 2]
    float *__restrict__ covars2d // [C, N, 2, 2]
) {
    // parallelize over C * N.
    uint32_t idx = cg::this_grid().thread_rank();
    if (idx >= C * N) {
        return;
    }
    const uint32_t cid = idx / N; // camera id
    // const uint32_t gid = idx % N; // gaussian id

    // shift pointers to the current camera and gaussian
    means += idx * 3;
    covars += idx * 9;
    Ks += cid * 9;
    means2d += idx * 2;
    covars2d += idx * 4;

    float fx = Ks[0], cx = Ks[2], fy = Ks[4], cy = Ks[5];
    mat2 covar2d(0.f);
    vec2 mean2d(0.f);
    const vec3 mean = glm::make_vec3(means);
    const mat3 covar = glm::make_mat3(covars);

    switch (camera_model) {
        case CameraModelType::PINHOLE: // perspective projection
            persp_proj(mean, covar, fx, fy, cx, cy, width, height, covar2d, mean2d);
            break;
        case CameraModelType::ORTHO: // orthographic projection
            ortho_proj(mean, covar, fx, fy, cx, cy, width, height, covar2d, mean2d);
            break;
        case CameraModelType::FISHEYE: // fisheye projection
            fisheye_proj(mean, covar, fx, fy, cx, cy, width, height, covar2d, mean2d);
            break;
    }

    // write to outputs: glm is column-major but we want row-major
    #pragma unroll
    for (uint32_t i = 0; i < 2; i++) { // rows
        #pragma unroll
        for (uint32_t j = 0; j < 2; j++) { // cols
            covars2d[i * 2 + j] = covar2d[j][i];
        }
    }
    #pragma unroll
    for (uint32_t i = 0; i < 2; i++) {
        means2d[i] = mean2d[i];
    }
}

void proj_naive_fwd_launcher(
    uint32_t shmem_size, 
    hipStream_t stream, 
    uint32_t n_elements, 
    // args
    const uint32_t C,
    const uint32_t N,
    const float *__restrict__ means,  // [C, N, 3]
    const float *__restrict__ covars, // [C, N, 3, 3]
    const float *__restrict__ Ks,     // [C, 3, 3]
    const uint32_t width,
    const uint32_t height,
    const CameraModelType camera_model,
    float *__restrict__ means2d, // [C, N, 2]
    float *__restrict__ covars2d // [C, N, 2, 2]
) {
    if (n_elements <= 0) {
        return;
    }
    proj_naive_fwd_kernel<<<n_blocks_linear(n_elements), N_THREADS, shmem_size, stream>>>(
        C,
        N,
        means,
        covars,
        Ks,
        width,
        height,
        camera_model,
        means2d,
        covars2d
    );
}


/****************************************************************************
 * Projection Backward Pass
 ****************************************************************************/

 __global__ void proj_naive_bwd_kernel(
    const uint32_t C,
    const uint32_t N,
    const float *__restrict__ means,  // [C, N, 3]
    const float *__restrict__ covars, // [C, N, 3, 3]
    const float *__restrict__ Ks,     // [C, 3, 3]
    const uint32_t width,
    const uint32_t height,
    const CameraModelType camera_model,
    const float *__restrict__ v_means2d,  // [C, N, 2]
    const float *__restrict__ v_covars2d, // [C, N, 2, 2]
    float *__restrict__ v_means,          // [C, N, 3]
    float *__restrict__ v_covars          // [C, N, 3, 3]
) {

    // parallelize over C * N.
    uint32_t idx = cg::this_grid().thread_rank();
    if (idx >= C * N) {
        return;
    }
    const uint32_t cid = idx / N; // camera id
    // const uint32_t gid = idx % N; // gaussian id

    // shift pointers to the current camera and gaussian
    means += idx * 3;
    covars += idx * 9;
    v_means += idx * 3;
    v_covars += idx * 9;
    Ks += cid * 9;
    v_means2d += idx * 2;
    v_covars2d += idx * 4;

    float fx = Ks[0], cx = Ks[2], fy = Ks[4], cy = Ks[5];
    mat3 v_covar(0.f);
    vec3 v_mean(0.f);
    const vec3 mean = glm::make_vec3(means);
    const mat3 covar = glm::make_mat3(covars);
    const vec2 v_mean2d = glm::make_vec2(v_means2d);
    const mat2 v_covar2d = glm::make_mat2(v_covars2d);

    switch (camera_model) {
        case CameraModelType::PINHOLE: // perspective projection
            persp_proj_vjp(
                mean,
                covar,
                fx,
                fy,
                cx,
                cy,
                width,
                height,
                glm::transpose(v_covar2d),
                v_mean2d,
                v_mean,
                v_covar
            );
            break;
        case CameraModelType::ORTHO: // orthographic projection
            ortho_proj_vjp(
                mean,
                covar,
                fx,
                fy,
                cx,
                cy,
                width,
                height,
                glm::transpose(v_covar2d),
                v_mean2d,
                v_mean,
                v_covar
            );
            break;
        case CameraModelType::FISHEYE: // fisheye projection
            fisheye_proj_vjp(
                mean,
                covar,
                fx,
                fy,
                cx,
                cy,
                width,
                height,
                glm::transpose(v_covar2d),
                v_mean2d,
                v_mean,
                v_covar
            );
            break;
    }

    // write to outputs: glm is column-major but we want row-major
    #pragma unroll
    for (uint32_t i = 0; i < 3; i++) { // rows
        #pragma unroll
        for (uint32_t j = 0; j < 3; j++) { // cols
            v_covars[i * 3 + j] = v_covar[j][i];
        }
    }

    #pragma unroll
    for (uint32_t i = 0; i < 3; i++) {
        v_means[i] = v_mean[i];
    }
}

void proj_naive_bwd_launcher(
    uint32_t shmem_size, 
    hipStream_t stream, 
    uint32_t n_elements, 
    // args
    const uint32_t C,
    const uint32_t N,
    const float *__restrict__ means,  // [C, N, 3]
    const float *__restrict__ covars, // [C, N, 3, 3]
    const float *__restrict__ Ks,     // [C, 3, 3]
    const uint32_t width,
    const uint32_t height,
    const CameraModelType camera_model,
    const float *__restrict__ v_means2d,  // [C, N, 2]
    const float *__restrict__ v_covars2d, // [C, N, 2, 2]
    float *__restrict__ v_means,          // [C, N, 3]
    float *__restrict__ v_covars          // [C, N, 3, 3]
) {
    if (n_elements <= 0) {
        return;
    }

    proj_naive_bwd_kernel<<<n_blocks_linear(n_elements), N_THREADS, shmem_size, stream>>>(
        C,
        N,
        means,
        covars,
        Ks,
        width,
        height,
        camera_model,
        v_means2d,
        v_covars2d,
        v_means,
        v_covars
    );
}
