#include "hip/hip_runtime.h"
#include <ATen/Dispatch.h>
#include <ATen/core/Tensor.h>
#include <ATen/cuda/Atomic.cuh>
#include <c10/cuda/CUDAStream.h>
#include <hip/hip_cooperative_groups.h>

#include "Common.h"
#include "Projection.h"
#include "Utils.cuh"

namespace gsplat {

namespace cg = cooperative_groups;

template <typename scalar_t>
__global__ void projection_ewa_3dgs_fused_fwd_kernel(
    const uint32_t C,
    const uint32_t N,
    const scalar_t *__restrict__ means,    // [N, 3]
    const scalar_t *__restrict__ covars,   // [N, 6] optional
    const scalar_t *__restrict__ quats,    // [N, 4] optional
    const scalar_t *__restrict__ scales,   // [N, 3] optional
    const scalar_t *__restrict__ viewmats, // [C, 4, 4]
    const scalar_t *__restrict__ Ks,       // [C, 3, 3]
    const int32_t image_width,
    const int32_t image_height,
    const float eps2d,
    const float near_plane,
    const float far_plane,
    const float radius_clip,
    const CameraModelType camera_model,
    // outputs
    int32_t *__restrict__ radii,         // [C, N]
    scalar_t *__restrict__ means2d,      // [C, N, 2]
    scalar_t *__restrict__ depths,       // [C, N]
    scalar_t *__restrict__ conics,       // [C, N, 3]
    scalar_t *__restrict__ compensations // [C, N] optional
) {
    // parallelize over C * N.
    uint32_t idx = cg::this_grid().thread_rank();
    if (idx >= C * N) {
        return;
    }
    const uint32_t cid = idx / N; // camera id
    const uint32_t gid = idx % N; // gaussian id

    // shift pointers to the current camera and gaussian
    means += gid * 3;
    viewmats += cid * 16;
    Ks += cid * 9;

    // glm is column-major but input is row-major
    mat3 R = mat3(
        viewmats[0],
        viewmats[4],
        viewmats[8], // 1st column
        viewmats[1],
        viewmats[5],
        viewmats[9], // 2nd column
        viewmats[2],
        viewmats[6],
        viewmats[10] // 3rd column
    );
    vec3 t = vec3(viewmats[3], viewmats[7], viewmats[11]);

    // transform Gaussian center to camera space
    vec3 mean_c;
    posW2C(R, t, glm::make_vec3(means), mean_c);
    if (mean_c.z < near_plane || mean_c.z > far_plane) {
        radii[idx] = 0;
        return;
    }

    // transform Gaussian covariance to camera space
    mat3 covar;
    if (covars != nullptr) {
        covars += gid * 6;
        covar = mat3(
            covars[0],
            covars[1],
            covars[2], // 1st column
            covars[1],
            covars[3],
            covars[4], // 2nd column
            covars[2],
            covars[4],
            covars[5] // 3rd column
        );
    } else {
        // compute from quaternions and scales
        quats += gid * 4;
        scales += gid * 3;
        quat_scale_to_covar_preci(
            glm::make_vec4(quats), glm::make_vec3(scales), &covar, nullptr
        );
    }
    mat3 covar_c;
    covarW2C(R, covar, covar_c);

    // perspective projection
    mat2 covar2d;
    vec2 mean2d;

    switch (camera_model) {
    case CameraModelType::PINHOLE: // perspective projection
        persp_proj(
            mean_c,
            covar_c,
            Ks[0],
            Ks[4],
            Ks[2],
            Ks[5],
            image_width,
            image_height,
            covar2d,
            mean2d
        );
        break;
    case CameraModelType::ORTHO: // orthographic projection
        ortho_proj(
            mean_c,
            covar_c,
            Ks[0],
            Ks[4],
            Ks[2],
            Ks[5],
            image_width,
            image_height,
            covar2d,
            mean2d
        );
        break;
    case CameraModelType::FISHEYE: // fisheye projection
        fisheye_proj(
            mean_c,
            covar_c,
            Ks[0],
            Ks[4],
            Ks[2],
            Ks[5],
            image_width,
            image_height,
            covar2d,
            mean2d
        );
        break;
    }

    float compensation;
    float det = add_blur(eps2d, covar2d, compensation);
    if (det <= 0.f) {
        radii[idx] = 0;
        return;
    }

    // compute the inverse of the 2d covariance
    mat2 covar2d_inv = glm::inverse(covar2d);

    // take 3 sigma as the radius (non differentiable)
    float b = 0.5f * (covar2d[0][0] + covar2d[1][1]);
    float v1 = b + sqrt(max(0.01f, b * b - det));
    float radius = ceil(3.f * sqrt(v1));
    // float v2 = b - sqrt(max(0.1f, b * b - det));
    // float radius = ceil(3.f * sqrt(max(v1, v2)));

    if (radius <= radius_clip) {
        radii[idx] = 0;
        return;
    }

    // mask out gaussians outside the image region
    if (mean2d.x + radius <= 0 || mean2d.x - radius >= image_width ||
        mean2d.y + radius <= 0 || mean2d.y - radius >= image_height) {
        radii[idx] = 0;
        return;
    }

    // write to outputs
    radii[idx] = (int32_t)radius;
    means2d[idx * 2] = mean2d.x;
    means2d[idx * 2 + 1] = mean2d.y;
    depths[idx] = mean_c.z;
    conics[idx * 3] = covar2d_inv[0][0];
    conics[idx * 3 + 1] = covar2d_inv[0][1];
    conics[idx * 3 + 2] = covar2d_inv[1][1];
    if (compensations != nullptr) {
        compensations[idx] = compensation;
    }
}

void launch_projection_ewa_3dgs_fused_fwd_kernel(
    // inputs
    const at::Tensor means,                // [N, 3]
    const at::optional<at::Tensor> covars, // [N, 6] optional
    const at::optional<at::Tensor> quats,  // [N, 4] optional
    const at::optional<at::Tensor> scales, // [N, 3] optional
    const at::Tensor viewmats,             // [C, 4, 4]
    const at::Tensor Ks,                   // [C, 3, 3]
    const uint32_t image_width,
    const uint32_t image_height,
    const float eps2d,
    const float near_plane,
    const float far_plane,
    const float radius_clip,
    const CameraModelType camera_model,
    // outputs
    at::Tensor radii,                      // [C, N]
    at::Tensor means2d,                    // [C, N, 2]
    at::Tensor depths,                     // [C, N]
    at::Tensor conics,                     // [C, N, 3]
    at::optional<at::Tensor> compensations // [C, N] optional
) {
    uint32_t N = means.size(0);    // number of gaussians
    uint32_t C = viewmats.size(0); // number of cameras

    int64_t n_elements = C * N;
    dim3 threads(256);
    dim3 grid((n_elements + threads.x - 1) / threads.x);
    int64_t shmem_size = 0; // No shared memory used in this kernel

    if (n_elements == 0) {
        // skip the kernel launch if there are no elements
        return;
    }

    AT_DISPATCH_FLOATING_TYPES(
        means.scalar_type(),
        "projection_ewa_3dgs_fused_fwd_kernel",
        [&]() {
            projection_ewa_3dgs_fused_fwd_kernel<scalar_t>
                <<<grid,
                   threads,
                   shmem_size,
                   at::cuda::getCurrentCUDAStream()>>>(
                    C,
                    N,
                    means.data_ptr<scalar_t>(),
                    covars.has_value() ? covars.value().data_ptr<scalar_t>()
                                       : nullptr,
                    quats.has_value() ? quats.value().data_ptr<scalar_t>()
                                      : nullptr,
                    scales.has_value() ? scales.value().data_ptr<scalar_t>()
                                       : nullptr,
                    viewmats.data_ptr<scalar_t>(),
                    Ks.data_ptr<scalar_t>(),
                    image_width,
                    image_height,
                    eps2d,
                    near_plane,
                    far_plane,
                    radius_clip,
                    camera_model,
                    radii.data_ptr<int32_t>(),
                    means2d.data_ptr<scalar_t>(),
                    depths.data_ptr<scalar_t>(),
                    conics.data_ptr<scalar_t>(),
                    compensations.has_value()
                        ? compensations.value().data_ptr<scalar_t>()
                        : nullptr
                );
        }
    );
}

template <typename scalar_t>
__global__ void projection_ewa_3dgs_fused_bwd_kernel(
    // fwd inputs
    const uint32_t C,
    const uint32_t N,
    const scalar_t *__restrict__ means,    // [N, 3]
    const scalar_t *__restrict__ covars,   // [N, 6] optional
    const scalar_t *__restrict__ quats,    // [N, 4] optional
    const scalar_t *__restrict__ scales,   // [N, 3] optional
    const scalar_t *__restrict__ viewmats, // [C, 4, 4]
    const scalar_t *__restrict__ Ks,       // [C, 3, 3]
    const int32_t image_width,
    const int32_t image_height,
    const float eps2d,
    const CameraModelType camera_model,
    // fwd outputs
    const int32_t *__restrict__ radii,          // [C, N]
    const scalar_t *__restrict__ conics,        // [C, N, 3]
    const scalar_t *__restrict__ compensations, // [C, N] optional
    // grad outputs
    const scalar_t *__restrict__ v_means2d,       // [C, N, 2]
    const scalar_t *__restrict__ v_depths,        // [C, N]
    const scalar_t *__restrict__ v_conics,        // [C, N, 3]
    const scalar_t *__restrict__ v_compensations, // [C, N] optional
    // grad inputs
    scalar_t *__restrict__ v_means,   // [N, 3]
    scalar_t *__restrict__ v_covars,  // [N, 6] optional
    scalar_t *__restrict__ v_quats,   // [N, 4] optional
    scalar_t *__restrict__ v_scales,  // [N, 3] optional
    scalar_t *__restrict__ v_viewmats // [C, 4, 4] optional
) {
    // parallelize over C * N.
    uint32_t idx = cg::this_grid().thread_rank();
    if (idx >= C * N || radii[idx] <= 0) {
        return;
    }
    const uint32_t cid = idx / N; // camera id
    const uint32_t gid = idx % N; // gaussian id

    // shift pointers to the current camera and gaussian
    means += gid * 3;
    viewmats += cid * 16;
    Ks += cid * 9;

    conics += idx * 3;

    v_means2d += idx * 2;
    v_depths += idx;
    v_conics += idx * 3;

    // vjp: compute the inverse of the 2d covariance
    mat2 covar2d_inv = mat2(conics[0], conics[1], conics[1], conics[2]);
    mat2 v_covar2d_inv =
        mat2(v_conics[0], v_conics[1] * .5f, v_conics[1] * .5f, v_conics[2]);
    mat2 v_covar2d(0.f);
    inverse_vjp(covar2d_inv, v_covar2d_inv, v_covar2d);

    if (v_compensations != nullptr) {
        // vjp: compensation term
        const float compensation = compensations[idx];
        const float v_compensation = v_compensations[idx];
        add_blur_vjp(
            eps2d, covar2d_inv, compensation, v_compensation, v_covar2d
        );
    }

    // transform Gaussian to camera space
    mat3 R = mat3(
        viewmats[0],
        viewmats[4],
        viewmats[8], // 1st column
        viewmats[1],
        viewmats[5],
        viewmats[9], // 2nd column
        viewmats[2],
        viewmats[6],
        viewmats[10] // 3rd column
    );
    vec3 t = vec3(viewmats[3], viewmats[7], viewmats[11]);

    mat3 covar;
    vec4 quat;
    vec3 scale;
    if (covars != nullptr) {
        covars += gid * 6;
        covar = mat3(
            covars[0],
            covars[1],
            covars[2], // 1st column
            covars[1],
            covars[3],
            covars[4], // 2nd column
            covars[2],
            covars[4],
            covars[5] // 3rd column
        );
    } else {
        // compute from quaternions and scales
        quat = glm::make_vec4(quats + gid * 4);
        scale = glm::make_vec3(scales + gid * 3);
        quat_scale_to_covar_preci(quat, scale, &covar, nullptr);
    }
    vec3 mean_c;
    posW2C(R, t, glm::make_vec3(means), mean_c);
    mat3 covar_c;
    covarW2C(R, covar, covar_c);

    // vjp: perspective projection
    float fx = Ks[0], cx = Ks[2], fy = Ks[4], cy = Ks[5];
    mat3 v_covar_c(0.f);
    vec3 v_mean_c(0.f);

    switch (camera_model) {
    case CameraModelType::PINHOLE: // perspective projection
        persp_proj_vjp(
            mean_c,
            covar_c,
            fx,
            fy,
            cx,
            cy,
            image_width,
            image_height,
            v_covar2d,
            glm::make_vec2(v_means2d),
            v_mean_c,
            v_covar_c
        );
        break;
    case CameraModelType::ORTHO: // orthographic projection
        ortho_proj_vjp(
            mean_c,
            covar_c,
            fx,
            fy,
            cx,
            cy,
            image_width,
            image_height,
            v_covar2d,
            glm::make_vec2(v_means2d),
            v_mean_c,
            v_covar_c
        );
        break;
    case CameraModelType::FISHEYE: // fisheye projection
        fisheye_proj_vjp(
            mean_c,
            covar_c,
            fx,
            fy,
            cx,
            cy,
            image_width,
            image_height,
            v_covar2d,
            glm::make_vec2(v_means2d),
            v_mean_c,
            v_covar_c
        );
        break;
    }

    // add contribution from v_depths
    v_mean_c.z += v_depths[0];

    // vjp: transform Gaussian covariance to camera space
    vec3 v_mean(0.f);
    mat3 v_covar(0.f);
    mat3 v_R(0.f);
    vec3 v_t(0.f);
    posW2C_VJP(R, t, glm::make_vec3(means), v_mean_c, v_R, v_t, v_mean);
    covarW2C_VJP(R, covar, v_covar_c, v_R, v_covar);

    // #if __CUDA_ARCH__ >= 700
    // write out results with warp-level reduction
    auto warp = cg::tiled_partition<32>(cg::this_thread_block());
    auto warp_group_g = cg::labeled_partition(warp, gid);
    if (v_means != nullptr) {
        warpSum(v_mean, warp_group_g);
        if (warp_group_g.thread_rank() == 0) {
            v_means += gid * 3;
#pragma unroll
            for (uint32_t i = 0; i < 3; i++) {
                gpuAtomicAdd(v_means + i, v_mean[i]);
            }
        }
    }
    if (v_covars != nullptr) {
        // Output gradients w.r.t. the covariance matrix
        warpSum(v_covar, warp_group_g);
        if (warp_group_g.thread_rank() == 0) {
            v_covars += gid * 6;
            gpuAtomicAdd(v_covars, v_covar[0][0]);
            gpuAtomicAdd(v_covars + 1, v_covar[0][1] + v_covar[1][0]);
            gpuAtomicAdd(v_covars + 2, v_covar[0][2] + v_covar[2][0]);
            gpuAtomicAdd(v_covars + 3, v_covar[1][1]);
            gpuAtomicAdd(v_covars + 4, v_covar[1][2] + v_covar[2][1]);
            gpuAtomicAdd(v_covars + 5, v_covar[2][2]);
        }
    } else {
        // Directly output gradients w.r.t. the quaternion and scale
        mat3 rotmat = quat_to_rotmat(quat);
        vec4 v_quat(0.f);
        vec3 v_scale(0.f);
        quat_scale_to_covar_vjp(quat, scale, rotmat, v_covar, v_quat, v_scale);
        warpSum(v_quat, warp_group_g);
        warpSum(v_scale, warp_group_g);
        if (warp_group_g.thread_rank() == 0) {
            v_quats += gid * 4;
            v_scales += gid * 3;
            gpuAtomicAdd(v_quats, v_quat[0]);
            gpuAtomicAdd(v_quats + 1, v_quat[1]);
            gpuAtomicAdd(v_quats + 2, v_quat[2]);
            gpuAtomicAdd(v_quats + 3, v_quat[3]);
            gpuAtomicAdd(v_scales, v_scale[0]);
            gpuAtomicAdd(v_scales + 1, v_scale[1]);
            gpuAtomicAdd(v_scales + 2, v_scale[2]);
        }
    }
    if (v_viewmats != nullptr) {
        auto warp_group_c = cg::labeled_partition(warp, cid);
        warpSum(v_R, warp_group_c);
        warpSum(v_t, warp_group_c);
        if (warp_group_c.thread_rank() == 0) {
            v_viewmats += cid * 16;
#pragma unroll
            for (uint32_t i = 0; i < 3; i++) { // rows
#pragma unroll
                for (uint32_t j = 0; j < 3; j++) { // cols
                    gpuAtomicAdd(v_viewmats + i * 4 + j, v_R[j][i]);
                }
                gpuAtomicAdd(v_viewmats + i * 4 + 3, v_t[i]);
            }
        }
    }
}

void launch_projection_ewa_3dgs_fused_bwd_kernel(
    // inputs
    // fwd inputs
    const at::Tensor means,                // [N, 3]
    const at::optional<at::Tensor> covars, // [N, 6] optional
    const at::optional<at::Tensor> quats,  // [N, 4] optional
    const at::optional<at::Tensor> scales, // [N, 3] optional
    const at::Tensor viewmats,             // [C, 4, 4]
    const at::Tensor Ks,                   // [C, 3, 3]
    const uint32_t image_width,
    const uint32_t image_height,
    const float eps2d,
    const CameraModelType camera_model,
    // fwd outputs
    const at::Tensor radii,                       // [C, N]
    const at::Tensor conics,                      // [C, N, 3]
    const at::optional<at::Tensor> compensations, // [C, N] optional
    // grad outputs
    const at::Tensor v_means2d,                     // [C, N, 2]
    const at::Tensor v_depths,                      // [C, N]
    const at::Tensor v_conics,                      // [C, N, 3]
    const at::optional<at::Tensor> v_compensations, // [C, N] optional
    const bool viewmats_requires_grad,
    // outputs
    at::Tensor v_means,   // [C, N, 3]
    at::Tensor v_covars,  // [C, N, 3, 3]
    at::Tensor v_quats,   // [C, N, 4]
    at::Tensor v_scales,  // [C, N, 3]
    at::Tensor v_viewmats // [C, 4, 4]
) {
    uint32_t N = means.size(0);    // number of gaussians
    uint32_t C = viewmats.size(0); // number of cameras

    int64_t n_elements = C * N;
    dim3 threads(256);
    dim3 grid((n_elements + threads.x - 1) / threads.x);
    int64_t shmem_size = 0; // No shared memory used in this kernel

    if (n_elements == 0) {
        // skip the kernel launch if there are no elements
        return;
    }

    AT_DISPATCH_FLOATING_TYPES(
        means.scalar_type(),
        "projection_ewa_3dgs_fused_bwd_kernel",
        [&]() {
            projection_ewa_3dgs_fused_bwd_kernel<scalar_t>
                <<<grid,
                   threads,
                   shmem_size,
                   at::cuda::getCurrentCUDAStream()>>>(
                    C,
                    N,
                    means.data_ptr<scalar_t>(),
                    covars.has_value() ? covars.value().data_ptr<scalar_t>()
                                       : nullptr,
                    covars.has_value() ? nullptr
                                       : quats.value().data_ptr<scalar_t>(),
                    covars.has_value() ? nullptr
                                       : scales.value().data_ptr<scalar_t>(),
                    viewmats.data_ptr<scalar_t>(),
                    Ks.data_ptr<scalar_t>(),
                    image_width,
                    image_height,
                    eps2d,
                    camera_model,
                    radii.data_ptr<int32_t>(),
                    conics.data_ptr<scalar_t>(),
                    compensations.has_value()
                        ? compensations.value().data_ptr<scalar_t>()
                        : nullptr,
                    v_means2d.data_ptr<scalar_t>(),
                    v_depths.data_ptr<scalar_t>(),
                    v_conics.data_ptr<scalar_t>(),
                    v_compensations.has_value()
                        ? v_compensations.value().data_ptr<scalar_t>()
                        : nullptr,
                    v_means.data_ptr<scalar_t>(),
                    covars.has_value() ? v_covars.data_ptr<scalar_t>()
                                       : nullptr,
                    covars.has_value() ? nullptr : v_quats.data_ptr<scalar_t>(),
                    covars.has_value() ? nullptr
                                       : v_scales.data_ptr<scalar_t>(),
                    viewmats_requires_grad ? v_viewmats.data_ptr<scalar_t>()
                                           : nullptr
                );
        }
    );
}

} // namespace gsplat
