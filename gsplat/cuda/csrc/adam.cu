#include "hip/hip_runtime.h"
#include "bindings.h"
#include "helpers.cuh"
#include "utils.cuh"

#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/reduce.h>
#include <hipcub/hipcub.hpp>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

namespace gsplat {

namespace cg = cooperative_groups;

template<typename T>
__global__ void selective_adam_update_kernel(
    T* __restrict__ param,
    const T* __restrict__ param_grad,
    T* __restrict__ exp_avg,
    T* __restrict__ exp_avg_sq,
    const bool* tiles_touched,
    const float lr,
    const float b1,
    const float b2,
    const float eps,
    const uint32_t N,
    const uint32_t M
) {
    auto p_idx = cg::this_grid().thread_rank();
    const uint32_t g_idx = p_idx / M;
    if (g_idx >= N) return;
    if (tiles_touched[g_idx]) {
        T Register_param_grad = param_grad[p_idx];
        T Register_exp_avg = exp_avg[p_idx];
        T Register_exp_avg_sq = exp_avg_sq[p_idx];
        Register_exp_avg = b1 * Register_exp_avg + (1.0f - b1) * Register_param_grad;
        Register_exp_avg_sq = b2 * Register_exp_avg_sq + (1.0f - b2) * Register_param_grad * Register_param_grad;
        T step = -lr * Register_exp_avg / (sqrt(Register_exp_avg_sq) + eps);

        param[p_idx] += step;
        exp_avg[p_idx] = Register_exp_avg;
        exp_avg_sq[p_idx] = Register_exp_avg_sq;
    }
}

void selective_adam_update(
    torch::Tensor &param,
    torch::Tensor &param_grad,
    torch::Tensor &exp_avg,
    torch::Tensor &exp_avg_sq,
    torch::Tensor &tiles_touched,
    const float lr,
    const float b1,
    const float b2,
    const float eps,
    const uint32_t N,
    const uint32_t M
) {
    GSPLAT_DEVICE_GUARD(param);
    GSPLAT_CHECK_INPUT(param);
    GSPLAT_CHECK_INPUT(param_grad);
    GSPLAT_CHECK_INPUT(exp_avg);
    GSPLAT_CHECK_INPUT(exp_avg_sq);
    GSPLAT_CHECK_INPUT(tiles_touched);

    const uint32_t cnt = N * M;
    at::cuda::CUDAStream stream = at::cuda::getCurrentCUDAStream();
    selective_adam_update_kernel<float><<<(cnt + 255) / 256, 256, 0, stream>>>(
        param.data_ptr<float>(),
        param_grad.data_ptr<float>(),
        exp_avg.data_ptr<float>(),
        exp_avg_sq.data_ptr<float>(),
        tiles_touched.data_ptr<bool>(),
        lr,
        b1,
        b2,
        eps,
        N,
        M
    );
}

} // namespace gsplat