#include "hip/hip_runtime.h"
#include "bindings.h"
#include "helpers.cuh"
#include "utils.cuh"

#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/reduce.h>
#include <hipcub/hipcub.hpp>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

namespace gsplat {

namespace cg = cooperative_groups;

/****************************************************************************
 * Projection of Gaussians (Batched) Backward Pass
 ****************************************************************************/

template <typename T>
__global__ void fully_fused_projection_packed_bwd_kernel(
    // fwd inputs
    const uint32_t C,
    const uint32_t N,
    const uint32_t nnz,
    const T *__restrict__ means,    // [N, 3]
    const T *__restrict__ covars,   // [N, 6] Optional
    const T *__restrict__ quats,    // [N, 4] Optional
    const T *__restrict__ scales,   // [N, 3] Optional
    const T *__restrict__ viewmats, // [C, 4, 4]
    const T *__restrict__ Ks,       // [C, 3, 3]
    const int32_t image_width,
    const int32_t image_height,
    const T eps2d,
    // fwd outputs
    const int64_t *__restrict__ camera_ids,   // [nnz]
    const int64_t *__restrict__ gaussian_ids, // [nnz]
    const T *__restrict__ conics,             // [nnz, 3]
    const T *__restrict__ compensations,      // [nnz] optional
    // grad outputs
    const T *__restrict__ v_means2d,       // [nnz, 2]
    const T *__restrict__ v_depths,        // [nnz]
    const T *__restrict__ v_conics,        // [nnz, 3]
    const T *__restrict__ v_compensations, // [nnz] optional
    const bool sparse_grad, // whether the outputs are in COO format [nnz, ...]
    // grad inputs
    T *__restrict__ v_means,   // [N, 3] or [nnz, 3]
    T *__restrict__ v_covars,  // [N, 6] or [nnz, 6] Optional
    T *__restrict__ v_quats,   // [N, 4] or [nnz, 4] Optional
    T *__restrict__ v_scales,  // [N, 3] or [nnz, 3] Optional
    T *__restrict__ v_viewmats // [C, 4, 4] Optional
) {
    // parallelize over nnz.
    uint32_t idx = cg::this_grid().thread_rank();
    if (idx >= nnz) {
        return;
    }
    const int64_t cid = camera_ids[idx];   // camera id
    const int64_t gid = gaussian_ids[idx]; // gaussian id

    // shift pointers to the current camera and gaussian
    means += gid * 3;
    viewmats += cid * 16;
    Ks += cid * 9;

    conics += idx * 3;

    v_means2d += idx * 2;
    v_depths += idx;
    v_conics += idx * 3;

    // vjp: compute the inverse of the 2d covariance
    mat2<T> covar2d_inv = mat2<T>(conics[0], conics[1], conics[1], conics[2]);
    mat2<T> v_covar2d_inv =
        mat2<T>(v_conics[0], v_conics[1] * .5f, v_conics[1] * .5f, v_conics[2]);
    mat2<T> v_covar2d(0.f);
    inverse_vjp(covar2d_inv, v_covar2d_inv, v_covar2d);

    if (v_compensations != nullptr) {
        // vjp: compensation term
        const T compensation = compensations[idx];
        const T v_compensation = v_compensations[idx];
        add_blur_vjp(
            eps2d, covar2d_inv, compensation, v_compensation, v_covar2d
        );
    }

    // transform Gaussian to camera space
    mat3<T> R = mat3<T>(
        viewmats[0],
        viewmats[4],
        viewmats[8], // 1st column
        viewmats[1],
        viewmats[5],
        viewmats[9], // 2nd column
        viewmats[2],
        viewmats[6],
        viewmats[10] // 3rd column
    );
    vec3<T> t = vec3<T>(viewmats[3], viewmats[7], viewmats[11]);
    mat3<T> covar;
    vec4<T> quat;
    vec3<T> scale;
    if (covars != nullptr) {
        // if a precomputed covariance is provided
        covars += gid * 6;
        covar = mat3<T>(
            covars[0],
            covars[1],
            covars[2], // 1st column
            covars[1],
            covars[3],
            covars[4], // 2nd column
            covars[2],
            covars[4],
            covars[5] // 3rd column
        );
    } else {
        // if not then compute it from quaternions and scales
        quat = glm::make_vec4(quats + gid * 4);
        scale = glm::make_vec3(scales + gid * 3);
        quat_scale_to_covar_preci<T>(quat, scale, &covar, nullptr);
    }
    vec3<T> mean_c;
    pos_world_to_cam(R, t, glm::make_vec3(means), mean_c);
    mat3<T> covar_c;
    covar_world_to_cam(R, covar, covar_c);

    // vjp: perspective projection
    T fx = Ks[0], cx = Ks[2], fy = Ks[4], cy = Ks[5];
    mat3<T> v_covar_c(0.f);
    vec3<T> v_mean_c(0.f);
    persp_proj_vjp<T>(
        mean_c,
        covar_c,
        fx,
        fy,
        cx,
        cy,
        image_width,
        image_height,
        v_covar2d,
        glm::make_vec2(v_means2d),
        v_mean_c,
        v_covar_c
    );

    // add contribution from v_depths
    v_mean_c.z += v_depths[0];

    // vjp: transform Gaussian covariance to camera space
    vec3<T> v_mean(0.f);
    mat3<T> v_covar(0.f);
    mat3<T> v_R(0.f);
    vec3<T> v_t(0.f);
    pos_world_to_cam_vjp(
        R, t, glm::make_vec3(means), v_mean_c, v_R, v_t, v_mean
    );
    covar_world_to_cam_vjp(R, covar, v_covar_c, v_R, v_covar);

    auto warp = cg::tiled_partition<32>(cg::this_thread_block());
    if (sparse_grad) {
        // write out results with sparse layout
        if (v_means != nullptr) {
            v_means += idx * 3;
            GSPLAT_PRAGMA_UNROLL
            for (uint32_t i = 0; i < 3; i++) {
                v_means[i] = v_mean[i];
            }
        }
        if (v_covars != nullptr) {
            v_covars += idx * 6;
            v_covars[0] = v_covar[0][0];
            v_covars[1] = v_covar[0][1] + v_covar[1][0];
            v_covars[2] = v_covar[0][2] + v_covar[2][0];
            v_covars[3] = v_covar[1][1];
            v_covars[4] = v_covar[1][2] + v_covar[2][1];
            v_covars[5] = v_covar[2][2];
        } else {
            mat3<T> rotmat = quat_to_rotmat<T>(quat);
            vec4<T> v_quat(0.f);
            vec3<T> v_scale(0.f);
            quat_scale_to_covar_vjp<T>(
                quat, scale, rotmat, v_covar, v_quat, v_scale
            );
            v_quats += idx * 4;
            v_scales += idx * 3;
            v_quats[0] = v_quat[0];
            v_quats[1] = v_quat[1];
            v_quats[2] = v_quat[2];
            v_quats[3] = v_quat[3];
            v_scales[0] = v_scale[0];
            v_scales[1] = v_scale[1];
            v_scales[2] = v_scale[2];
        }
    } else {
        // write out results with dense layout
        // #if __CUDA_ARCH__ >= 700
        // write out results with warp-level reduction
        auto warp_group_g = cg::labeled_partition(warp, gid);
        if (v_means != nullptr) {
            warpSum(v_mean, warp_group_g);
            if (warp_group_g.thread_rank() == 0) {
                v_means += gid * 3;
                GSPLAT_PRAGMA_UNROLL
                for (uint32_t i = 0; i < 3; i++) {
                    gpuAtomicAdd(v_means + i, v_mean[i]);
                }
            }
        }
        if (v_covars != nullptr) {
            // Directly output gradients w.r.t. the covariance
            warpSum(v_covar, warp_group_g);
            if (warp_group_g.thread_rank() == 0) {
                v_covars += gid * 6;
                gpuAtomicAdd(v_covars, v_covar[0][0]);
                gpuAtomicAdd(v_covars + 1, v_covar[0][1] + v_covar[1][0]);
                gpuAtomicAdd(v_covars + 2, v_covar[0][2] + v_covar[2][0]);
                gpuAtomicAdd(v_covars + 3, v_covar[1][1]);
                gpuAtomicAdd(v_covars + 4, v_covar[1][2] + v_covar[2][1]);
                gpuAtomicAdd(v_covars + 5, v_covar[2][2]);
            }
        } else {
            // Directly output gradients w.r.t. the quaternion and scale
            mat3<T> rotmat = quat_to_rotmat<T>(quat);
            vec4<T> v_quat(0.f);
            vec3<T> v_scale(0.f);
            quat_scale_to_covar_vjp<T>(
                quat, scale, rotmat, v_covar, v_quat, v_scale
            );
            warpSum(v_quat, warp_group_g);
            warpSum(v_scale, warp_group_g);
            if (warp_group_g.thread_rank() == 0) {
                v_quats += gid * 4;
                v_scales += gid * 3;
                gpuAtomicAdd(v_quats, v_quat[0]);
                gpuAtomicAdd(v_quats + 1, v_quat[1]);
                gpuAtomicAdd(v_quats + 2, v_quat[2]);
                gpuAtomicAdd(v_quats + 3, v_quat[3]);
                gpuAtomicAdd(v_scales, v_scale[0]);
                gpuAtomicAdd(v_scales + 1, v_scale[1]);
                gpuAtomicAdd(v_scales + 2, v_scale[2]);
            }
        }
    }
    // v_viewmats is always in dense layout
    if (v_viewmats != nullptr) {
        auto warp_group_c = cg::labeled_partition(warp, cid);
        warpSum(v_R, warp_group_c);
        warpSum(v_t, warp_group_c);
        if (warp_group_c.thread_rank() == 0) {
            v_viewmats += cid * 16;
            GSPLAT_PRAGMA_UNROLL
            for (uint32_t i = 0; i < 3; i++) { // rows
                GSPLAT_PRAGMA_UNROLL
                for (uint32_t j = 0; j < 3; j++) { // cols
                    gpuAtomicAdd(v_viewmats + i * 4 + j, v_R[j][i]);
                }
                gpuAtomicAdd(v_viewmats + i * 4 + 3, v_t[i]);
            }
        }
    }
}

std::tuple<
    torch::Tensor,
    torch::Tensor,
    torch::Tensor,
    torch::Tensor,
    torch::Tensor>
fully_fused_projection_packed_bwd_tensor(
    // fwd inputs
    const torch::Tensor &means,                // [N, 3]
    const at::optional<torch::Tensor> &covars, // [N, 6]
    const at::optional<torch::Tensor> &quats,  // [N, 4]
    const at::optional<torch::Tensor> &scales, // [N, 3]
    const torch::Tensor &viewmats,             // [C, 4, 4]
    const torch::Tensor &Ks,                   // [C, 3, 3]
    const uint32_t image_width,
    const uint32_t image_height,
    const float eps2d,
    // fwd outputs
    const torch::Tensor &camera_ids,                  // [nnz]
    const torch::Tensor &gaussian_ids,                // [nnz]
    const torch::Tensor &conics,                      // [nnz, 3]
    const at::optional<torch::Tensor> &compensations, // [nnz] optional
    // grad outputs
    const torch::Tensor &v_means2d,                     // [nnz, 2]
    const torch::Tensor &v_depths,                      // [nnz]
    const torch::Tensor &v_conics,                      // [nnz, 3]
    const at::optional<torch::Tensor> &v_compensations, // [nnz] optional
    const bool viewmats_requires_grad,
    const bool sparse_grad
) {
    GSPLAT_DEVICE_GUARD(means);
    GSPLAT_CHECK_INPUT(means);
    if (covars.has_value()) {
        GSPLAT_CHECK_INPUT(covars.value());
    } else {
        assert(quats.has_value() && scales.has_value());
        GSPLAT_CHECK_INPUT(quats.value());
        GSPLAT_CHECK_INPUT(scales.value());
    }
    GSPLAT_CHECK_INPUT(viewmats);
    GSPLAT_CHECK_INPUT(Ks);
    GSPLAT_CHECK_INPUT(camera_ids);
    GSPLAT_CHECK_INPUT(gaussian_ids);
    GSPLAT_CHECK_INPUT(conics);
    GSPLAT_CHECK_INPUT(v_means2d);
    GSPLAT_CHECK_INPUT(v_depths);
    GSPLAT_CHECK_INPUT(v_conics);
    if (compensations.has_value()) {
        GSPLAT_CHECK_INPUT(compensations.value());
    }
    if (v_compensations.has_value()) {
        GSPLAT_CHECK_INPUT(v_compensations.value());
        assert(compensations.has_value());
    }

    uint32_t N = means.size(0);    // number of gaussians
    uint32_t C = viewmats.size(0); // number of cameras
    uint32_t nnz = camera_ids.size(0);
    at::cuda::CUDAStream stream = at::cuda::getCurrentCUDAStream();

    torch::Tensor v_means, v_covars, v_quats, v_scales, v_viewmats;
    if (sparse_grad) {
        v_means = torch::zeros({nnz, 3}, means.options());
        if (covars.has_value()) {
            v_covars = torch::zeros({nnz, 6}, covars.value().options());
        } else {
            v_quats = torch::zeros({nnz, 4}, quats.value().options());
            v_scales = torch::zeros({nnz, 3}, scales.value().options());
        }
        if (viewmats_requires_grad) {
            v_viewmats = torch::zeros({C, 4, 4}, viewmats.options());
        }
    } else {
        v_means = torch::zeros_like(means);
        if (covars.has_value()) {
            v_covars = torch::zeros_like(covars.value());
        } else {
            v_quats = torch::zeros_like(quats.value());
            v_scales = torch::zeros_like(scales.value());
        }
        if (viewmats_requires_grad) {
            v_viewmats = torch::zeros_like(viewmats);
        }
    }
    if (nnz) {
        fully_fused_projection_packed_bwd_kernel<float>
            <<<(nnz + GSPLAT_N_THREADS - 1) / GSPLAT_N_THREADS,
               GSPLAT_N_THREADS,
               0,
               stream>>>(
                C,
                N,
                nnz,
                means.data_ptr<float>(),
                covars.has_value() ? covars.value().data_ptr<float>() : nullptr,
                covars.has_value() ? nullptr : quats.value().data_ptr<float>(),
                covars.has_value() ? nullptr : scales.value().data_ptr<float>(),
                viewmats.data_ptr<float>(),
                Ks.data_ptr<float>(),
                image_width,
                image_height,
                eps2d,
                camera_ids.data_ptr<int64_t>(),
                gaussian_ids.data_ptr<int64_t>(),
                conics.data_ptr<float>(),
                compensations.has_value()
                    ? compensations.value().data_ptr<float>()
                    : nullptr,
                v_means2d.data_ptr<float>(),
                v_depths.data_ptr<float>(),
                v_conics.data_ptr<float>(),
                v_compensations.has_value()
                    ? v_compensations.value().data_ptr<float>()
                    : nullptr,
                sparse_grad,
                v_means.data_ptr<float>(),
                covars.has_value() ? v_covars.data_ptr<float>() : nullptr,
                covars.has_value() ? nullptr : v_quats.data_ptr<float>(),
                covars.has_value() ? nullptr : v_scales.data_ptr<float>(),
                viewmats_requires_grad ? v_viewmats.data_ptr<float>() : nullptr
            );
    }
    return std::make_tuple(v_means, v_covars, v_quats, v_scales, v_viewmats);
}

} // namespace gsplat