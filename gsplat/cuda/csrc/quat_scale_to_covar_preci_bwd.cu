#include "hip/hip_runtime.h"
#include "bindings.h"
#include "helpers.cuh"
#include "utils.cuh"

#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/reduce.h>
#include <hipcub/hipcub.hpp>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

namespace gsplat {

namespace cg = cooperative_groups;

/****************************************************************************
 * Quat-Scale to Covariance and Precision Backward Pass
 ****************************************************************************/

template <typename T>
__global__ void quat_scale_to_covar_preci_bwd_kernel(
    const uint32_t N,
    // fwd inputs
    const T *__restrict__ quats,  // [N, 4]
    const T *__restrict__ scales, // [N, 3]
    // grad outputs
    const T *__restrict__ v_covars, // [N, 3, 3] or [N, 6]
    const T *__restrict__ v_precis, // [N, 3, 3] or [N, 6]
    const bool triu,
    // grad inputs
    T *__restrict__ v_scales, // [N, 3]
    T *__restrict__ v_quats   // [N, 4]
) {

    // For now we'll upcast float16 and bfloat16 to float32
    using OpT = typename OpType<T>::type;

    // parallelize over N.
    uint32_t idx = cg::this_grid().thread_rank();
    if (idx >= N) {
        return;
    }

    // shift pointers to the current gaussian
    v_scales += idx * 3;
    v_quats += idx * 4;

    vec4<OpT> quat = glm::make_vec4(quats + idx * 4);
    vec3<OpT> scale = glm::make_vec3(scales + idx * 3);
    mat3<OpT> rotmat = quat_to_rotmat<OpT>(quat);

    vec4<OpT> v_quat(0.f);
    vec3<OpT> v_scale(0.f);
    if (v_covars != nullptr) {
        // glm is column-major, input is row-major
        mat3<OpT> v_covar;
        if (triu) {
            v_covars += idx * 6;
            v_covar = mat3<OpT>(
                v_covars[0],
                v_covars[1] * .5f,
                v_covars[2] * .5f,
                v_covars[1] * .5f,
                v_covars[3],
                v_covars[4] * .5f,
                v_covars[2] * .5f,
                v_covars[4] * .5f,
                v_covars[5]
            );
        } else {
            v_covars += idx * 9;
            mat3<OpT> v_covar_cast = glm::make_mat3(v_covars);
            v_covar = glm::transpose(v_covar_cast);
        }
        quat_scale_to_covar_vjp<OpT>(
            quat, scale, rotmat, v_covar, v_quat, v_scale
        );
    }
    if (v_precis != nullptr) {
        // glm is column-major, input is row-major
        mat3<OpT> v_preci;
        if (triu) {
            v_precis += idx * 6;
            v_preci = mat3<OpT>(
                v_precis[0],
                v_precis[1] * .5f,
                v_precis[2] * .5f,
                v_precis[1] * .5f,
                v_precis[3],
                v_precis[4] * .5f,
                v_precis[2] * .5f,
                v_precis[4] * .5f,
                v_precis[5]
            );
        } else {
            v_precis += idx * 9;
            mat3<OpT> v_precis_cast = glm::make_mat3(v_precis);
            v_preci = glm::transpose(v_precis_cast);
        }
        quat_scale_to_preci_vjp<OpT>(
            quat, scale, rotmat, v_preci, v_quat, v_scale
        );
    }

    // write out results
    GSPLAT_PRAGMA_UNROLL
    for (uint32_t k = 0; k < 3; ++k) {
        v_scales[k] = T(v_scale[k]);
    }
    GSPLAT_PRAGMA_UNROLL
    for (uint32_t k = 0; k < 4; ++k) {
        v_quats[k] = T(v_quat[k]);
    }
}

std::tuple<torch::Tensor, torch::Tensor> quat_scale_to_covar_preci_bwd_tensor(
    const torch::Tensor &quats,                  // [N, 4]
    const torch::Tensor &scales,                 // [N, 3]
    const at::optional<torch::Tensor> &v_covars, // [N, 3, 3] or [N, 6]
    const at::optional<torch::Tensor> &v_precis, // [N, 3, 3] or [N, 6]
    const bool triu
) {
    GSPLAT_DEVICE_GUARD(quats);
    GSPLAT_CHECK_INPUT(quats);
    GSPLAT_CHECK_INPUT(scales);
    if (v_covars.has_value()) {
        GSPLAT_CHECK_INPUT(v_covars.value());
    }
    if (v_precis.has_value()) {
        GSPLAT_CHECK_INPUT(v_precis.value());
    }

    uint32_t N = quats.size(0);

    torch::Tensor v_scales = torch::empty_like(scales);
    torch::Tensor v_quats = torch::empty_like(quats);

    if (N) {
        at::cuda::CUDAStream stream = at::cuda::getCurrentCUDAStream();
        AT_DISPATCH_FLOATING_TYPES_AND2(
            at::ScalarType::Half,
            at::ScalarType::BFloat16,
            quats.scalar_type(),
            "quat_scale_to_covar_preci_bwd",
            [&]() {
                quat_scale_to_covar_preci_bwd_kernel<scalar_t>
                    <<<(N + GSPLAT_N_THREADS - 1) / GSPLAT_N_THREADS,
                       GSPLAT_N_THREADS,
                       0,
                       stream>>>(
                        N,
                        quats.data_ptr<scalar_t>(),
                        scales.data_ptr<scalar_t>(),
                        v_covars.has_value()
                            ? v_covars.value().data_ptr<scalar_t>()
                            : nullptr,
                        v_precis.has_value()
                            ? v_precis.value().data_ptr<scalar_t>()
                            : nullptr,
                        triu,
                        v_scales.data_ptr<scalar_t>(),
                        v_quats.data_ptr<scalar_t>()
                    );
            }
        );
    }

    return std::make_tuple(v_quats, v_scales);
}

} // namespace gsplat