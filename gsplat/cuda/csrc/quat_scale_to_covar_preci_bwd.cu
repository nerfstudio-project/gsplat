#include "hip/hip_runtime.h"
#include "bindings.h"
#include "quaternion.cuh"

#include <hip/hip_cooperative_groups.h>

namespace gsplat {

namespace cg = cooperative_groups;

/****************************************************************************
 * Quat-Scale to Covariance and Precision Backward Pass
 ****************************************************************************/


__global__ void quat_scale_to_covar_preci_bwd_kernel(
    const uint32_t N,
    // fwd inputs
    const float *__restrict__ quats,  // [N, 4]
    const float *__restrict__ scales, // [N, 3]
    // grad outputs
    const float *__restrict__ v_covars, // [N, 3, 3] or [N, 6]
    const float *__restrict__ v_precis, // [N, 3, 3] or [N, 6]
    const bool triu,
    // grad inputs
    float *__restrict__ v_scales, // [N, 3]
    float *__restrict__ v_quats   // [N, 4]
) {


    // parallelize over N.
    uint32_t idx = cg::this_grid().thread_rank();
    if (idx >= N) {
        return;
    }

    // shift pointers to the current gaussian
    v_scales += idx * 3;
    v_quats += idx * 4;

    vec4 quat = glm::make_vec4(quats + idx * 4);
    vec3 scale = glm::make_vec3(scales + idx * 3);
    mat3 rotmat = quat_to_rotmat(quat);

    vec4 v_quat(0.f);
    vec3 v_scale(0.f);
    if (v_covars != nullptr) {
        // glm is column-major, input is row-major
        mat3 v_covar;
        if (triu) {
            v_covars += idx * 6;
            v_covar = mat3(
                v_covars[0],
                v_covars[1] * .5f,
                v_covars[2] * .5f,
                v_covars[1] * .5f,
                v_covars[3],
                v_covars[4] * .5f,
                v_covars[2] * .5f,
                v_covars[4] * .5f,
                v_covars[5]
            );
        } else {
            v_covars += idx * 9;
            mat3 v_covar_cast = glm::make_mat3(v_covars);
            v_covar = glm::transpose(v_covar_cast);
        }
        quat_scale_to_covar_vjp(
            quat, scale, rotmat, v_covar, v_quat, v_scale
        );
    }
    if (v_precis != nullptr) {
        // glm is column-major, input is row-major
        mat3 v_preci;
        if (triu) {
            v_precis += idx * 6;
            v_preci = mat3(
                v_precis[0],
                v_precis[1] * .5f,
                v_precis[2] * .5f,
                v_precis[1] * .5f,
                v_precis[3],
                v_precis[4] * .5f,
                v_precis[2] * .5f,
                v_precis[4] * .5f,
                v_precis[5]
            );
        } else {
            v_precis += idx * 9;
            mat3 v_precis_cast = glm::make_mat3(v_precis);
            v_preci = glm::transpose(v_precis_cast);
        }
        quat_scale_to_preci_vjp(
            quat, scale, rotmat, v_preci, v_quat, v_scale
        );
    }

    // write out results
    GSPLAT_PRAGMA_UNROLL
    for (uint32_t k = 0; k < 3; ++k) {
        v_scales[k] = v_scale[k];
    }
    GSPLAT_PRAGMA_UNROLL
    for (uint32_t k = 0; k < 4; ++k) {
        v_quats[k] = v_quat[k];
    }
}

std::tuple<torch::Tensor, torch::Tensor> quat_scale_to_covar_preci_bwd_tensor(
    const torch::Tensor &quats,                  // [N, 4]
    const torch::Tensor &scales,                 // [N, 3]
    const at::optional<torch::Tensor> &v_covars, // [N, 3, 3] or [N, 6]
    const at::optional<torch::Tensor> &v_precis, // [N, 3, 3] or [N, 6]
    const bool triu
) {
    GSPLAT_DEVICE_GUARD(quats);
    GSPLAT_CHECK_INPUT(quats);
    GSPLAT_CHECK_INPUT(scales);
    if (v_covars.has_value()) {
        GSPLAT_CHECK_INPUT(v_covars.value());
    }
    if (v_precis.has_value()) {
        GSPLAT_CHECK_INPUT(v_precis.value());
    }

    uint32_t N = quats.size(0);

    torch::Tensor v_scales = torch::empty_like(scales);
    torch::Tensor v_quats = torch::empty_like(quats);

    if (N) {
        at::cuda::CUDAStream stream = at::cuda::getCurrentCUDAStream();
                quat_scale_to_covar_preci_bwd_kernel
                    <<<(N + GSPLAT_N_THREADS - 1) / GSPLAT_N_THREADS,
                       GSPLAT_N_THREADS,
                       0,
                       stream>>>(
                        N,
                        quats.data_ptr<float>(),
                        scales.data_ptr<float>(),
                        v_covars.has_value()
                            ? v_covars.value().data_ptr<float>()
                            : nullptr,
                        v_precis.has_value()
                            ? v_precis.value().data_ptr<float>()
                            : nullptr,
                        triu,
                        v_scales.data_ptr<float>(),
                        v_quats.data_ptr<float>()
                    );
    }

    return std::make_tuple(v_quats, v_scales);
}

} // namespace gsplat