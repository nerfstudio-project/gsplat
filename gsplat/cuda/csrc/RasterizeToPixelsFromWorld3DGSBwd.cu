#include "hip/hip_runtime.h"
#include <ATen/Dispatch.h>
#include <ATen/core/Tensor.h>
#include <ATen/cuda/Atomic.cuh>
#include <c10/cuda/CUDAStream.h>
#include <hip/hip_cooperative_groups.h>

#include "Common.h"
#include "Rasterization.h"
#include "Utils.cuh"
#include "Cameras.cuh"

namespace gsplat {

namespace cg = cooperative_groups;

template <uint32_t CDIM, typename scalar_t>
__global__ void rasterize_to_pixels_from_world_3dgs_bwd_kernel(
    const uint32_t C,
    const uint32_t N,
    const uint32_t n_isects,
    const bool packed,
    // fwd inputs
    const vec3 *__restrict__ means,       // [N, 3]
    const vec4 *__restrict__ quats,       // [N, 4]
    const vec3 *__restrict__ scales,      // [N, 3]
    const scalar_t *__restrict__ colors,      // [C, N, CDIM] or [nnz, CDIM]
    const scalar_t *__restrict__ opacities,   // [C, N] or [nnz]
    const scalar_t *__restrict__ backgrounds, // [C, CDIM] or [nnz, CDIM]
    const bool *__restrict__ masks,           // [C, tile_height, tile_width]
    const uint32_t image_width,
    const uint32_t image_height,
    const uint32_t tile_size,
    const uint32_t tile_width,
    const uint32_t tile_height,
    // camera model
    const scalar_t *__restrict__ viewmats0, // [C, 4, 4]
    const scalar_t *__restrict__ viewmats1, // [C, 4, 4] optional for rolling shutter
    const scalar_t *__restrict__ Ks,        // [C, 3, 3]
    const CameraModelType camera_model_type,
    // uncented transform
    const UnscentedTransformParameters ut_params,    
    const ShutterType rs_type,
    const scalar_t *__restrict__ radial_coeffs, // [C, 6] or [C, 4] optional
    const scalar_t *__restrict__ tangential_coeffs, // [C, 2] optional
    const scalar_t *__restrict__ thin_prism_coeffs, // [C, 2] optional
    // intersections
    const int32_t *__restrict__ tile_offsets, // [C, tile_height, tile_width]
    const int32_t *__restrict__ flatten_ids,  // [n_isects]
    // fwd outputs
    const scalar_t
        *__restrict__ render_alphas,      // [C, image_height, image_width, 1]
    const int32_t *__restrict__ last_ids, // [C, image_height, image_width]
    // grad outputs
    const scalar_t *__restrict__ v_render_colors, // [C, image_height,
                                                  // image_width, CDIM]
    const scalar_t
        *__restrict__ v_render_alphas, // [C, image_height, image_width, 1]
    // grad inputs
    vec3 *__restrict__ v_means,      // [N, 3]
    vec4 *__restrict__ v_quats,       // [N, 4]
    vec3 *__restrict__ v_scales,      // [N, 3]
    scalar_t *__restrict__ v_colors,   // [C, N, CDIM] or [nnz, CDIM]
    scalar_t *__restrict__ v_opacities // [C, N] or [nnz]
) {
    auto block = cg::this_thread_block();
    uint32_t cid = block.group_index().x;
    uint32_t tile_id =
        block.group_index().y * tile_width + block.group_index().z;
    uint32_t i = block.group_index().y * tile_size + block.thread_index().y;
    uint32_t j = block.group_index().z * tile_size + block.thread_index().x;

    tile_offsets += cid * tile_height * tile_width;
    render_alphas += cid * image_height * image_width;
    last_ids += cid * image_height * image_width;
    v_render_colors += cid * image_height * image_width * CDIM;
    v_render_alphas += cid * image_height * image_width;
    if (backgrounds != nullptr) {
        backgrounds += cid * CDIM;
    }
    if (masks != nullptr) {
        masks += cid * tile_height * tile_width;
    }

    // when the mask is provided, do nothing and return if
    // this tile is labeled as False
    if (masks != nullptr && !masks[tile_id]) {
        return;
    }

    const float px = (float)j + 0.5f;
    const float py = (float)i + 0.5f;
    // clamp this value to the last pixel
    const int32_t pix_id =
        min(i * image_width + j, image_width * image_height - 1);

    // Create rolling shutter parameter
    auto rs_params = RollingShutterParameters(
        viewmats0 + cid * 16,
        viewmats1 == nullptr ? nullptr : viewmats1 + cid * 16
    );
    // shift pointers to the current camera. note that glm is colume-major.
    const vec2 focal_length = {Ks[cid * 9 + 0], Ks[cid * 9 + 4]};
    const vec2 principal_point = {Ks[cid * 9 + 2], Ks[cid * 9 + 5]};
    
    // Create ray from pixel
    WorldRay ray;
    if (camera_model_type == CameraModelType::PINHOLE) {
        if (radial_coeffs == nullptr && tangential_coeffs == nullptr && thin_prism_coeffs == nullptr) {
            PerfectPinholeCameraModel::Parameters cm_params = {};
            cm_params.resolution = {image_width, image_height};
            cm_params.shutter_type = rs_type;
            cm_params.principal_point = { principal_point.x, principal_point.y };
            cm_params.focal_length = { focal_length.x, focal_length.y };
            PerfectPinholeCameraModel camera_model(cm_params);
            ray = camera_model.image_point_to_world_ray_shutter_pose(vec2(px, py), rs_params);
        } else {
            OpenCVPinholeCameraModel<>::Parameters cm_params = {};
            cm_params.resolution = {image_width, image_height};
            cm_params.shutter_type = rs_type;
            cm_params.principal_point = { principal_point.x, principal_point.y };
            cm_params.focal_length = { focal_length.x, focal_length.y };
            if (radial_coeffs != nullptr) {
                cm_params.radial_coeffs = make_array<float, 6>(radial_coeffs + cid * 6);
            }
            if (tangential_coeffs != nullptr) {
                cm_params.tangential_coeffs = make_array<float, 2>(tangential_coeffs + cid * 2);
            }
            if (thin_prism_coeffs != nullptr) {
                cm_params.thin_prism_coeffs = make_array<float, 4>(thin_prism_coeffs + cid * 4);
            }
            OpenCVPinholeCameraModel camera_model(cm_params);
            ray = camera_model.image_point_to_world_ray_shutter_pose(vec2(px, py), rs_params);
        }
    } else if (camera_model_type == CameraModelType::FISHEYE) {
        OpenCVFisheyeCameraModel<>::Parameters cm_params = {};
        cm_params.resolution = {image_width, image_height};
        cm_params.shutter_type = rs_type;
        cm_params.principal_point = { principal_point.x, principal_point.y };
        cm_params.focal_length = { focal_length.x, focal_length.y };
        if (radial_coeffs != nullptr) {
            cm_params.radial_coeffs = make_array<float, 4>(radial_coeffs + cid * 4);
        }
        OpenCVFisheyeCameraModel camera_model(cm_params);
        ray = camera_model.image_point_to_world_ray_shutter_pose(vec2(px, py), rs_params);
    } else {
        // should never reach here
        assert(false);
        return;
    }
    const vec3 ray_d = ray.ray_dir;
    const vec3 ray_o = ray.ray_org;

    // keep not rasterizing threads around for reading data
    bool done = (i < image_height && j < image_width) && ray.valid_flag;

    // have all threads in tile process the same gaussians in batches
    // first collect gaussians between range.x and range.y in batches
    // which gaussians to look through in this tile
    int32_t range_start = tile_offsets[tile_id];
    int32_t range_end =
        (cid == C - 1) && (tile_id == tile_width * tile_height - 1)
            ? n_isects
            : tile_offsets[tile_id + 1];
    const uint32_t block_size = block.size();
    const uint32_t num_batches =
        (range_end - range_start + block_size - 1) / block_size;

    extern __shared__ int s[];
    int32_t *id_batch = (int32_t *)s; // [block_size]
    vec4 *xyz_opacity_batch =
        reinterpret_cast<vec4 *>(&id_batch[block_size]); // [block_size]
    vec3 *scale_batch =
        reinterpret_cast<vec3 *>(&xyz_opacity_batch[block_size]); // [block_size]
    vec4 *quat_batch =
        reinterpret_cast<vec4 *>(&scale_batch[block_size]); // [block_size]
    float *rgbs_batch =
        (float *)&quat_batch[block_size]; // [block_size * CDIM]

    // this is the T AFTER the last gaussian in this pixel
    float T_final = 1.0f - render_alphas[pix_id];
    float T = T_final;
    // the contribution from gaussians behind the current one
    float buffer[CDIM] = {0.f};
    // index of last gaussian to contribute to this pixel
    const int32_t bin_final = done ? last_ids[pix_id] : 0;

    // df/d_out for this pixel
    float v_render_c[CDIM];
#pragma unroll
    for (uint32_t k = 0; k < CDIM; ++k) {
        v_render_c[k] = v_render_colors[pix_id * CDIM + k];
    }
    const float v_render_a = v_render_alphas[pix_id];

    // collect and process batches of gaussians
    // each thread loads one gaussian at a time before rasterizing
    const uint32_t tr = block.thread_rank();
    cg::thread_block_tile<32> warp = cg::tiled_partition<32>(block);
    const int32_t warp_bin_final =
        cg::reduce(warp, bin_final, cg::greater<int>());
    for (uint32_t b = 0; b < num_batches; ++b) {
        // resync all threads before writing next batch of shared mem
        block.sync();

        // each thread fetch 1 gaussian from back to front
        // 0 index will be furthest back in batch
        // index of gaussian to load
        // batch end is the index of the last gaussian in the batch
        // These values can be negative so must be int32 instead of uint32
        const int32_t batch_end = range_end - 1 - block_size * b;
        const int32_t batch_size = min(block_size, batch_end + 1 - range_start);
        const int32_t idx = batch_end - tr;
        if (idx >= range_start) {
            // TODO: only support 1 camera for now so it is ok to abuse the index.
            int32_t g = flatten_ids[idx]; // flatten index in [C * N] or [nnz]
            id_batch[tr] = g;
            const vec3 xyz = means[g];
            const float opac = opacities[g];
            xyz_opacity_batch[tr] = {xyz.x, xyz.y, xyz.z, opac};
            scale_batch[tr] = scales[g];
            quat_batch[tr] = quats[g];
#pragma unroll
            for (uint32_t k = 0; k < CDIM; ++k) {
                rgbs_batch[tr * CDIM + k] = colors[g * CDIM + k];
            }
        }
        // wait for other threads to collect the gaussians in batch
        block.sync();
        // process gaussians in the current batch for this pixel
        // 0 index is the furthest back gaussian in the batch
        for (uint32_t t = max(0, batch_end - warp_bin_final); t < batch_size;
             ++t) {
            bool valid = done;
            if (batch_end - t > bin_final) {
                valid = 0;
            }
            float alpha;
            float opac;
            float vis;

            mat3 R, S;
            vec3 xyz;
            vec3 scale;
            vec4 quat;
            mat3 Mt;
            vec3 o_minus_mu, gro, grd, grd_n, gcrod;
            float grayDist, power;
            if (valid) {
                const vec4 xyz_opac = xyz_opacity_batch[t];
                opac = xyz_opac[3];
                xyz = {xyz_opac[0], xyz_opac[1], xyz_opac[2]};
                scale = scale_batch[t];
                quat = quat_batch[t];
                
                R = quat_to_rotmat(quat);
                S = mat3(
                    1.0f / scale[0],
                    0.f,
                    0.f,
                    0.f,
                    1.0f / scale[1],
                    0.f,
                    0.f,
                    0.f,
                    1.0f / scale[2]
                );
                Mt = glm::transpose(R * S);
                o_minus_mu = ray_o - xyz;
                gro = Mt * o_minus_mu;
                grd = Mt * ray_d;
                grd_n = safe_normalize(grd);
                gcrod = glm::cross(grd_n, gro);
                grayDist = glm::dot(gcrod, gcrod);
                power = -0.5f * grayDist;

                vis = __expf(power);
                alpha = min(0.999f, opac * vis);
                if (power > 0.f || alpha < 1.f / 255.f) {
                    valid = false;
                }
            }

            // if all threads are inactive in this warp, skip this loop
            if (!warp.any(valid)) {
                continue;
            }
            float v_rgb_local[CDIM] = {0.f};
            vec3 v_mean_local = {0.f, 0.f, 0.f};
            vec3 v_scale_local = {0.f, 0.f, 0.f};
            vec4 v_quat_local = {0.f, 0.f, 0.f, 0.f};
            float v_opacity_local = 0.f;
            // initialize everything to 0, only set if the lane is valid
            if (valid) {
                // compute the current T for this gaussian
                float ra = 1.0f / (1.0f - alpha);
                T *= ra;
                // update v_rgb for this gaussian
                const float fac = alpha * T;
#pragma unroll
                for (uint32_t k = 0; k < CDIM; ++k) {
                    v_rgb_local[k] = fac * v_render_c[k];
                }
                // contribution from this pixel
                float v_alpha = 0.f;
#pragma unroll
                for (uint32_t k = 0; k < CDIM; ++k) {
                    v_alpha += (rgbs_batch[t * CDIM + k] * T - buffer[k] * ra) *
                               v_render_c[k];
                }

                v_alpha += T_final * ra * v_render_a;
                // contribution from background pixel
                if (backgrounds != nullptr) {
                    float accum = 0.f;
#pragma unroll
                    for (uint32_t k = 0; k < CDIM; ++k) {
                        accum += backgrounds[k] * v_render_c[k];
                    }
                    v_alpha += -T_final * ra * accum;
                }

                if (opac * vis <= 0.999f) {
                    const float v_vis = opac * v_alpha;
                    float v_gradDist = -0.5f * vis * v_vis;
                    vec3 v_gcrod = 2.0f * v_gradDist * gcrod;
                    vec3 v_grd_n = - glm::cross(v_gcrod, gro);
                    vec3 v_gro = glm::cross(v_gcrod, grd_n);
                    vec3 v_grd = safe_normalize_bw(grd, v_grd_n);
                    mat3 v_Mt = glm::outerProduct(v_grd, ray_d) + 
                        glm::outerProduct(v_gro, o_minus_mu);
                    vec3 v_o_minus_mu = glm::transpose(Mt) * v_gro;

                    v_mean_local += -v_o_minus_mu;
                    quat_scale_to_preci_half_vjp(
                        quat, scale, R, glm::transpose(v_Mt), v_quat_local, v_scale_local
                    );
                    v_opacity_local = vis * v_alpha;
                }

#pragma unroll
                for (uint32_t k = 0; k < CDIM; ++k) {
                    buffer[k] += rgbs_batch[t * CDIM + k] * fac;
                }
            }
            warpSum<CDIM>(v_rgb_local, warp);
            warpSum(v_mean_local, warp);
            warpSum(v_scale_local, warp);
            warpSum(v_quat_local, warp);
            warpSum(v_opacity_local, warp);
            if (warp.thread_rank() == 0) {
                int32_t g = id_batch[t]; // flatten index in [C * N] or [nnz]
                float *v_rgb_ptr = (float *)(v_colors) + CDIM * g;
#pragma unroll
                for (uint32_t k = 0; k < CDIM; ++k) {
                    gpuAtomicAdd(v_rgb_ptr + k, v_rgb_local[k]);
                }

                float *v_mean_ptr = (float *)(v_means) + 3 * g;
                gpuAtomicAdd(v_mean_ptr, v_mean_local.x);
                gpuAtomicAdd(v_mean_ptr + 1, v_mean_local.y);
                gpuAtomicAdd(v_mean_ptr + 2, v_mean_local.z);

                float *v_scale_ptr = (float *)(v_scales) + 3 * g;
                gpuAtomicAdd(v_scale_ptr, v_scale_local.x);
                gpuAtomicAdd(v_scale_ptr + 1, v_scale_local.y);
                gpuAtomicAdd(v_scale_ptr + 2, v_scale_local.z);

                float *v_quat_ptr = (float *)(v_quats) + 4 * g;
                gpuAtomicAdd(v_quat_ptr, v_quat_local.x);
                gpuAtomicAdd(v_quat_ptr + 1, v_quat_local.y);
                gpuAtomicAdd(v_quat_ptr + 2, v_quat_local.z);
                gpuAtomicAdd(v_quat_ptr + 3, v_quat_local.w);

                gpuAtomicAdd(v_opacities + g, v_opacity_local);
            }
        }
    }
}

template <uint32_t CDIM>
void launch_rasterize_to_pixels_from_world_3dgs_bwd_kernel(
    // Gaussian parameters
    const at::Tensor means, // [N, 3]
    const at::Tensor quats, // [N, 4]
    const at::Tensor scales, // [N, 3]
    const at::Tensor colors,                    // [C, N, 3] or [nnz, 3]
    const at::Tensor opacities,                 // [C, N] or [nnz]
    const at::optional<at::Tensor> backgrounds, // [C, 3]
    const at::optional<at::Tensor> masks,       // [C, tile_height, tile_width]
    // image size
    const uint32_t image_width,
    const uint32_t image_height,
    const uint32_t tile_size,
    // camera
    const at::Tensor viewmats0,             // [C, 4, 4]
    const at::optional<at::Tensor> viewmats1, // [C, 4, 4] optional for rolling shutter
    const at::Tensor Ks,                   // [C, 3, 3]
    const CameraModelType camera_model,
    // uncented transform
    const UnscentedTransformParameters ut_params,
    ShutterType rs_type,
    const at::optional<at::Tensor> radial_coeffs, // [C, 6] or [C, 4] optional
    const at::optional<at::Tensor> tangential_coeffs, // [C, 2] optional
    const at::optional<at::Tensor> thin_prism_coeffs, // [C, 2] optional
    // intersections
    const at::Tensor tile_offsets, // [C, tile_height, tile_width]
    const at::Tensor flatten_ids,  // [n_isects]
    // forward outputs
    const at::Tensor render_alphas, // [C, image_height, image_width, 1]
    const at::Tensor last_ids,      // [C, image_height, image_width]
    // gradients of outputs
    const at::Tensor v_render_colors, // [C, image_height, image_width, 3]
    const at::Tensor v_render_alphas, // [C, image_height, image_width, 1]
    // outputs
    at::Tensor v_means,      // [N, 3]
    at::Tensor v_quats,      // [N, 4]
    at::Tensor v_scales,     // [N, 3]
    at::Tensor v_colors,                    // [C, N, 3] or [nnz, 3]
    at::Tensor v_opacities                  // [C, N] or [nnz]
) {
    bool packed = opacities.dim() == 1;
    assert (packed == false); // only support non-packed for now

    uint32_t C = tile_offsets.size(0);         // number of cameras
    uint32_t N = packed ? 0 : means.size(0); // number of gaussians
    uint32_t tile_height = tile_offsets.size(1);
    uint32_t tile_width = tile_offsets.size(2);
    uint32_t n_isects = flatten_ids.size(0);

    // Each block covers a tile on the image. In total there are
    // C * tile_height * tile_width blocks.
    dim3 threads = {tile_size, tile_size, 1};
    dim3 grid = {C, tile_height, tile_width};

    int64_t shmem_size =
        tile_size * tile_size *
        (sizeof(int32_t) + sizeof(vec4) + sizeof(vec3) + sizeof(vec4) + sizeof(float) * CDIM);

    if (n_isects == 0) {
        // skip the kernel launch if there are no elements
        return;
    }

    // TODO: an optimization can be done by passing the actual number of
    // channels into the kernel functions and avoid necessary global memory
    // writes. This requires moving the channel padding from python to C side.
    if (hipFuncSetAttribute(reinterpret_cast<const void*>(
            rasterize_to_pixels_from_world_3dgs_bwd_kernel<CDIM), float>,
            hipFuncAttributeMaxDynamicSharedMemorySize,
            shmem_size
        ) != hipSuccess) {
        AT_ERROR(
            "Failed to set maximum shared memory size (requested ",
            shmem_size,
            " bytes), try lowering tile_size."
        );
    }

    rasterize_to_pixels_from_world_3dgs_bwd_kernel<CDIM, float>
        <<<grid, threads, shmem_size, at::cuda::getCurrentCUDAStream()>>>(
            C,
            N,
            n_isects,
            packed,
            reinterpret_cast<vec3 *>(means.data_ptr<float>()),
            reinterpret_cast<vec4 *>(quats.data_ptr<float>()),
            reinterpret_cast<vec3 *>(scales.data_ptr<float>()),
            colors.data_ptr<float>(),
            opacities.data_ptr<float>(),
            backgrounds.has_value() ? backgrounds.value().data_ptr<float>()
                                    : nullptr,
            masks.has_value() ? masks.value().data_ptr<bool>() : nullptr,
            image_width,
            image_height,
            tile_size,
            tile_width,
            tile_height,
            // camera model
            viewmats0.data_ptr<float>(),
            viewmats1.has_value() ? viewmats1.value().data_ptr<float>()
                                : nullptr,
            Ks.data_ptr<float>(),
            camera_model,
            // uncented transform
            ut_params,
            rs_type,
            radial_coeffs.has_value() ? radial_coeffs.value().data_ptr<float>()
                                    : nullptr,
            tangential_coeffs.has_value()
                ? tangential_coeffs.value().data_ptr<float>()
                : nullptr,
            thin_prism_coeffs.has_value()
                ? thin_prism_coeffs.value().data_ptr<float>()
                : nullptr,
            // intersections
            tile_offsets.data_ptr<int32_t>(),
            flatten_ids.data_ptr<int32_t>(),
            render_alphas.data_ptr<float>(),
            last_ids.data_ptr<int32_t>(),
            v_render_colors.data_ptr<float>(),
            v_render_alphas.data_ptr<float>(),
            // outputs
            reinterpret_cast<vec3 *>(v_means.data_ptr<float>()),
            reinterpret_cast<vec4 *>(v_quats.data_ptr<float>()),
            reinterpret_cast<vec3 *>(v_scales.data_ptr<float>()),
            v_colors.data_ptr<float>(),
            v_opacities.data_ptr<float>()
        );
}

// Explicit Instantiation: this should match how it is being called in .cpp
// file.
// TODO: this is slow to compile, can we do something about it?
#define __INS__(CDIM)                                                          \
    template void launch_rasterize_to_pixels_from_world_3dgs_bwd_kernel<CDIM>( \
        const at::Tensor means,                                                \
        const at::Tensor quats,                                                \
        const at::Tensor scales,                                               \
        const at::Tensor colors,                                               \
        const at::Tensor opacities,                                            \
        const at::optional<at::Tensor> backgrounds,                            \
        const at::optional<at::Tensor> masks,                                  \
        const uint32_t image_width,                                            \
        const uint32_t image_height,                                           \
        const uint32_t tile_size,                                              \
        const at::Tensor viewmats0,                                            \
        const at::optional<at::Tensor> viewmats1,                              \
        const at::Tensor Ks,                                                   \
        const CameraModelType camera_model,                                    \
        const UnscentedTransformParameters ut_params,                         \
        const ShutterType rs_type,                                             \
        const at::optional<at::Tensor> radial_coeffs,                         \
        const at::optional<at::Tensor> tangential_coeffs,                     \
        const at::optional<at::Tensor> thin_prism_coeffs,                     \
        const at::Tensor tile_offsets,                                         \
        const at::Tensor flatten_ids,                                          \
        const at::Tensor render_alphas,                                        \
        const at::Tensor last_ids,                                             \
        const at::Tensor v_render_colors,                                      \
        const at::Tensor v_render_alphas,                                      \
        at::Tensor v_means,                                                    \
        at::Tensor v_quats,                                                    \
        at::Tensor v_scales,                                                   \
        at::Tensor v_colors,                                                   \
        at::Tensor v_opacities                                                 \
    );

__INS__(1)
__INS__(2)
__INS__(3)
__INS__(4)
__INS__(5)
__INS__(8)
__INS__(9)
__INS__(16)
__INS__(17)
__INS__(32)
__INS__(33)
__INS__(64)
__INS__(65)
__INS__(128)
__INS__(129)
__INS__(256)
__INS__(257)
__INS__(512)
__INS__(513)
    
#undef __INS__

} // namespace gsplat
