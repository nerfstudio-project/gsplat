#include "hip/hip_runtime.h"
#include <ATen/Dispatch.h>
#include <ATen/core/Tensor.h>
#include <ATen/cuda/Atomic.cuh>
#include <c10/cuda/CUDAStream.h>
#include <hip/hip_cooperative_groups.h>
#include <hipcub/hipcub.hpp>

#include "Common.h"
#include "Projection.h"
#include "Projection2DGS.cuh" // Utils for 2DGS Projection
#include "Utils.cuh"

namespace gsplat {

namespace cg = cooperative_groups;

template <typename scalar_t>
__global__ void projection_2dgs_packed_fwd_kernel(
    const uint32_t C,
    const uint32_t N,
    const scalar_t *__restrict__ means,    // [N, 3]
    const scalar_t *__restrict__ quats,    // [N, 4]
    const scalar_t *__restrict__ scales,   // [N, 3]
    const scalar_t *__restrict__ viewmats, // [C, 4, 4]
    const scalar_t *__restrict__ Ks,       // [C, 3, 3]
    const int32_t image_width,
    const int32_t image_height,
    const scalar_t near_plane,
    const scalar_t far_plane,
    const scalar_t radius_clip,
    const int32_t
        *__restrict__ block_accum,    // [C * blocks_per_row] packing helper
    int32_t *__restrict__ block_cnts, // [C * blocks_per_row] packing helper
    // outputs
    int32_t *__restrict__ indptr,          // [C + 1]
    int64_t *__restrict__ camera_ids,      // [nnz]
    int64_t *__restrict__ gaussian_ids,    // [nnz]
    int32_t *__restrict__ radii,           // [nnz]
    scalar_t *__restrict__ means2d,        // [nnz, 2]
    scalar_t *__restrict__ depths,         // [nnz]
    scalar_t *__restrict__ ray_transforms, // [nnz, 3, 3]
    scalar_t *__restrict__ normals         // [nnz, 3]
) {
    int32_t blocks_per_row = gridDim.x;

    int32_t row_idx = blockIdx.y; // cid
    int32_t block_col_idx = blockIdx.x;
    int32_t block_idx = row_idx * blocks_per_row + block_col_idx;

    int32_t col_idx = block_col_idx * blockDim.x + threadIdx.x; // gid

    bool valid = (row_idx < C) && (col_idx < N);

    // check if points are with camera near and far plane
    vec3 mean_c;
    mat3 R;
    if (valid) {
        // shift pointers to the current camera and gaussian
        means += col_idx * 3;
        viewmats += row_idx * 16;

        // glm is column-major but input is row-major
        R = mat3(
            viewmats[0],
            viewmats[4],
            viewmats[8], // 1st column
            viewmats[1],
            viewmats[5],
            viewmats[9], // 2nd column
            viewmats[2],
            viewmats[6],
            viewmats[10] // 3rd column
        );
        vec3 t = vec3(viewmats[3], viewmats[7], viewmats[11]);

        // transform Gaussian center to camera space
        posW2C(R, t, glm::make_vec3(means), mean_c);
        if (mean_c.z < near_plane || mean_c.z > far_plane) {
            valid = false;
        }
    }

    vec2 mean2d;
    mat3 M;
    float radius;
    vec3 normal;
    if (valid) {
        // build ray transformation matrix and transform from world space to
        // camera space
        quats += col_idx * 4;
        scales += col_idx * 3;
        Ks += row_idx * 9;

        mat3 RS_camera =
            R * quat_to_rotmat(glm::make_vec4(quats)) *
            mat3(scales[0], 0.0, 0.0, 0.0, scales[1], 0.0, 0.0, 0.0, 1.0);
        ;
        mat3 WH = mat3(RS_camera[0], RS_camera[1], mean_c);

        mat3 world_2_pix =
            mat3(Ks[0], 0.0, Ks[2], 0.0, Ks[4], Ks[5], 0.0, 0.0, 1.0);
        M = glm::transpose(WH) * world_2_pix;

        // compute AABB
        const vec3 M0 = vec3(M[0][0], M[0][1], M[0][2]);
        const vec3 M1 = vec3(M[1][0], M[1][1], M[1][2]);
        const vec3 M2 = vec3(M[2][0], M[2][1], M[2][2]);

        const vec3 temp_point = vec3(1.0f, 1.0f, -1.0f);
        const float distance = sum(temp_point * M2 * M2);

        if (distance == 0.0f)
            valid = false;

        const vec3 f = (1 / distance) * temp_point;
        mean2d = vec2(sum(f * M0 * M2), sum(f * M1 * M2));

        const vec2 temp = {sum(f * M0 * M0), sum(f * M1 * M1)};
        const vec2 half_extend = mean2d * mean2d - temp;
        radius = ceil(3.f * sqrt(max(1e-4, max(half_extend.x, half_extend.y))));

        if (radius <= radius_clip) {
            valid = false;
        }

        // mask out gaussians outside the image region
        if (mean2d.x + radius <= 0 || mean2d.x - radius >= image_width ||
            mean2d.y + radius <= 0 || mean2d.y - radius >= image_height) {
            valid = false;
        }

        // normal dual visible
        normal = RS_camera[2];
        float multipler = glm::dot(-normal, mean_c) > 0 ? 1 : -1;
        normal *= multipler;
    }

    int32_t thread_data = static_cast<int32_t>(valid);
    if (block_cnts != nullptr) {
        // First pass: compute the block-wide sum
        int32_t aggregate;
        if (__syncthreads_or(thread_data)) {
            typedef hipcub::BlockReduce<int32_t, N_THREADS_PACKED> BlockReduce;
            __shared__ typename BlockReduce::TempStorage temp_storage;
            aggregate = BlockReduce(temp_storage).Sum(thread_data);
        } else {
            aggregate = 0;
        }
        if (threadIdx.x == 0) {
            block_cnts[block_idx] = aggregate;
        }
    } else {
        // Second pass: write out the indices of the non zero elements
        if (__syncthreads_or(thread_data)) {
            typedef hipcub::BlockScan<int32_t, N_THREADS_PACKED> BlockScan;
            __shared__ typename BlockScan::TempStorage temp_storage;
            BlockScan(temp_storage).ExclusiveSum(thread_data, thread_data);
        }
        if (valid) {
            if (block_idx > 0) {
                int32_t offset = block_accum[block_idx - 1];
                thread_data += offset;
            }
            // write to outputs
            camera_ids[thread_data] = row_idx;   // cid
            gaussian_ids[thread_data] = col_idx; // gid
            radii[thread_data] = (int32_t)radius;
            means2d[thread_data * 2] = mean2d.x;
            means2d[thread_data * 2 + 1] = mean2d.y;
            depths[thread_data] = mean_c.z;
            ray_transforms[thread_data * 9] = M[0][0];
            ray_transforms[thread_data * 9 + 1] = M[0][1];
            ray_transforms[thread_data * 9 + 2] = M[0][2];
            ray_transforms[thread_data * 9 + 3] = M[1][0];
            ray_transforms[thread_data * 9 + 4] = M[1][1];
            ray_transforms[thread_data * 9 + 5] = M[1][2];
            ray_transforms[thread_data * 9 + 6] = M[2][0];
            ray_transforms[thread_data * 9 + 7] = M[2][1];
            ray_transforms[thread_data * 9 + 8] = M[2][2];
            normals[thread_data * 3] = normal.x;
            normals[thread_data * 3 + 1] = normal.y;
            normals[thread_data * 3 + 2] = normal.z;
        }
        // lane 0 of the first block in each row writes the indptr
        if (threadIdx.x == 0 && block_col_idx == 0) {
            if (row_idx == 0) {
                indptr[0] = 0;
                indptr[C] = block_accum[C * blocks_per_row - 1];
            } else {
                indptr[row_idx] = block_accum[block_idx - 1];
            }
        }
    }
}

void launch_projection_2dgs_packed_fwd_kernel(
    // inputs
    const at::Tensor means,    // [N, 3]
    const at::Tensor quats,    // [N, 4]
    const at::Tensor scales,   // [N, 3]
    const at::Tensor viewmats, // [C, 4, 4]
    const at::Tensor Ks,       // [C, 3, 3]
    const uint32_t image_width,
    const uint32_t image_height,
    const float near_plane,
    const float far_plane,
    const float radius_clip,
    const at::optional<at::Tensor>
        block_accum, // [C * blocks_per_row] packing helper
    // outputs
    at::optional<at::Tensor> block_cnts, // [C * blocks_per_row] packing helper
    at::optional<at::Tensor> indptr,     // [C + 1]
    at::optional<at::Tensor> camera_ids, // [nnz]
    at::optional<at::Tensor> gaussian_ids,   // [nnz]
    at::optional<at::Tensor> radii,          // [nnz]
    at::optional<at::Tensor> means2d,        // [nnz, 2]
    at::optional<at::Tensor> depths,         // [nnz]
    at::optional<at::Tensor> ray_transforms, // [nnz, 3, 3]
    at::optional<at::Tensor> normals         // [nnz]
) {
    uint32_t N = means.size(0);    // number of gaussians
    uint32_t C = viewmats.size(0); // number of cameras

    uint32_t nrows = C;
    uint32_t ncols = N;
    uint32_t blocks_per_row = (ncols + N_THREADS_PACKED - 1) / N_THREADS_PACKED;

    dim3 threads(N_THREADS_PACKED);
    // limit on the number of blocks: [2**31 - 1, 65535, 65535]
    dim3 grid(blocks_per_row, nrows, 1);
    int64_t shmem_size = 0; // No shared memory used in this kernel

    if (N == 0 || C == 0) {
        // skip the kernel launch if there are no elements
        return;
    }

    projection_2dgs_packed_fwd_kernel<float>
        <<<grid, threads, shmem_size, at::cuda::getCurrentCUDAStream()>>>(
            C,
            N,
            means.data_ptr<float>(),
            quats.data_ptr<float>(),
            scales.data_ptr<float>(),
            viewmats.data_ptr<float>(),
            Ks.data_ptr<float>(),
            image_width,
            image_height,
            near_plane,
            far_plane,
            radius_clip,
            block_accum.has_value() ? block_accum.value().data_ptr<int32_t>()
                                    : nullptr,
            block_cnts.has_value() ? block_cnts.value().data_ptr<int32_t>()
                                   : nullptr,
            indptr.has_value() ? indptr.value().data_ptr<int32_t>() : nullptr,
            camera_ids.has_value() ? camera_ids.value().data_ptr<int64_t>()
                                   : nullptr,
            gaussian_ids.has_value() ? gaussian_ids.value().data_ptr<int64_t>()
                                     : nullptr,
            radii.has_value() ? radii.value().data_ptr<int32_t>() : nullptr,
            means2d.has_value() ? means2d.value().data_ptr<float>() : nullptr,
            depths.has_value() ? depths.value().data_ptr<float>() : nullptr,
            ray_transforms.has_value()
                ? ray_transforms.value().data_ptr<float>()
                : nullptr,
            normals.has_value() ? normals.value().data_ptr<float>() : nullptr
        );
}

template <typename scalar_t>
__global__ void projection_2dgs_packed_bwd_kernel(
    // fwd inputs
    const uint32_t C,
    const uint32_t N,
    const uint32_t nnz,
    const scalar_t *__restrict__ means,    // [N, 3]
    const scalar_t *__restrict__ quats,    // [N, 4]
    const scalar_t *__restrict__ scales,   // [N, 3]
    const scalar_t *__restrict__ viewmats, // [C, 4, 4]
    const scalar_t *__restrict__ Ks,       // [C, 3, 3]
    const int32_t image_width,
    const int32_t image_height,
    // fwd outputs
    const int64_t *__restrict__ camera_ids,      // [nnz]
    const int64_t *__restrict__ gaussian_ids,    // [nnz]
    const scalar_t *__restrict__ ray_transforms, // [nnz, 3]
    // grad outputs
    const scalar_t *__restrict__ v_means2d,        // [nnz, 2]
    const scalar_t *__restrict__ v_depths,         // [nnz]
    const scalar_t *__restrict__ v_ray_transforms, // [nnz, 3, 3]
    const scalar_t *__restrict__ v_normals,        // [nnz, 3]
    const bool sparse_grad, // whether the outputs are in COO format [nnz, ...]
    // grad inputs
    scalar_t *__restrict__ v_means,   // [N, 3] or [nnz, 3]
    scalar_t *__restrict__ v_quats,   // [N, 4] or [nnz, 4] Optional
    scalar_t *__restrict__ v_scales,  // [N, 3] or [nnz, 3] Optional
    scalar_t *__restrict__ v_viewmats // [C, 4, 4] Optional
) {
    // parallelize over nnz.
    uint32_t idx = cg::this_grid().thread_rank();
    if (idx >= nnz) {
        return;
    }
    const int64_t cid = camera_ids[idx];   // camera id
    const int64_t gid = gaussian_ids[idx]; // gaussian id

    // shift pointers to the current camera and gaussian
    means += gid * 3;
    viewmats += cid * 16;
    Ks += cid * 9;

    ray_transforms += idx * 9;

    v_means2d += idx * 2;
    v_normals += idx * 3;
    v_depths += idx;
    v_ray_transforms += idx * 9;

    // transform Gaussian to camera space
    mat3 R = mat3(
        viewmats[0],
        viewmats[4],
        viewmats[8], // 1st column
        viewmats[1],
        viewmats[5],
        viewmats[9], // 2nd column
        viewmats[2],
        viewmats[6],
        viewmats[10] // 3rd column
    );
    vec3 t = vec3(viewmats[3], viewmats[7], viewmats[11]);
    vec3 mean_c;
    posW2C(R, t, glm::make_vec3(means), mean_c);

    vec4 quat = glm::make_vec4(quats + gid * 4);
    vec2 scale = glm::make_vec2(scales + gid * 3);
    mat3 P = mat3(Ks[0], 0.0, Ks[2], 0.0, Ks[4], Ks[5], 0.0, 0.0, 1.0);

    mat3 _v_ray_transforms = mat3(
        v_ray_transforms[0],
        v_ray_transforms[1],
        v_ray_transforms[2],
        v_ray_transforms[3],
        v_ray_transforms[4],
        v_ray_transforms[5],
        v_ray_transforms[6],
        v_ray_transforms[7],
        v_ray_transforms[8]
    );

    _v_ray_transforms[2][2] += v_depths[0];

    vec3 v_normal = glm::make_vec3(v_normals);

    vec3 v_mean(0.f);
    vec2 v_scale(0.f);
    vec4 v_quat(0.f);
    compute_ray_transforms_aabb_vjp(
        ray_transforms,
        v_means2d,
        v_normal,
        R,
        P,
        t,
        mean_c,
        quat,
        scale,
        _v_ray_transforms,
        v_quat,
        v_scale,
        v_mean
    );

    auto warp = cg::tiled_partition<32>(cg::this_thread_block());
    if (sparse_grad) {
        // write out results with sparse layout
        if (v_means != nullptr) {
            v_means += idx * 3;
#pragma unroll
            for (uint32_t i = 0; i < 3; i++) {
                v_means[i] = v_mean[i];
            }
        }
        v_quats += idx * 4;
        v_scales += idx * 3;
        v_quats[0] = v_quat[0];
        v_quats[1] = v_quat[1];
        v_quats[2] = v_quat[2];
        v_quats[3] = v_quat[3];
        v_scales[0] = v_scale[0];
        v_scales[1] = v_scale[1];
    } else {
        // write out results with dense layout
        // #if __CUDA_ARCH__ >= 700
        // write out results with warp-level reduction
        auto warp_group_g = cg::labeled_partition(warp, gid);
        if (v_means != nullptr) {
            warpSum(v_mean, warp_group_g);
            if (warp_group_g.thread_rank() == 0) {
                v_means += gid * 3;
#pragma unroll
                for (uint32_t i = 0; i < 3; i++) {
                    gpuAtomicAdd(v_means + i, v_mean[i]);
                }
            }
        }
        // Directly output gradients w.r.t. the quaternion and scale
        warpSum(v_quat, warp_group_g);
        warpSum(v_scale, warp_group_g);
        if (warp_group_g.thread_rank() == 0) {
            v_quats += gid * 4;
            v_scales += gid * 3;
            gpuAtomicAdd(v_quats, v_quat[0]);
            gpuAtomicAdd(v_quats + 1, v_quat[1]);
            gpuAtomicAdd(v_quats + 2, v_quat[2]);
            gpuAtomicAdd(v_quats + 3, v_quat[3]);
            gpuAtomicAdd(v_scales, v_scale[0]);
            gpuAtomicAdd(v_scales + 1, v_scale[1]);
        }
    }
}

void launch_projection_2dgs_packed_bwd_kernel(
    // fwd inputs
    const at::Tensor means,    // [N, 3]
    const at::Tensor quats,    // [N, 4]
    const at::Tensor scales,   // [N, 3]
    const at::Tensor viewmats, // [C, 4, 4]
    const at::Tensor Ks,       // [C, 3, 3]
    const uint32_t image_width,
    const uint32_t image_height,
    // fwd outputs
    const at::Tensor camera_ids,     // [nnz]
    const at::Tensor gaussian_ids,   // [nnz]
    const at::Tensor ray_transforms, // [nnz, 3, 3]
    // grad outputs
    const at::Tensor v_means2d,        // [nnz, 2]
    const at::Tensor v_depths,         // [nnz]
    const at::Tensor v_ray_transforms, // [nnz, 3, 3]
    const at::Tensor v_normals,        // [nnz, 3]
    const bool sparse_grad,
    // grad inputs
    at::Tensor v_means,                 // [N, 3] or [nnz, 3]
    at::Tensor v_quats,                 // [N, 4] or [nnz, 4]
    at::Tensor v_scales,                // [N, 3] or [nnz, 3]
    at::optional<at::Tensor> v_viewmats // [C, 4, 4] Optional
) {
    uint32_t N = means.size(0);    // number of gaussians
    uint32_t C = viewmats.size(0); // number of cameras
    uint32_t nnz = camera_ids.size(0);

    dim3 threads(256);
    dim3 grid((nnz + threads.x - 1) / threads.x);
    int64_t shmem_size = 0; // No shared memory used in this kernel

    if (nnz == 0) {
        // skip the kernel launch if there are no elements
        return;
    }

    projection_2dgs_packed_bwd_kernel<float>
        <<<grid, threads, shmem_size, at::cuda::getCurrentCUDAStream()>>>(
            C,
            N,
            nnz,
            means.data_ptr<float>(),
            quats.data_ptr<float>(),
            scales.data_ptr<float>(),
            viewmats.data_ptr<float>(),
            Ks.data_ptr<float>(),
            image_width,
            image_height,
            camera_ids.data_ptr<int64_t>(),
            gaussian_ids.data_ptr<int64_t>(),
            ray_transforms.data_ptr<float>(),
            v_means2d.data_ptr<float>(),
            v_depths.data_ptr<float>(),
            v_ray_transforms.data_ptr<float>(),
            v_normals.data_ptr<float>(),
            sparse_grad,
            v_means.data_ptr<float>(),
            v_quats.data_ptr<float>(),
            v_scales.data_ptr<float>(),
            v_viewmats.has_value() ? v_viewmats.value().data_ptr<float>()
                                   : nullptr
        );
}

} // namespace gsplat