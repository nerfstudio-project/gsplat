#include "hip/hip_runtime.h"
#include "bindings.h"
#include "helpers.cuh"
#include "utils.cuh"

#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/reduce.h>
#include <hipcub/hipcub.hpp>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

namespace gsplat {

namespace cg = cooperative_groups;

/****************************************************************************
 * Projection of Gaussians (Batched) Backward Pass
 ****************************************************************************/
template <typename T>
__global__ void fully_fused_projection_bwd_2dgs_kernel(
    // fwd inputs
    const uint32_t C,
    const uint32_t N,
    const T *__restrict__ means,    // [N, 3]
    const T *__restrict__ quats,    // [N, 4]
    const T *__restrict__ scales,   // [N, 3]
    const T *__restrict__ viewmats, // [C, 4, 4]
    const T *__restrict__ Ks,       // [C, 3, 3]
    const int32_t image_width,
    const int32_t image_height,
    // fwd outputs
    const int32_t *__restrict__ radii, // [C, N]
    const T *__restrict__ ray_transforms,      // [C, N, 3, 3]
    // grad outputs
    const T *__restrict__ v_means2d, // [C, N, 2]
    const T *__restrict__ v_depths,  // [C, N]
    const T *__restrict__ v_normals, // [C, N, 3]
    // grad inputs
    T *__restrict__ v_ray_transforms,  // [C, N, 3, 3]
    T *__restrict__ v_means,   // [N, 3]
    T *__restrict__ v_quats,   // [N, 4]
    T *__restrict__ v_scales,  // [N, 3]
    T *__restrict__ v_viewmats // [C, 4, 4]
) {
    // parallelize over C * N.
    uint32_t idx = cg::this_grid().thread_rank();
    if (idx >= C * N || radii[idx] <= 0) {
        return;
    }
    const uint32_t cid = idx / N; // camera id
    const uint32_t gid = idx % N; // gaussian id

    // shift pointers to the current camera and gaussian
    means += gid * 3;
    viewmats += cid * 16;
    Ks += cid * 9;

    ray_transforms += idx * 9;

    v_means2d += idx * 2;
    v_depths += idx;
    v_normals += idx * 3;
    v_ray_transforms += idx * 9;

    // transform Gaussian to camera space
    mat3<T> R = mat3<T>(
        viewmats[0],
        viewmats[4],
        viewmats[8], // 1st column
        viewmats[1],
        viewmats[5],
        viewmats[9], // 2nd column
        viewmats[2],
        viewmats[6],
        viewmats[10] // 3rd column
    );
    vec3<T> t = vec3<T>(viewmats[3], viewmats[7], viewmats[11]);
    vec3<T> mean_c;
    pos_world_to_cam(R, t, glm::make_vec3(means), mean_c);

    vec4<T> quat = glm::make_vec4(quats + gid * 4);
    vec2<T> scale = glm::make_vec2(scales + gid * 3);

    mat3<T> P = mat3<T>(Ks[0], 0.0, Ks[2], 0.0, Ks[4], Ks[5], 0.0, 0.0, 1.0);

    mat3<T> _v_ray_transforms = mat3<T>(
        v_ray_transforms[0],
        v_ray_transforms[1],
        v_ray_transforms[2],
        v_ray_transforms[3],
        v_ray_transforms[4],
        v_ray_transforms[5],
        v_ray_transforms[6],
        v_ray_transforms[7],
        v_ray_transforms[8]
    );

    _v_ray_transforms[2][2] += v_depths[0];

    vec3<T> v_normal = glm::make_vec3(v_normals);

    vec3<T> v_mean(0.f);
    vec2<T> v_scale(0.f);
    vec4<T> v_quat(0.f);
    compute_ray_transforms_aabb_vjp(
        ray_transforms,
        v_means2d,
        v_normal,
        R,
        P,
        t,
        mean_c,
        quat,
        scale,
        _v_ray_transforms,
        v_quat,
        v_scale,
        v_mean
    );

    // #if __CUDA_ARCH__ >= 700
    // write out results with warp-level reduction
    auto warp = cg::tiled_partition<32>(cg::this_thread_block());
    auto warp_group_g = cg::labeled_partition(warp, gid);
    if (v_means != nullptr) {
        warpSum(v_mean, warp_group_g);
        if (warp_group_g.thread_rank() == 0) {
            v_means += gid * 3;
            GSPLAT_PRAGMA_UNROLL
            for (uint32_t i = 0; i < 3; i++) {
                gpuAtomicAdd(v_means + i, v_mean[i]);
            }
        }
    }

    // Directly output gradients w.r.t. the quaternion and scale
    warpSum(v_quat, warp_group_g);
    warpSum(v_scale, warp_group_g);
    if (warp_group_g.thread_rank() == 0) {
        v_quats += gid * 4;
        v_scales += gid * 3;
        gpuAtomicAdd(v_quats, v_quat[0]);
        gpuAtomicAdd(v_quats + 1, v_quat[1]);
        gpuAtomicAdd(v_quats + 2, v_quat[2]);
        gpuAtomicAdd(v_quats + 3, v_quat[3]);
        gpuAtomicAdd(v_scales, v_scale[0]);
        gpuAtomicAdd(v_scales + 1, v_scale[1]);
    }
}

std::tuple<torch::Tensor, torch::Tensor, torch::Tensor, torch::Tensor>
fully_fused_projection_bwd_2dgs_tensor(
    // fwd inputs
    const torch::Tensor &means,    // [N, 3]
    const torch::Tensor &quats,    // [N, 4]
    const torch::Tensor &scales,   // [N, 2]
    const torch::Tensor &viewmats, // [C, 4, 4]
    const torch::Tensor &Ks,       // [C, 3, 3]
    const uint32_t image_width,
    const uint32_t image_height,
    // fwd outputs
    const torch::Tensor &radii,  // [C, N]
    const torch::Tensor &ray_transforms, // [C, N, 3, 3]
    // grad outputs
    const torch::Tensor &v_means2d, // [C, N, 2]
    const torch::Tensor &v_depths,  // [C, N]
    const torch::Tensor &v_normals, // [C, N, 3]
    const torch::Tensor &v_ray_transforms,  // [C, N, 3, 3]
    const bool viewmats_requires_grad
) {
    GSPLAT_DEVICE_GUARD(means);
    GSPLAT_CHECK_INPUT(means);
    GSPLAT_CHECK_INPUT(quats);
    GSPLAT_CHECK_INPUT(scales);
    GSPLAT_CHECK_INPUT(viewmats);
    GSPLAT_CHECK_INPUT(Ks);
    GSPLAT_CHECK_INPUT(radii);
    GSPLAT_CHECK_INPUT(ray_transforms);
    GSPLAT_CHECK_INPUT(v_means2d);
    GSPLAT_CHECK_INPUT(v_depths);
    GSPLAT_CHECK_INPUT(v_normals);
    GSPLAT_CHECK_INPUT(v_ray_transforms);

    uint32_t N = means.size(0);    // number of gaussians
    uint32_t C = viewmats.size(0); // number of cameras
    at::cuda::CUDAStream stream = at::cuda::getCurrentCUDAStream();

    torch::Tensor v_means = torch::zeros_like(means);
    torch::Tensor v_quats = torch::zeros_like(quats);
    torch::Tensor v_scales = torch::zeros_like(scales);
    torch::Tensor v_viewmats;
    if (viewmats_requires_grad) {
        v_viewmats = torch::zeros_like(viewmats);
    }
    if (C && N) {
        fully_fused_projection_bwd_2dgs_kernel<float>
            <<<(C * N + GSPLAT_N_THREADS - 1) / GSPLAT_N_THREADS,
               GSPLAT_N_THREADS,
               0,
               stream>>>(
                C,
                N,
                means.data_ptr<float>(),
                quats.data_ptr<float>(),
                scales.data_ptr<float>(),
                viewmats.data_ptr<float>(),
                Ks.data_ptr<float>(),
                image_width,
                image_height,
                radii.data_ptr<int32_t>(),
                ray_transforms.data_ptr<float>(),
                v_means2d.data_ptr<float>(),
                v_depths.data_ptr<float>(),
                v_normals.data_ptr<float>(),
                v_ray_transforms.data_ptr<float>(),
                v_means.data_ptr<float>(),
                v_quats.data_ptr<float>(),
                v_scales.data_ptr<float>(),
                viewmats_requires_grad ? v_viewmats.data_ptr<float>() : nullptr
            );
    }
    return std::make_tuple(v_means, v_quats, v_scales, v_viewmats);
}

} // namespace gsplat