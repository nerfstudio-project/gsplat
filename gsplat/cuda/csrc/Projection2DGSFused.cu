#include "hip/hip_runtime.h"
#include <ATen/Dispatch.h>
#include <ATen/core/Tensor.h>
#include <ATen/cuda/Atomic.cuh>
#include <c10/cuda/CUDAStream.h>
#include <hip/hip_cooperative_groups.h>

#include "Common.h"
#include "Projection.h"
#include "Projection2DGS.cuh" // Utils for 2DGS Projection
#include "Utils.cuh"

namespace gsplat {

namespace cg = cooperative_groups;

template <typename scalar_t>
__global__ void projection_2dgs_fused_fwd_kernel(
    const uint32_t C,
    const uint32_t N,
    const scalar_t
        *__restrict__ means, // [N, 3]:  Gaussian means. (i.e. source points)
    const scalar_t
        *__restrict__ quats, // [N, 4]:  Quaternions (No need to be normalized):
                             // This is the rotation component (for 2D)
    const scalar_t
        *__restrict__ scales, // [N, 3]:  Scales. [N, 3] scales for x, y, z
    const scalar_t *__restrict__ viewmats, // [C, 4, 4]:  World-to-Camera
                                           // coordinate mat [R t] [0 1]
    const scalar_t
        *__restrict__ Ks, // [C, 3, 3]:  Projective transformation matrix
                          // [f_x 0  c_x]
                          // [0  f_y c_y]
                          // [0   0   1]  : f_x, f_y are focal lengths, c_x, c_y
                          // is coords for camera center on screen space
    const int32_t image_width,  // Image width  pixels
    const int32_t image_height, // Image height pixels
    const scalar_t
        near_plane, // Near clipping plane (for finite range used in z sorting)
    const scalar_t
        far_plane, // Far clipping plane (for finite range used in z sorting)
    const scalar_t radius_clip, // Radius clipping threshold (through away small
                                // primitives)
    // outputs
    int32_t *__restrict__ radii, // [C, N]   The maximum radius of the projected
                                 // Gaussians in pixel unit. Int32 tensor of
                                 // shape [C, N].
    scalar_t
        *__restrict__ means2d, // [C, N, 2] 2D means of the projected Gaussians.
    scalar_t
        *__restrict__ depths, // [C, N] The z-depth of the projected Gaussians.
    scalar_t
        *__restrict__ ray_transforms, // [C, N, 3, 3] Transformation matrices
                                      // that transform xy-planes in pixel
                                      // spaces into splat coordinates (WH)^T in
                                      // equation (9) in paper
    scalar_t *__restrict__ normals    // [C, N, 3] The normals in camera spaces.
) {

    /**
     * ===============================================
     * Initialize execution and threading variables:
     * idx: global thread index
     * cid: camera id (N is the total number of primitives, C is the number of
     cameras)
     * gid: gaussian id (N is the total number of primitives, C is the number of
     cameras)

     * THIS KERNEL LAUNCHES PER PRIMITIVE PER CAMERA i.e. C*N THREADS IN TOTAL
     * ===============================================
    */

    // parallelize over C * N.
    uint32_t idx =
        cg::this_grid().thread_rank(); // get the thread index from grid
    if (idx >= C * N) {
        return;
    }
    const uint32_t cid = idx / N; // camera id
    const uint32_t gid = idx % N; // gaussian id

    /**
     * ===============================================
     * Load data and put together camera rotation / translation
     * ===============================================
     */

    // shift pointers to the current camera and gaussian
    means += gid *
             3; // find the mean of the primitive this thread is responsible for
    viewmats += cid * 16; // step 4x4 camera matrix
    Ks += cid * 9;        // step 3x3 intrinsic matrix

    // glm is column-major but input is row-major
    // rotation component of the camera. Explicit Transpose
    mat3 R = mat3(
        viewmats[0],
        viewmats[4],
        viewmats[8], // 1st column
        viewmats[1],
        viewmats[5],
        viewmats[9], // 2nd column
        viewmats[2],
        viewmats[6],
        viewmats[10] // 3rd column
    );
    // translation component of the camera
    vec3 t = vec3(viewmats[3], viewmats[7], viewmats[11]);

    /**
     * ===============================================
     * Build ray transformation matrix from Primitive to Camera
     * in the original paper, q_ray [xz, yz, z, 1] = WH * q_uv : [u,v,1,1]
     *
     * Thus: RS_camera = R * H(P->W)

     * Since H matrix (4x4) is defined as:
     * [v_x v_y 0_vec3  t]
     * [0   0   0       1]
     *
     * thus RS_Camera defined as R * [v_x v_y 0], which gives
     * [R⋅v_x R⋅v_y 0]
     * Thus the only non zero terms will be the first two columns of R
     *
     * This gives the "affine rotation component" from uv to camera space as
     RS_camera
     *
     * the final addition component will be mean_c, which is the center of
     primitive in camera space, as
     * q_cam = RS_camera * q_uv + mean_c
     *
     * Like with homogeneous coordinates. if we encode incoming 2d points as
     [u,v,1], we can have:
     * q_cam = [RS_camera[0,1] | mean_c] * [u,v,1]
     * ===============================================
    */

    // transform Gaussian center to camera space
    vec3 mean_c;
    posW2C(R, t, glm::make_vec3(means), mean_c);

    // return this thread for overly small primitives
    if (mean_c.z < near_plane || mean_c.z > far_plane) {
        radii[idx] = 0;
        return;
    }

    quats += gid * 4;
    scales += gid * 3;

    mat3 RS_camera =
        R * quat_to_rotmat(glm::make_vec4(quats)) *
        mat3(scales[0], 0.0, 0.0, 0.0, scales[1], 0.0, 0.0, 0.0, 1.0);

    mat3 WH = mat3(RS_camera[0], RS_camera[1], mean_c);

    // projective transformation matrix: Camera -> Screen
    // when write in this order, the matrix is actually K^T as glm will read it
    // in column major order [Ks[0],  0,  0] [0,   Ks[4],  0] [Ks[2], Ks[5],  1]
    mat3 world_2_pix =
        mat3(Ks[0], 0.0, Ks[2], 0.0, Ks[4], Ks[5], 0.0, 0.0, 1.0);

    // WH is defined as [R⋅v_x, R⋅v_y, mean_c]: q_uv = [u,v,-1] -> q_cam =
    // [c1,c2,c3] here is the issue, world_2_pix is actually K^T M is thus
    // (KWH)^T = (WH)^T * K^T = (WH)^T * world_2_pix thus M stores the "row
    // majored" version of KWH, or column major version of (KWH)^T
    mat3 M = glm::transpose(WH) * world_2_pix;
    /**
     * ===============================================
     * Compute AABB
     * ===============================================
     */

    // compute AABB
    const vec3 M0 = vec3(
        M[0][0], M[0][1], M[0][2]
    ); // the first column of KWH^T, thus first row of KWH
    const vec3 M1 = vec3(
        M[1][0], M[1][1], M[1][2]
    ); // the second column of KWH^T, thus second row of KWH
    const vec3 M2 = vec3(
        M[2][0], M[2][1], M[2][2]
    ); // the third column of KWH^T, thus third row of KWH

    // we know that KWH brings [u,v,-1] to ray1, ray2, ray3] = [xz, yz, z]
    // temp_point is [1,1,-1], which is a "corner" of the UV space.
    const vec3 temp_point = vec3(1.0f, 1.0f, -1.0f);

    // ==============================================
    // trivial implementation to find mean and 1 sigma radius
    // ==============================================
    // const vec3 mean_ray = glm::transpose(M) * vec3(0.0f, 0.0f, -1.0f);
    // const vec3 temp_point_ray = glm::transpose(M) * temp_point;

    // const vec2 mean2d = vec2(mean_ray.x / mean_ray.z, mean_ray.y /
    // mean_ray.z); const vec2 half_extend_p = vec2(temp_point_ray.x /
    // temp_point_ray.z, temp_point_ray.y / temp_point_ray.z) - mean2d; const
    // vec2 half_extend = vec2(half_extend_p.x * half_extend_p.x,
    // half_extend_p.y * half_extend_p.y);

    // ==============================================
    // pro implementation
    // ==============================================
    // this is purely resulted from algebraic manipulation
    // check here for details:
    // https://github.com/hbb1/diff-surfel-rasterization/issues/8#issuecomment-2138069016
    const float distance = sum(temp_point * M2 * M2);

    // ill-conditioned primitives will have distance = 0.0f, we ignore them
    if (distance == 0.0f)
        return;

    const vec3 f = (1 / distance) * temp_point;
    const vec2 mean2d = vec2(sum(f * M0 * M2), sum(f * M1 * M2));

    const vec2 temp = {sum(f * M0 * M0), sum(f * M1 * M1)};
    const vec2 half_extend = mean2d * mean2d - temp;

    // ==============================================
    const float radius =
        ceil(3.f * sqrt(max(1e-4, max(half_extend.x, half_extend.y))));

    if (radius <= radius_clip) {
        radii[idx] = 0;
        return;
    }

    // CULLING STEP:
    // mask out gaussians outside the image region
    if (mean2d.x + radius <= 0 || mean2d.x - radius >= image_width ||
        mean2d.y + radius <= 0 || mean2d.y - radius >= image_height) {
        radii[idx] = 0;
        return;
    }

    // normals dual visible
    vec3 normal = RS_camera[2];
    // flip normal if it is pointing away from the camera
    float multipler = glm::dot(-normal, mean_c) > 0 ? 1 : -1;
    normal *= multipler;

    // write to outputs
    radii[idx] = (int32_t)radius;
    means2d[idx * 2] = mean2d.x;
    means2d[idx * 2 + 1] = mean2d.y;
    depths[idx] = mean_c.z;

    // row major storing (KWH)
    ray_transforms[idx * 9] = M0.x;
    ray_transforms[idx * 9 + 1] = M0.y;
    ray_transforms[idx * 9 + 2] = M0.z;
    ray_transforms[idx * 9 + 3] = M1.x;
    ray_transforms[idx * 9 + 4] = M1.y;
    ray_transforms[idx * 9 + 5] = M1.z;
    ray_transforms[idx * 9 + 6] = M2.x;
    ray_transforms[idx * 9 + 7] = M2.y;
    ray_transforms[idx * 9 + 8] = M2.z;

    // primitive normals
    normals[idx * 3] = normal.x;
    normals[idx * 3 + 1] = normal.y;
    normals[idx * 3 + 2] = normal.z;
}

void launch_projection_2dgs_fused_fwd_kernel(
    // inputs
    const at::Tensor means,    // [N, 3]
    const at::Tensor quats,    // [N, 4]
    const at::Tensor scales,   // [N, 3]
    const at::Tensor viewmats, // [C, 4, 4]
    const at::Tensor Ks,       // [C, 3, 3]
    const uint32_t image_width,
    const uint32_t image_height,
    const float near_plane,
    const float far_plane,
    const float radius_clip,
    // outputs
    at::Tensor radii,          // [C, N]
    at::Tensor means2d,        // [C, N, 2]
    at::Tensor depths,         // [C, N]
    at::Tensor ray_transforms, // [C, N, 3, 3]
    at::Tensor normals         // [C, N, 3]
) {
    uint32_t N = means.size(0);    // number of gaussians
    uint32_t C = viewmats.size(0); // number of cameras

    int64_t n_elements = C * N;
    dim3 threads(256);
    dim3 grid((n_elements + threads.x - 1) / threads.x);
    int64_t shmem_size = 0; // No shared memory used in this kernel

    if (n_elements == 0) {
        // skip the kernel launch if there are no elements
        return;
    }

    projection_2dgs_fused_fwd_kernel<float>
        <<<grid, threads, shmem_size, at::cuda::getCurrentCUDAStream()>>>(
            C,
            N,
            means.data_ptr<float>(),
            quats.data_ptr<float>(),
            scales.data_ptr<float>(),
            viewmats.data_ptr<float>(),
            Ks.data_ptr<float>(),
            image_width,
            image_height,
            near_plane,
            far_plane,
            radius_clip,
            radii.data_ptr<int32_t>(),
            means2d.data_ptr<float>(),
            depths.data_ptr<float>(),
            ray_transforms.data_ptr<float>(),
            normals.data_ptr<float>()
        );
}

template <typename scalar_t>
__global__ void projection_2dgs_fused_bwd_kernel(
    // fwd inputs
    const uint32_t C,
    const uint32_t N,
    const scalar_t *__restrict__ means,    // [N, 3]
    const scalar_t *__restrict__ quats,    // [N, 4]
    const scalar_t *__restrict__ scales,   // [N, 3]
    const scalar_t *__restrict__ viewmats, // [C, 4, 4]
    const scalar_t *__restrict__ Ks,       // [C, 3, 3]
    const int32_t image_width,
    const int32_t image_height,
    // fwd outputs
    const int32_t *__restrict__ radii,           // [C, N]
    const scalar_t *__restrict__ ray_transforms, // [C, N, 3, 3]
    // grad outputs
    const scalar_t *__restrict__ v_means2d, // [C, N, 2]
    const scalar_t *__restrict__ v_depths,  // [C, N]
    const scalar_t *__restrict__ v_normals, // [C, N, 3]
    // grad inputs
    scalar_t *__restrict__ v_ray_transforms, // [C, N, 3, 3]
    scalar_t *__restrict__ v_means,          // [N, 3]
    scalar_t *__restrict__ v_quats,          // [N, 4]
    scalar_t *__restrict__ v_scales,         // [N, 3]
    scalar_t *__restrict__ v_viewmats        // [C, 4, 4]
) {
    // parallelize over C * N.
    uint32_t idx = cg::this_grid().thread_rank();
    if (idx >= C * N || radii[idx] <= 0) {
        return;
    }
    const uint32_t cid = idx / N; // camera id
    const uint32_t gid = idx % N; // gaussian id

    // shift pointers to the current camera and gaussian
    means += gid * 3;
    viewmats += cid * 16;
    Ks += cid * 9;

    ray_transforms += idx * 9;

    v_means2d += idx * 2;
    v_depths += idx;
    v_normals += idx * 3;
    v_ray_transforms += idx * 9;

    // transform Gaussian to camera space
    mat3 R = mat3(
        viewmats[0],
        viewmats[4],
        viewmats[8], // 1st column
        viewmats[1],
        viewmats[5],
        viewmats[9], // 2nd column
        viewmats[2],
        viewmats[6],
        viewmats[10] // 3rd column
    );
    vec3 t = vec3(viewmats[3], viewmats[7], viewmats[11]);
    vec3 mean_c;
    posW2C(R, t, glm::make_vec3(means), mean_c);

    vec4 quat = glm::make_vec4(quats + gid * 4);
    vec2 scale = glm::make_vec2(scales + gid * 3);

    mat3 P = mat3(Ks[0], 0.0, Ks[2], 0.0, Ks[4], Ks[5], 0.0, 0.0, 1.0);

    mat3 _v_ray_transforms = mat3(
        v_ray_transforms[0],
        v_ray_transforms[1],
        v_ray_transforms[2],
        v_ray_transforms[3],
        v_ray_transforms[4],
        v_ray_transforms[5],
        v_ray_transforms[6],
        v_ray_transforms[7],
        v_ray_transforms[8]
    );

    _v_ray_transforms[2][2] += v_depths[0];

    vec3 v_normal = glm::make_vec3(v_normals);

    vec3 v_mean(0.f);
    vec2 v_scale(0.f);
    vec4 v_quat(0.f);
    compute_ray_transforms_aabb_vjp(
        ray_transforms,
        v_means2d,
        v_normal,
        R,
        P,
        t,
        mean_c,
        quat,
        scale,
        _v_ray_transforms,
        v_quat,
        v_scale,
        v_mean
    );

    // #if __CUDA_ARCH__ >= 700
    // write out results with warp-level reduction
    auto warp = cg::tiled_partition<32>(cg::this_thread_block());
    auto warp_group_g = cg::labeled_partition(warp, gid);
    if (v_means != nullptr) {
        warpSum(v_mean, warp_group_g);
        if (warp_group_g.thread_rank() == 0) {
            v_means += gid * 3;
#pragma unroll
            for (uint32_t i = 0; i < 3; i++) {
                gpuAtomicAdd(v_means + i, v_mean[i]);
            }
        }
    }

    // Directly output gradients w.r.t. the quaternion and scale
    warpSum(v_quat, warp_group_g);
    warpSum(v_scale, warp_group_g);
    if (warp_group_g.thread_rank() == 0) {
        v_quats += gid * 4;
        v_scales += gid * 3;
        gpuAtomicAdd(v_quats, v_quat[0]);
        gpuAtomicAdd(v_quats + 1, v_quat[1]);
        gpuAtomicAdd(v_quats + 2, v_quat[2]);
        gpuAtomicAdd(v_quats + 3, v_quat[3]);
        gpuAtomicAdd(v_scales, v_scale[0]);
        gpuAtomicAdd(v_scales + 1, v_scale[1]);
    }
}

void launch_projection_2dgs_fused_bwd_kernel(
    // fwd inputs
    const at::Tensor means,    // [N, 3]
    const at::Tensor quats,    // [N, 4]
    const at::Tensor scales,   // [N, 3]
    const at::Tensor viewmats, // [C, 4, 4]
    const at::Tensor Ks,       // [C, 3, 3]
    const uint32_t image_width,
    const uint32_t image_height,
    // fwd outputs
    const at::Tensor radii,          // [C, N]
    const at::Tensor ray_transforms, // [C, N, 3, 3]
    // grad outputs
    const at::Tensor v_means2d,        // [C, N, 2]
    const at::Tensor v_depths,         // [C, N]
    const at::Tensor v_normals,        // [C, N, 3]
    const at::Tensor v_ray_transforms, // [C, N, 3, 3]
    const bool viewmats_requires_grad,
    // outputs
    at::Tensor v_means,   // [C, N, 3]
    at::Tensor v_quats,   // [C, N, 4]
    at::Tensor v_scales,  // [C, N, 3]
    at::Tensor v_viewmats // [C, 4, 4]
) {
    uint32_t N = means.size(0);    // number of gaussians
    uint32_t C = viewmats.size(0); // number of cameras

    int64_t n_elements = C * N;
    dim3 threads(256);
    dim3 grid((n_elements + threads.x - 1) / threads.x);
    int64_t shmem_size = 0; // No shared memory used in this kernel

    if (n_elements == 0) {
        // skip the kernel launch if there are no elements
        return;
    }

    projection_2dgs_fused_bwd_kernel<float>
        <<<grid, threads, shmem_size, at::cuda::getCurrentCUDAStream()>>>(
            C,
            N,
            means.data_ptr<float>(),
            quats.data_ptr<float>(),
            scales.data_ptr<float>(),
            viewmats.data_ptr<float>(),
            Ks.data_ptr<float>(),
            image_width,
            image_height,
            radii.data_ptr<int32_t>(),
            ray_transforms.data_ptr<float>(),
            v_means2d.data_ptr<float>(),
            v_depths.data_ptr<float>(),
            v_normals.data_ptr<float>(),
            v_ray_transforms.data_ptr<float>(),
            v_means.data_ptr<float>(),
            v_quats.data_ptr<float>(),
            v_scales.data_ptr<float>(),
            viewmats_requires_grad ? v_viewmats.data_ptr<float>() : nullptr
        );
}

} // namespace gsplat
