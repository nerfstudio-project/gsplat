#include "hip/hip_runtime.h"
#include "bindings.h"
#include "helpers.cuh"
#include "types.cuh"
#include <hip/hip_cooperative_groups.h>
#include <hipcub/hipcub.hpp>
#include <hip/hip_runtime.h>

namespace cg = cooperative_groups;

/****************************************************************************
 * Rasterization to Pixels Backward Pass
 ****************************************************************************/

template <uint32_t COLOR_DIM, typename S>
__global__ void raytracing_to_pixels_bwd_kernel(
    const uint32_t C, const uint32_t N, const uint32_t n_isects, const bool packed,
    // fwd inputs
    const vec2<S> *__restrict__ means2d, // [C, N, 2] or [nnz, 2]
    const vec3<S> *__restrict__ conics,  // [C, N, 3] or [nnz, 3]
    const S *__restrict__ colors,        // [C, N, COLOR_DIM] or [nnz, COLOR_DIM]
    const S *__restrict__ opacities,     // [C, N] or [nnz]
    const vec10<S> *__restrict__ view2gaussians, // [C, N, 10] or [nnz, 10]
    const S *__restrict__ Ks,                     // [C, 3, 3]
    const S *__restrict__ backgrounds,   // [C, COLOR_DIM] or [nnz, COLOR_DIM]
    const uint32_t image_width, const uint32_t image_height, const uint32_t tile_size,
    const uint32_t tile_width, const uint32_t tile_height,
    const int32_t *__restrict__ tile_offsets, // [C, tile_height, tile_width]
    const int32_t *__restrict__ flatten_ids,  // [n_isects]
    // fwd outputs
    const S *__restrict__ render_alphas,  // [C, image_height, image_width, 1]
    const int32_t *__restrict__ last_ids, // [C, image_height, image_width]
    // grad outputs
    const S *__restrict__ v_render_colors, // [C, image_height, image_width,
                                           // COLOR_DIM]
    const S *__restrict__ v_render_alphas, // [C, image_height, image_width, 1]
    // grad inputs
    vec2<S> *__restrict__ v_means2d_abs, // [C, N, 2] or [nnz, 2]
    vec2<S> *__restrict__ v_means2d,     // [C, N, 2] or [nnz, 2]
    vec3<S> *__restrict__ v_conics,      // [C, N, 3] or [nnz, 3]
    S *__restrict__ v_colors,            // [C, N, COLOR_DIM] or [nnz, COLOR_DIM]
    S *__restrict__ v_opacities,         // [C, N] or [nnz]
    S *__restrict__ v_view2gaussians     // [C, N, 10] or [nnz, 10]
) {
    auto block = cg::this_thread_block();
    uint32_t camera_id = block.group_index().x;
    uint32_t tile_id = block.group_index().y * tile_width + block.group_index().z;
    uint32_t i = block.group_index().y * tile_size + block.thread_index().y;
    uint32_t j = block.group_index().z * tile_size + block.thread_index().x;

    tile_offsets += camera_id * tile_height * tile_width;
    render_alphas += camera_id * image_height * image_width;
    last_ids += camera_id * image_height * image_width * 2;
    v_render_colors += camera_id * image_height * image_width * (COLOR_DIM + 1 + 3);
    v_render_alphas += camera_id * image_height * image_width;
    Ks += camera_id * 9;
    if (backgrounds != nullptr) {
        backgrounds += camera_id * COLOR_DIM;
    }

    const S px = (S)j + 0.5f;
    const S py = (S)i + 0.5f;
    // clamp this value to the last pixel
    const int32_t pix_id = min(i * image_width + j, image_width * image_height - 1);

    const S focal_x = Ks[0];
    const S focal_y = Ks[4];
    const S cx = Ks[2];
    const S cy = Ks[5];
    const vec3<S> ray = {(px - cx) / focal_x, (py - cy) / focal_y, 1.0};

    // keep not rasterizing threads around for reading data
    bool inside = (i < image_height && j < image_width);

    // have all threads in tile process the same gaussians in batches
    // first collect gaussians between range.x and range.y in batches
    // which gaussians to look through in this tile
    int32_t range_start = tile_offsets[tile_id];
    int32_t range_end =
        (camera_id == C - 1) && (tile_id == tile_width * tile_height - 1)
            ? n_isects
            : tile_offsets[tile_id + 1];
    const uint32_t block_size = block.size();
    const uint32_t num_batches =
        (range_end - range_start + block_size - 1) / block_size;

    extern __shared__ int s[];
    int32_t *id_batch = (int32_t *)s; // [block_size]
    vec3<S> *xy_opacity_batch =
        reinterpret_cast<vec3<float> *>(&id_batch[block_size]); // [block_size]
    vec3<S> *conic_batch =
        reinterpret_cast<vec3<float> *>(&xy_opacity_batch[block_size]); // [block_size]
    vec10<S> *view2gaussian_batch =
        reinterpret_cast<vec10<float> *>(&conic_batch[block_size]); // [block_size]
    S *rgbs_batch = (S *)&view2gaussian_batch[block_size]; // [block_size * COLOR_DIM]
    

    // this is the T AFTER the last gaussian in this pixel
    S T_final = 1.0f - render_alphas[pix_id];
    S T = T_final;
    // the contribution from gaussians behind the current one
    S buffer[COLOR_DIM] = {0.f};
    S buffer_normal[3] = {0.f};
    // index of last gaussian to contribute to this pixel
    const int32_t bin_final = inside ? last_ids[pix_id * 2] : 0;
    const int32_t bin_max = inside? last_ids[pix_id * 2 + 1] : 0;
    // df/d_out for this pixel
    S v_render_c[COLOR_DIM];
    PRAGMA_UNROLL
    for (uint32_t k = 0; k < COLOR_DIM; ++k) {
        v_render_c[k] = v_render_colors[pix_id * (COLOR_DIM + 1 + 3) + k];
    }
    const S v_render_a = v_render_alphas[pix_id];
    // gradient for normal and depth
    S v_render_normal[3];
    PRAGMA_UNROLL
    for (uint32_t k = 0; k < 3; ++k) {
        v_render_normal[k] = v_render_colors[pix_id * (COLOR_DIM + 1 + 3) + COLOR_DIM + k];
    }
    S v_render_depth = v_render_colors[pix_id * (COLOR_DIM + 1 + 3) + COLOR_DIM + 3];

    // collect and process batches of gaussians
    // each thread loads one gaussian at a time before rasterizing
    const uint32_t tr = block.thread_rank();
    cg::thread_block_tile<32> warp = cg::tiled_partition<32>(block);
    const int32_t warp_bin_final = cg::reduce(warp, bin_final, cg::greater<int>());
    for (uint32_t b = 0; b < num_batches; ++b) {
        // resync all threads before writing next batch of shared mem
        block.sync();

        // each thread fetch 1 gaussian from back to front
        // 0 index will be furthest back in batch
        // index of gaussian to load
        // batch end is the index of the last gaussian in the batch
        // These values can be negative so must be int32 instead of uint32
        const int32_t batch_end = range_end - 1 - block_size * b;
        const int32_t batch_size = min(block_size, batch_end + 1 - range_start);
        const int32_t idx = batch_end - tr;
        if (idx >= range_start) {
            int32_t g = flatten_ids[idx]; // flatten index in [C * N] or [nnz]
            id_batch[tr] = g;
            const vec2<S> xy = means2d[g];
            const S opac = opacities[g];
            xy_opacity_batch[tr] = {xy.x, xy.y, opac};
            conic_batch[tr] = conics[g];
            view2gaussian_batch[tr] = view2gaussians[g];
            PRAGMA_UNROLL
            for (uint32_t k = 0; k < COLOR_DIM; ++k) {
                rgbs_batch[tr * COLOR_DIM + k] = colors[g * COLOR_DIM + k];
            }
        }
        // wait for other threads to collect the gaussians in batch
        block.sync();
        // process gaussians in the current batch for this pixel
        // 0 index is the furthest back gaussian in the batch
        for (uint32_t t = max(0, batch_end - warp_bin_final); t < batch_size; ++t) {
            bool valid = inside;
            if (batch_end - t > bin_final) {
                valid = 0;
            }
            S alpha;
            S opac;
            vec2<S> delta;
            vec3<S> conic;
            const vec10<S> view2gaussian = view2gaussian_batch[t];
            vec3<S> normal;
            S vis;
            S AA, BB, CC, depth, min_value, power;

            if (valid) {
                conic = conic_batch[t];
                vec3<S> xy_opac = xy_opacity_batch[t];
                
                opac = xy_opac.z;
                delta = {xy_opac.x - px, xy_opac.y - py};
                // S sigma =
                //     0.5f * (conic.x * delta.x * delta.x + conic.z * delta.y * delta.y) +
                //     conic.y * delta.x * delta.y;
                // vis = __expf(-sigma);
                // alpha = min(0.999f, opac * vis);

                normal = {
                    view2gaussian[0] * ray.x + view2gaussian[1] * ray.y + view2gaussian[2], 
                    view2gaussian[1] * ray.x + view2gaussian[3] * ray.y + view2gaussian[4],
                    view2gaussian[2] * ray.x + view2gaussian[4] * ray.y + view2gaussian[5]
                };
            
                // use AA, BB, CC so that the name is unique
                AA = ray.x * normal[0] + ray.y * normal[1] + normal[2];
                BB = 2 * (view2gaussian[6] * ray.x + view2gaussian[7] * ray.y + view2gaussian[8]);
                CC = view2gaussian[9];
                
                // t is the depth of the gaussian
                depth = -BB/(2*AA);
                //TODO take near plane as input
                #define NEAR_PLANE 0.01f
                // depth must be positive otherwise it is not valid and we skip it
                if (depth <= NEAR_PLANE)
                    valid = false;

                // the scale of the gaussian is 1.f / sqrt(AA)
			    min_value = -(BB/AA) * (BB/4.) + CC;

                power = -0.5f * min_value;
                if (power > 0.0f){
                    power = 0.0f;
                }
                
                vis = exp(power);
                alpha = min(0.999f, opac * vis);

                if (alpha < 1.f / 255.f) {
                    valid = false;
                }
            }

            // if all threads are inactive in this warp, skip this loop
            if (!warp.any(valid)) {
                continue;
            }
            S v_rgb_local[COLOR_DIM] = {0.f};
            S dL_dnormal_normalized[3] = {0.f};
            vec3<S> v_conic_local = {0.f, 0.f, 0.f};
            vec2<S> v_xy_local = {0.f, 0.f};
            vec2<S> v_xy_abs_local = {0.f, 0.f};
            S v_opacity_local = 0.f;
            S v_view2gaussian_local[10] = {0.f};
            // initialize everything to 0, only set if the lane is valid
            if (valid) {
                
                const S length = sqrt(normal[0] * normal[0] + normal[1] * normal[1] + normal[2] * normal[2] + 1e-7);
			    const vec3<S> normal_normalized = { -normal[0] / length, -normal[1] / length, -normal[2] / length };

                // compute the current T for this gaussian
                S ra = 1.0f / (1.0f - alpha);
                T *= ra;
                // update v_rgb for this gaussian
                const S fac = alpha * T;
                PRAGMA_UNROLL
                for (uint32_t k = 0; k < COLOR_DIM; ++k) {
                    v_rgb_local[k] = fac * v_render_c[k];
                }
                // contribution from normal and depth
                PRAGMA_UNROLL
                for (uint32_t k = 0; k < 3; ++k) {
                    dL_dnormal_normalized[k] = fac * v_render_normal[k];
                }
                // contribution from this pixel
                S v_alpha = 0.f;
                for (uint32_t k = 0; k < COLOR_DIM; ++k) {
                    v_alpha += (rgbs_batch[t * COLOR_DIM + k] * T - buffer[k] * ra) *
                               v_render_c[k];
                }
                // contribution from this pixel's normal
                for (uint32_t k = 0; k < 3; ++k) {
                    v_alpha += (normal_normalized[k] * T - buffer_normal[k] * ra) *
                               v_render_normal[k];
                }

                v_alpha += T_final * ra * v_render_a;
                // contribution from background pixel
                if (backgrounds != nullptr) {
                    S accum = 0.f;
                    PRAGMA_UNROLL
                    for (uint32_t k = 0; k < COLOR_DIM; ++k) {
                        accum += backgrounds[k] * v_render_c[k];
                    }
                    v_alpha += -T_final * ra * accum;
                }
                // here is different to 3DGS, in 3DGS the gradient is computed even if opac * vis > 0.999f
                if (opac * vis <= 0.999f) {
                    const S dL_dG = opac * v_alpha;
                    const S v_sigma = -opac * vis * v_alpha;
                    v_xy_local = {v_sigma * (conic.x * delta.x + conic.y * delta.y),
                                  v_sigma * (conic.y * delta.x + conic.z * delta.y)};

                    if (v_means2d_abs != nullptr) {
                        v_xy_abs_local = {abs(v_xy_local.x), abs(v_xy_local.y)};
                    }
                    v_opacity_local = vis * v_alpha;

                    // gradient of depth
                    S dL_dt = 0.0f;
                    vec3<S> dL_dnormal = {0.0f, 0.0f, 0.0f};
                    // float length = sqrt(normal[0] * normal[0] + normal[1] * normal[1] + normal[2] * normal[2] + 1e-7);
                    // const float normal_normalized[3] = { -normal[0] / length, -normal[1] / length, -normal[2] / length};
                    S dL_dlength = (dL_dnormal_normalized[0] * normal[0] + dL_dnormal_normalized[1] * normal[1] + dL_dnormal_normalized[2] * normal[2]);
                    dL_dlength *= 1.f / (length * length);
                    dL_dnormal += vec3<S>(
                        (-dL_dnormal_normalized[0] + dL_dlength * normal[0]) / length,
                        (-dL_dnormal_normalized[1] + dL_dlength * normal[1]) / length,
                        (-dL_dnormal_normalized[2] + dL_dlength * normal[2]) / length
                    );

                    if (batch_end - t == bin_max){
                        dL_dt += v_render_depth;
                    }

                    // vis = exp(power);
                    const S dG_dpower = vis;
                    const S dL_dpower = dL_dG * dG_dpower;

                    // // float power = -0.5f * min_value;
                    const S dL_dmin_value = dL_dpower * -0.5f;
                    // float min_value = -(BB*BB)/(4*AA) + CC;
                    // const float dL_dA = dL_dmin_value * (BB*BB)/4 *  1. / (AA*AA);
                    S dL_dA = dL_dmin_value * (BB / AA) * (BB / AA) / 4.f;
                    S dL_dB = dL_dmin_value * -BB / (2 *AA);
                    S dL_dC = dL_dmin_value * 1.0f;
                    // from depth = -BB/(2*AA)
                    dL_dA += dL_dt * BB / (2 * AA * AA);
                    dL_dB += dL_dt * -1.f / (2 * AA);

                    // const float normal[3] = { view2gaussian_j[0] * ray.x + view2gaussian_j[1] * ray.y + view2gaussian_j[2], 
                    // 						view2gaussian_j[1] * ray.x + view2gaussian_j[3] * ray.y + view2gaussian_j[4],
                    // 						view2gaussian_j[2] * ray.x + view2gaussian_j[4] * ray.y + view2gaussian_j[5]};

                    // use AA, BB, CC so that the name is unique
                    // float AA = ray.x * normal[0] + ray.y * normal[1] + normal[2];
                    // float BB = 2 * (view2gaussian_j[6] * ray_point.x + view2gaussian_j[7] * ray_point.y + view2gaussian_j[8]);
                    // float CC = view2gaussian_j[9];
                    dL_dnormal[0] += dL_dA * ray.x;
                    dL_dnormal[1] += dL_dA * ray.y;
                    dL_dnormal[2] += dL_dA;

                    // write the gradients to global memory directly
                    // atomicAdd(&(dL_dview2gaussian[global_id * 10 + 0]), dL_dnormal[0] * ray.x);
                    // atomicAdd(&(dL_dview2gaussian[global_id * 10 + 1]), dL_dnormal[0] * ray.y + dL_dnormal[1] * ray.x);
                    // atomicAdd(&(dL_dview2gaussian[global_id * 10 + 2]), dL_dnormal[0] + dL_dnormal[2] * ray.x);
                    // atomicAdd(&(dL_dview2gaussian[global_id * 10 + 3]), dL_dnormal[1] * ray.y);
                    // atomicAdd(&(dL_dview2gaussian[global_id * 10 + 4]), dL_dnormal[1] + dL_dnormal[2] * ray.y);
                    // atomicAdd(&(dL_dview2gaussian[global_id * 10 + 5]), dL_dnormal[2]);
                    // atomicAdd(&(dL_dview2gaussian[global_id * 10 + 6]), dL_dB * 2 * ray.x);
                    // atomicAdd(&(dL_dview2gaussian[global_id * 10 + 7]), dL_dB * 2 * ray.y);
                    // atomicAdd(&(dL_dview2gaussian[global_id * 10 + 8]), dL_dB * 2);
                    // atomicAdd(&(dL_dview2gaussian[global_id * 10 + 9]), dL_dC);
                    v_view2gaussian_local[0] = dL_dnormal[0] * ray.x;
                    v_view2gaussian_local[1] = dL_dnormal[0] * ray.y + dL_dnormal[1] * ray.x;
                    v_view2gaussian_local[2] = dL_dnormal[0] + dL_dnormal[2] * ray.x;
                    v_view2gaussian_local[3] = dL_dnormal[1] * ray.y;
                    v_view2gaussian_local[4] = dL_dnormal[1] + dL_dnormal[2] * ray.y;
                    v_view2gaussian_local[5] = dL_dnormal[2];
                    v_view2gaussian_local[6] = dL_dB * 2 * ray.x;
                    v_view2gaussian_local[7] = dL_dB * 2 * ray.y;
                    v_view2gaussian_local[8] = dL_dB * 2;
                    v_view2gaussian_local[9] = dL_dC;
                }

                PRAGMA_UNROLL
                for (uint32_t k = 0; k < COLOR_DIM; ++k) {
                    buffer[k] += rgbs_batch[t * COLOR_DIM + k] * fac;
                }
                PRAGMA_UNROLL
                for (uint32_t k = 0; k < 3; ++k) {
                    buffer_normal[k] += normal_normalized[k] * fac;
                }
            }
            warpSum<COLOR_DIM, S>(v_rgb_local, warp);
            warpSum<10, S>(v_view2gaussian_local, warp);
            warpSum<decltype(warp), S>(v_conic_local, warp);
            warpSum<decltype(warp), S>(v_xy_local, warp);
            if (v_means2d_abs != nullptr) {
                warpSum<decltype(warp), S>(v_xy_abs_local, warp);
            }
            warpSum<decltype(warp), S>(v_opacity_local, warp);
            if (warp.thread_rank() == 0) {
                int32_t g = id_batch[t]; // flatten index in [C * N] or [nnz]
                S *v_rgb_ptr = (S *)(v_colors) + COLOR_DIM * g;
                PRAGMA_UNROLL
                for (uint32_t k = 0; k < COLOR_DIM; ++k) {
                    gpuAtomicAdd(v_rgb_ptr + k, v_rgb_local[k]);
                }

                S *v_conic_ptr = (S *)(v_conics) + 3 * g;
                gpuAtomicAdd(v_conic_ptr, v_conic_local.x);
                gpuAtomicAdd(v_conic_ptr + 1, v_conic_local.y);
                gpuAtomicAdd(v_conic_ptr + 2, v_conic_local.z);

                S *v_xy_ptr = (S *)(v_means2d) + 2 * g;
                gpuAtomicAdd(v_xy_ptr, v_xy_local.x);
                gpuAtomicAdd(v_xy_ptr + 1, v_xy_local.y);

                if (v_means2d_abs != nullptr) {
                    S *v_xy_abs_ptr = (S *)(v_means2d_abs) + 2 * g;
                    gpuAtomicAdd(v_xy_abs_ptr, v_xy_abs_local.x);
                    gpuAtomicAdd(v_xy_abs_ptr + 1, v_xy_abs_local.y);
                }

                gpuAtomicAdd(v_opacities + g, v_opacity_local);
                PRAGMA_UNROLL
                for (uint32_t k = 0; k < 10; ++k) {
                    gpuAtomicAdd((S *)(v_view2gaussians) + 10 * g + k, v_view2gaussian_local[k]);
                }
            }
        }
    }
}

template <uint32_t CDIM>
std::tuple<torch::Tensor, torch::Tensor, torch::Tensor, torch::Tensor, torch::Tensor, torch::Tensor>
call_kernel_with_dim(
    // Gaussian parameters
    const torch::Tensor &means2d,                   // [C, N, 2] or [nnz, 2]
    const torch::Tensor &conics,                    // [C, N, 3] or [nnz, 3]
    const torch::Tensor &colors,                    // [C, N, 3] or [nnz, 3]
    const torch::Tensor &opacities,                 // [C, N] or [nnz]
    const torch::Tensor &view2gaussians,            // [C, N, 10] or [nnz, 10]
    const torch::Tensor &Ks,                        // [C, 3, 3]
    const at::optional<torch::Tensor> &backgrounds, // [C, 3]
    // image size
    const uint32_t image_width, const uint32_t image_height, const uint32_t tile_size,
    // intersections
    const torch::Tensor &tile_offsets, // [C, tile_height, tile_width]
    const torch::Tensor &flatten_ids,  // [n_isects]
    // forward outputs
    const torch::Tensor &render_alphas, // [C, image_height, image_width, 1]
    const torch::Tensor &last_ids,      // [C, image_height, image_width]
    // gradients of outputs
    const torch::Tensor &v_render_colors, // [C, image_height, image_width, 3]
    const torch::Tensor &v_render_alphas, // [C, image_height, image_width, 1]
    // options
    bool absgrad) {

    DEVICE_GUARD(means2d);
    CHECK_INPUT(means2d);
    CHECK_INPUT(conics);
    CHECK_INPUT(colors);
    CHECK_INPUT(opacities);
    CHECK_INPUT(view2gaussians);
    CHECK_INPUT(Ks);
    CHECK_INPUT(tile_offsets);
    CHECK_INPUT(flatten_ids);
    CHECK_INPUT(render_alphas);
    CHECK_INPUT(last_ids);
    CHECK_INPUT(v_render_colors);
    CHECK_INPUT(v_render_alphas);
    if (backgrounds.has_value()) {
        CHECK_INPUT(backgrounds.value());
    }

    bool packed = means2d.dim() == 2;

    uint32_t C = tile_offsets.size(0);         // number of cameras
    uint32_t N = packed ? 0 : means2d.size(1); // number of gaussians
    uint32_t n_isects = flatten_ids.size(0);
    uint32_t COLOR_DIM = colors.size(-1);
    uint32_t tile_height = tile_offsets.size(1);
    uint32_t tile_width = tile_offsets.size(2);

    // Each block covers a tile on the image. In total there are
    // C * tile_height * tile_width blocks.
    dim3 threads = {tile_size, tile_size, 1};
    dim3 blocks = {C, tile_height, tile_width};

    torch::Tensor v_means2d = torch::zeros_like(means2d);
    torch::Tensor v_conics = torch::zeros_like(conics);
    torch::Tensor v_colors = torch::zeros_like(colors);
    torch::Tensor v_opacities = torch::zeros_like(opacities);
    torch::Tensor v_means2d_abs;
    torch::Tensor v_view2gaussians = torch::zeros_like(view2gaussians);
    if (absgrad) {
        v_means2d_abs = torch::zeros_like(means2d);
    }

    if (n_isects) {
        const uint32_t shared_mem = tile_size * tile_size *
                                    (sizeof(int32_t) + sizeof(vec3<float>) +
                                     sizeof(vec3<float>) + sizeof(vec10<float>) + sizeof(float) * COLOR_DIM);
        at::cuda::CUDAStream stream = at::cuda::getCurrentCUDAStream();

        if (hipFuncSetAttribute(reinterpret_cast<const void*>(raytracing_to_pixels_bwd_kernel<CDIM), float>,
                                 hipFuncAttributeMaxDynamicSharedMemorySize,
                                 shared_mem) != hipSuccess) {
            AT_ERROR("Failed to set maximum shared memory size (requested ", shared_mem,
                     " bytes), try lowering tile_size.");
        }
        raytracing_to_pixels_bwd_kernel<CDIM, float>
            <<<blocks, threads, shared_mem, stream>>>(
                C, N, n_isects, packed,
                reinterpret_cast<vec2<float> *>(means2d.data_ptr<float>()),
                reinterpret_cast<vec3<float> *>(conics.data_ptr<float>()),
                colors.data_ptr<float>(), opacities.data_ptr<float>(),
                reinterpret_cast<vec10<float> *>(view2gaussians.data_ptr<float>()),
                Ks.data_ptr<float>(),
                backgrounds.has_value() ? backgrounds.value().data_ptr<float>()
                                        : nullptr,
                image_width, image_height, tile_size, tile_width, tile_height,
                tile_offsets.data_ptr<int32_t>(), flatten_ids.data_ptr<int32_t>(),
                render_alphas.data_ptr<float>(), last_ids.data_ptr<int32_t>(),
                v_render_colors.data_ptr<float>(), v_render_alphas.data_ptr<float>(),
                absgrad
                    ? reinterpret_cast<vec2<float> *>(v_means2d_abs.data_ptr<float>())
                    : nullptr,
                reinterpret_cast<vec2<float> *>(v_means2d.data_ptr<float>()),
                reinterpret_cast<vec3<float> *>(v_conics.data_ptr<float>()),
                v_colors.data_ptr<float>(), v_opacities.data_ptr<float>(),
                v_view2gaussians.data_ptr<float>());
    }

    return std::make_tuple(v_means2d_abs, v_means2d, v_conics, v_colors, v_opacities, v_view2gaussians);
}

std::tuple<torch::Tensor, torch::Tensor, torch::Tensor, torch::Tensor, torch::Tensor, torch::Tensor>
raytracing_to_pixels_bwd_tensor(
    // Gaussian parameters
    const torch::Tensor &means2d,                   // [C, N, 2] or [nnz, 2]
    const torch::Tensor &conics,                    // [C, N, 3] or [nnz, 3]
    const torch::Tensor &colors,                    // [C, N, 3] or [nnz, 3]
    const torch::Tensor &opacities,                 // [C, N] or [nnz]
    const torch::Tensor &view2gaussians,            // [C, N, 10] or [nnz, 10]
    const torch::Tensor &Ks,                        // [C, 3, 3]
    const at::optional<torch::Tensor> &backgrounds, // [C, 3]
    // image size
    const uint32_t image_width, const uint32_t image_height, const uint32_t tile_size,
    // intersections
    const torch::Tensor &tile_offsets, // [C, tile_height, tile_width]
    const torch::Tensor &flatten_ids,  // [n_isects]
    // forward outputs
    const torch::Tensor &render_alphas, // [C, image_height, image_width, 1]
    const torch::Tensor &last_ids,      // [C, image_height, image_width]
    // gradients of outputs
    const torch::Tensor &v_render_colors, // [C, image_height, image_width, 3]
    const torch::Tensor &v_render_alphas, // [C, image_height, image_width, 1]
    // options
    bool absgrad) {

    CHECK_INPUT(colors);
    uint32_t COLOR_DIM = colors.size(-1);

#define __GS__CALL_(N)                                                                 \
    case N:                                                                            \
        return call_kernel_with_dim<N>(                                                \
            means2d, conics, colors, opacities, view2gaussians, Ks,                     \
            backgrounds, image_width,                                                  \
            image_height, tile_size, tile_offsets, flatten_ids, render_alphas,         \
            last_ids, v_render_colors, v_render_alphas, absgrad);

    switch (COLOR_DIM) {
        __GS__CALL_(1)
        __GS__CALL_(2)
        __GS__CALL_(3)
        __GS__CALL_(4)
        __GS__CALL_(5)
        __GS__CALL_(8)
        __GS__CALL_(9)
        __GS__CALL_(16)
        __GS__CALL_(17)
        __GS__CALL_(32)
        __GS__CALL_(33)
        __GS__CALL_(64)
        __GS__CALL_(65)
        __GS__CALL_(128)
        __GS__CALL_(129)
        __GS__CALL_(256)
        __GS__CALL_(257)
        __GS__CALL_(512)
        __GS__CALL_(513)
    default:
        AT_ERROR("Unsupported number of channels: ", COLOR_DIM);
    }
}
