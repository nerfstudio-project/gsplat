#include "hip/hip_runtime.h"
#include <ATen/Dispatch.h>
#include <ATen/core/Tensor.h>
#include <c10/cuda/CUDAStream.h>
#include <hip/hip_cooperative_groups.h>

// for CUB_WRAPPER
#include <c10/cuda/CUDACachingAllocator.h>
#include <hipcub/hipcub.hpp>

#include "Common.h"
#include "Intersect.h"
#include "Utils.cuh"

namespace gsplat {

namespace cg = cooperative_groups;

// Evaluate spherical harmonics bases at unit direction for high orders using
// approach described by Efficient Spherical Harmonic Evaluation, Peter-Pike
// Sloan, JCGT 2013 See https://jcgt.org/published/0002/02/06/ for reference
// implementation

template <typename scalar_t>
__global__ void intersect_tile_kernel(
    // if the data is [C, N, ...] or [nnz, ...] (packed)
    const bool packed,
    // parallelize over C * N, only used if packed is False
    const uint32_t C,
    const uint32_t N,
    // parallelize over nnz, only used if packed is True
    const uint32_t nnz,
    const int64_t *__restrict__ camera_ids,   // [nnz] optional
    const int64_t *__restrict__ gaussian_ids, // [nnz] optional
    // data
    const scalar_t *__restrict__ means2d,            // [C, N, 2] or [nnz, 2]
    const int32_t *__restrict__ radii,               // [C, N] or [nnz]
    const scalar_t *__restrict__ depths,             // [C, N] or [nnz]
    const int64_t *__restrict__ cum_tiles_per_gauss, // [C, N] or [nnz]
    const uint32_t tile_size,
    const uint32_t tile_width,
    const uint32_t tile_height,
    const uint32_t tile_n_bits,
    int32_t *__restrict__ tiles_per_gauss, // [C, N] or [nnz]
    int64_t *__restrict__ isect_ids,       // [n_isects]
    int32_t *__restrict__ flatten_ids      // [n_isects]
) {
    // parallelize over C * N.
    uint32_t idx = cg::this_grid().thread_rank();
    bool first_pass = cum_tiles_per_gauss == nullptr;
    if (idx >= (packed ? nnz : C * N)) {
        return;
    }

    const float radius = radii[idx];
    if (radius <= 0) {
        if (first_pass) {
            tiles_per_gauss[idx] = 0;
        }
        return;
    }

    vec2 mean2d = glm::make_vec2(means2d + 2 * idx);

    float tile_radius = radius / static_cast<float>(tile_size);
    float tile_x = mean2d.x / static_cast<float>(tile_size);
    float tile_y = mean2d.y / static_cast<float>(tile_size);

    // tile_min is inclusive, tile_max is exclusive
    uint2 tile_min, tile_max;
    tile_min.x = min(max(0, (uint32_t)floor(tile_x - tile_radius)), tile_width);
    tile_min.y =
        min(max(0, (uint32_t)floor(tile_y - tile_radius)), tile_height);
    tile_max.x = min(max(0, (uint32_t)ceil(tile_x + tile_radius)), tile_width);
    tile_max.y = min(max(0, (uint32_t)ceil(tile_y + tile_radius)), tile_height);

    if (first_pass) {
        // first pass only writes out tiles_per_gauss
        tiles_per_gauss[idx] = static_cast<int32_t>(
            (tile_max.y - tile_min.y) * (tile_max.x - tile_min.x)
        );
        return;
    }

    int64_t cid; // camera id
    if (packed) {
        // parallelize over nnz
        cid = camera_ids[idx];
        // gid = gaussian_ids[idx];
    } else {
        // parallelize over C * N
        cid = idx / N;
        // gid = idx % N;
    }
    const int64_t cid_enc = cid << (32 + tile_n_bits);

    int64_t depth_id_enc = (int64_t) * (int32_t *)&(depths[idx]);
    int64_t cur_idx = (idx == 0) ? 0 : cum_tiles_per_gauss[idx - 1];
    for (int32_t i = tile_min.y; i < tile_max.y; ++i) {
        for (int32_t j = tile_min.x; j < tile_max.x; ++j) {
            int64_t tile_id = i * tile_width + j;
            // e.g. tile_n_bits = 22:
            // camera id (10 bits) | tile id (22 bits) | depth (32 bits)
            isect_ids[cur_idx] = cid_enc | (tile_id << 32) | depth_id_enc;
            // the flatten index in [C * N] or [nnz]
            flatten_ids[cur_idx] = static_cast<int32_t>(idx);
            ++cur_idx;
        }
    }
}

void launch_intersect_tile_kernel(
    // inputs
    const at::Tensor means2d,                    // [C, N, 2] or [nnz, 2]
    const at::Tensor radii,                      // [C, N] or [nnz]
    const at::Tensor depths,                     // [C, N] or [nnz]
    const at::optional<at::Tensor> camera_ids,   // [nnz]
    const at::optional<at::Tensor> gaussian_ids, // [nnz]
    const uint32_t C,
    const uint32_t tile_size,
    const uint32_t tile_width,
    const uint32_t tile_height,
    const at::optional<at::Tensor> cum_tiles_per_gauss, // [C, N] or [nnz]
    // outputs
    at::optional<at::Tensor> tiles_per_gauss, // [C, N] or [nnz]
    at::optional<at::Tensor> isect_ids,       // [n_isects]
    at::optional<at::Tensor> flatten_ids      // [n_isects]
) {
    bool packed = means2d.dim() == 2;

    uint32_t N, nnz;
    int64_t n_elements;
    if (packed) {
        nnz = means2d.size(0); // total number of gaussians
        n_elements = nnz;
    } else {
        N = means2d.size(1); // number of gaussians per camera
        n_elements = C * N;
    }

    uint32_t n_tiles = tile_width * tile_height;
    // the number of bits needed to encode the camera id and tile id
    // Note: std::bit_width requires C++20
    // uint32_t tile_n_bits = std::bit_width(n_tiles);
    // uint32_t cam_n_bits = std::bit_width(C);
    uint32_t tile_n_bits = (uint32_t)floor(log2(n_tiles)) + 1;
    uint32_t cam_n_bits = (uint32_t)floor(log2(C)) + 1;
    // the first 32 bits are used for the camera id and tile id altogether, so
    // check if we have enough bits for them.
    assert(tile_n_bits + cam_n_bits <= 32);

    dim3 threads(256);
    dim3 grid((n_elements + threads.x - 1) / threads.x);
    int64_t shmem_size = 0; // No shared memory used in this kernel

    if (n_elements == 0) {
        // skip the kernel launch if there are no elements
        return;
    }

    AT_DISPATCH_FLOATING_TYPES(
        means2d.scalar_type(),
        "intersect_tile_kernel",
        [&]() {
            intersect_tile_kernel<scalar_t>
                <<<grid,
                   threads,
                   shmem_size,
                   at::cuda::getCurrentCUDAStream()>>>(
                    packed,
                    C,
                    N,
                    nnz,
                    camera_ids.has_value()
                        ? camera_ids.value().data_ptr<int64_t>()
                        : nullptr,
                    gaussian_ids.has_value()
                        ? gaussian_ids.value().data_ptr<int64_t>()
                        : nullptr,
                    means2d.data_ptr<scalar_t>(),
                    radii.data_ptr<int32_t>(),
                    depths.data_ptr<scalar_t>(),
                    cum_tiles_per_gauss.has_value()
                        ? cum_tiles_per_gauss.value().data_ptr<int64_t>()
                        : nullptr,
                    tile_size,
                    tile_width,
                    tile_height,
                    tile_n_bits,
                    tiles_per_gauss.has_value()
                        ? tiles_per_gauss.value().data_ptr<int32_t>()
                        : nullptr,
                    isect_ids.has_value()
                        ? isect_ids.value().data_ptr<int64_t>()
                        : nullptr,
                    flatten_ids.has_value()
                        ? flatten_ids.value().data_ptr<int32_t>()
                        : nullptr
                );
        }
    );
}

__global__ void intersect_offset_kernel(
    const uint32_t n_isects,
    const int64_t *__restrict__ isect_ids,
    const uint32_t C,
    const uint32_t n_tiles,
    const uint32_t tile_n_bits,
    int32_t *__restrict__ offsets // [C, n_tiles]
) {
    // e.g., ids: [1, 1, 1, 3, 3], n_tiles = 6
    // counts: [0, 3, 0, 2, 0, 0]
    // cumsum: [0, 3, 3, 5, 5, 5]
    // offsets: [0, 0, 3, 3, 5, 5]
    uint32_t idx = cg::this_grid().thread_rank();
    if (idx >= n_isects)
        return;

    int64_t isect_id_curr = isect_ids[idx] >> 32;
    int64_t cid_curr = isect_id_curr >> tile_n_bits;
    int64_t tid_curr = isect_id_curr & ((1 << tile_n_bits) - 1);
    int64_t id_curr = cid_curr * n_tiles + tid_curr;

    if (idx == 0) {
        // write out the offsets until the first valid tile (inclusive)
        for (uint32_t i = 0; i < id_curr + 1; ++i)
            offsets[i] = static_cast<int32_t>(idx);
    }
    if (idx == n_isects - 1) {
        // write out the rest of the offsets
        for (uint32_t i = id_curr + 1; i < C * n_tiles; ++i)
            offsets[i] = static_cast<int32_t>(n_isects);
    }

    if (idx > 0) {
        // visit the current and previous isect_id and check if the (cid,
        // tile_id) pair changes.
        int64_t isect_id_prev = isect_ids[idx - 1] >> 32; // shift out the depth
        if (isect_id_prev == isect_id_curr)
            return;

        // write out the offsets between the previous and current tiles
        int64_t cid_prev = isect_id_prev >> tile_n_bits;
        int64_t tid_prev = isect_id_prev & ((1 << tile_n_bits) - 1);
        int64_t id_prev = cid_prev * n_tiles + tid_prev;
        for (uint32_t i = id_prev + 1; i < id_curr + 1; ++i)
            offsets[i] = static_cast<int32_t>(idx);
    }
}

void launch_intersect_offset_kernel(
    // inputs
    const at::Tensor isect_ids, // [n_isects]
    const uint32_t C,
    const uint32_t tile_width,
    const uint32_t tile_height,
    // outputs
    at::Tensor offsets // [C, tile_height, tile_width]
) {
    int64_t n_elements = isect_ids.size(0); // total number of intersections
    dim3 threads(256);
    dim3 grid((n_elements + threads.x - 1) / threads.x);
    int64_t shmem_size = 0; // No shared memory used in this kernel

    if (n_elements == 0) {
        offsets.fill_(0);
        return;
    }

    uint32_t n_tiles = tile_width * tile_height;
    uint32_t tile_n_bits = (uint32_t)floor(log2(n_tiles)) + 1;
    intersect_offset_kernel<<<
        grid,
        threads,
        shmem_size,
        at::cuda::getCurrentCUDAStream()>>>(
        n_elements,
        isect_ids.data_ptr<int64_t>(),
        C,
        n_tiles,
        tile_n_bits,
        offsets.data_ptr<int32_t>()
    );
}

// https://nvidia.github.io/cccl/cub/api/structcub_1_1DeviceRadixSort.html
// DoubleBuffer reduce the auxiliary memory usage from O(N+P) to O(P)
void radix_sort_double_buffer(
    const int64_t n_isects,
    const uint32_t tile_n_bits,
    const uint32_t cam_n_bits,
    at::Tensor isect_ids,
    at::Tensor flatten_ids,
    at::Tensor isect_ids_sorted,
    at::Tensor flatten_ids_sorted
) {
    if (n_isects <= 0) {
        return;
    }

    // Create a set of DoubleBuffers to wrap pairs of device pointers
    hipcub::DoubleBuffer<int64_t> d_keys(
        isect_ids.data_ptr<int64_t>(), isect_ids_sorted.data_ptr<int64_t>()
    );
    hipcub::DoubleBuffer<int32_t> d_values(
        flatten_ids.data_ptr<int32_t>(), flatten_ids_sorted.data_ptr<int32_t>()
    );
    CUB_WRAPPER(
        hipcub::DeviceRadixSort::SortPairs,
        d_keys,
        d_values,
        n_isects,
        0,
        32 + tile_n_bits + cam_n_bits,
        at::cuda::getCurrentCUDAStream()
    );
    switch (d_keys.selector) {
    case 0: // sorted items are stored in isect_ids
        isect_ids_sorted.set_(isect_ids);
        break;
    case 1: // sorted items are stored in isect_ids_sorted
        break;
    }
    switch (d_values.selector) {
    case 0: // sorted items are stored in flatten_ids
        flatten_ids_sorted.set_(flatten_ids);
        break;
    case 1: // sorted items are stored in flatten_ids_sorted
        break;
    }

    // Double buffer is better than naive radix sort, in terms of mem usage.
    // CUB_WRAPPER(
    //     hipcub::DeviceRadixSort::SortPairs,
    //     isect_ids,
    //     isect_ids_sorted,
    //     flatten_ids,
    //     flatten_ids_sorted,
    //     n_isects,
    //     0,
    //     32 + tile_n_bits + cam_n_bits,
    //     stream
    // );
}

} // namespace gsplat
