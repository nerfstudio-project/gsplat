#include "hip/hip_runtime.h"
#include "bindings.h"
#include "helpers.cuh"
#include "utils.cuh"

#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/reduce.h>
#include <hipcub/hipcub.hpp>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

namespace gsplat {

namespace cg = cooperative_groups;

/****************************************************************************
 * Projection of Gaussians (Batched) Backward Pass 2DGS
 ****************************************************************************/

template <typename T>
__global__ void fully_fused_projection_packed_bwd_2dgs_kernel(
    // fwd inputs
    const uint32_t C,
    const uint32_t N,
    const uint32_t nnz,
    const T *__restrict__ means,    // [N, 3]
    const T *__restrict__ quats,    // [N, 4]
    const T *__restrict__ scales,   // [N, 3]
    const T *__restrict__ viewmats, // [C, 4, 4]
    const T *__restrict__ Ks,       // [C, 3, 3]
    const int32_t image_width,
    const int32_t image_height,
    // fwd outputs
    const int64_t *__restrict__ camera_ids,   // [nnz]
    const int64_t *__restrict__ gaussian_ids, // [nnz]
    const T *__restrict__ ray_transforms,             // [nnz, 3]
    // grad outputs
    const T *__restrict__ v_means2d, // [nnz, 2]
    const T *__restrict__ v_depths,  // [nnz]
    const T *__restrict__ v_normals, // [nnz, 3]
    const bool sparse_grad, // whether the outputs are in COO format [nnz, ...]
    // grad inputs
    T *__restrict__ v_ray_transforms,
    T *__restrict__ v_means,   // [N, 3] or [nnz, 3]
    T *__restrict__ v_quats,   // [N, 4] or [nnz, 4] Optional
    T *__restrict__ v_scales,  // [N, 3] or [nnz, 3] Optional
    T *__restrict__ v_viewmats // [C, 4, 4] Optional
) {
    // parallelize over nnz.
    uint32_t idx = cg::this_grid().thread_rank();
    if (idx >= nnz) {
        return;
    }
    const int64_t cid = camera_ids[idx];   // camera id
    const int64_t gid = gaussian_ids[idx]; // gaussian id

    // shift pointers to the current camera and gaussian
    means += gid * 3;
    viewmats += cid * 16;
    Ks += cid * 9;

    ray_transforms += idx * 9;

    v_means2d += idx * 2;
    v_normals += idx * 3;
    v_depths += idx;
    v_ray_transforms += idx * 9;

    // transform Gaussian to camera space
    mat3<T> R = mat3<T>(
        viewmats[0],
        viewmats[4],
        viewmats[8], // 1st column
        viewmats[1],
        viewmats[5],
        viewmats[9], // 2nd column
        viewmats[2],
        viewmats[6],
        viewmats[10] // 3rd column
    );
    vec3<T> t = vec3<T>(viewmats[3], viewmats[7], viewmats[11]);
    vec3<T> mean_c;
    pos_world_to_cam(R, t, glm::make_vec3(means), mean_c);

    vec4<T> quat = glm::make_vec4(quats + gid * 4);
    vec2<T> scale = glm::make_vec2(scales + gid * 3);
    mat3<T> P = mat3<T>(Ks[0], 0.0, Ks[2], 0.0, Ks[4], Ks[5], 0.0, 0.0, 1.0);

    mat3<T> _v_ray_transforms = mat3<T>(
        v_ray_transforms[0],
        v_ray_transforms[1],
        v_ray_transforms[2],
        v_ray_transforms[3],
        v_ray_transforms[4],
        v_ray_transforms[5],
        v_ray_transforms[6],
        v_ray_transforms[7],
        v_ray_transforms[8]
    );

    _v_ray_transforms[2][2] += v_depths[0];

    vec3<T> v_normal = glm::make_vec3(v_normals);

    vec3<T> v_mean(0.f);
    vec2<T> v_scale(0.f);
    vec4<T> v_quat(0.f);
    compute_ray_transforms_aabb_vjp<T>(
        ray_transforms,
        v_means2d,
        v_normal,
        R,
        P,
        t,
        mean_c,
        quat,
        scale,
        _v_ray_transforms,
        v_quat,
        v_scale,
        v_mean
    );

    auto warp = cg::tiled_partition<32>(cg::this_thread_block());
    if (sparse_grad) {
        // write out results with sparse layout
        if (v_means != nullptr) {
            v_means += idx * 3;
            GSPLAT_PRAGMA_UNROLL
            for (uint32_t i = 0; i < 3; i++) {
                v_means[i] = v_mean[i];
            }
        }
        v_quats += idx * 4;
        v_scales += idx * 3;
        v_quats[0] = v_quat[0];
        v_quats[1] = v_quat[1];
        v_quats[2] = v_quat[2];
        v_quats[3] = v_quat[3];
        v_scales[0] = v_scale[0];
        v_scales[1] = v_scale[1];
    } else {
        // write out results with dense layout
        // #if __CUDA_ARCH__ >= 700
        // write out results with warp-level reduction
        auto warp_group_g = cg::labeled_partition(warp, gid);
        if (v_means != nullptr) {
            warpSum(v_mean, warp_group_g);
            if (warp_group_g.thread_rank() == 0) {
                v_means += gid * 3;
                GSPLAT_PRAGMA_UNROLL
                for (uint32_t i = 0; i < 3; i++) {
                    gpuAtomicAdd(v_means + i, v_mean[i]);
                }
            }
        }
        // Directly output gradients w.r.t. the quaternion and scale
        warpSum(v_quat, warp_group_g);
        warpSum(v_scale, warp_group_g);
        if (warp_group_g.thread_rank() == 0) {
            v_quats += gid * 4;
            v_scales += gid * 3;
            gpuAtomicAdd(v_quats, v_quat[0]);
            gpuAtomicAdd(v_quats + 1, v_quat[1]);
            gpuAtomicAdd(v_quats + 2, v_quat[2]);
            gpuAtomicAdd(v_quats + 3, v_quat[3]);
            gpuAtomicAdd(v_scales, v_scale[0]);
            gpuAtomicAdd(v_scales + 1, v_scale[1]);
        }
    }
}

std::tuple<torch::Tensor, torch::Tensor, torch::Tensor, torch::Tensor>
fully_fused_projection_packed_bwd_2dgs_tensor(
    // fwd inputs
    const torch::Tensor &means,    // [N, 3]
    const torch::Tensor &quats,    // [N, 4]
    const torch::Tensor &scales,   // [N, 3]
    const torch::Tensor &viewmats, // [C, 4, 4]
    const torch::Tensor &Ks,       // [C, 3, 3]
    const uint32_t image_width,
    const uint32_t image_height,
    // fwd outputs
    const torch::Tensor &camera_ids,   // [nnz]
    const torch::Tensor &gaussian_ids, // [nnz]
    const torch::Tensor &ray_transforms,       // [nnz, 3, 3]
    // grad outputs
    const torch::Tensor &v_means2d, // [nnz, 2]
    const torch::Tensor &v_depths,  // [nnz]
    const torch::Tensor &v_ray_transforms,  // [nnz, 3, 3]
    const torch::Tensor &v_normals, // [nnz, 3]
    const bool viewmats_requires_grad,
    const bool sparse_grad
) {

    GSPLAT_DEVICE_GUARD(means);
    GSPLAT_CHECK_INPUT(means);
    GSPLAT_CHECK_INPUT(quats);
    GSPLAT_CHECK_INPUT(scales);
    GSPLAT_CHECK_INPUT(viewmats);
    GSPLAT_CHECK_INPUT(Ks);
    GSPLAT_CHECK_INPUT(camera_ids);
    GSPLAT_CHECK_INPUT(gaussian_ids);
    GSPLAT_CHECK_INPUT(ray_transforms);
    GSPLAT_CHECK_INPUT(v_means2d);
    GSPLAT_CHECK_INPUT(v_depths);
    GSPLAT_CHECK_INPUT(v_normals);
    GSPLAT_CHECK_INPUT(v_ray_transforms);

    uint32_t N = means.size(0);    // number of gaussians
    uint32_t C = viewmats.size(0); // number of cameras
    uint32_t nnz = camera_ids.size(0);

    at::cuda::CUDAStream stream = at::cuda::getCurrentCUDAStream();

    torch::Tensor v_means, v_quats, v_scales, v_viewmats;
    if (sparse_grad) {
        v_means = torch::zeros({nnz, 3}, means.options());

        v_quats = torch::zeros({nnz, 4}, quats.options());
        v_scales = torch::zeros({nnz, 3}, scales.options());

        if (viewmats_requires_grad) {
            v_viewmats = torch::zeros({C, 4, 4}, viewmats.options());
        }

    } else {
        v_means = torch::zeros_like(means);

        v_quats = torch::zeros_like(quats);
        v_scales = torch::zeros_like(scales);

        if (viewmats_requires_grad) {
            v_viewmats = torch::zeros_like(viewmats);
        }
    }
    if (nnz) {

        fully_fused_projection_packed_bwd_2dgs_kernel<float>
            <<<(nnz + GSPLAT_N_THREADS - 1) / GSPLAT_N_THREADS,
               GSPLAT_N_THREADS,
               0,
               stream>>>(
                C,
                N,
                nnz,
                means.data_ptr<float>(),
                quats.data_ptr<float>(),
                scales.data_ptr<float>(),
                viewmats.data_ptr<float>(),
                Ks.data_ptr<float>(),
                image_width,
                image_height,
                camera_ids.data_ptr<int64_t>(),
                gaussian_ids.data_ptr<int64_t>(),
                ray_transforms.data_ptr<float>(),
                v_means2d.data_ptr<float>(),
                v_depths.data_ptr<float>(),
                v_normals.data_ptr<float>(),
                sparse_grad,
                v_ray_transforms.data_ptr<float>(),
                v_means.data_ptr<float>(),
                v_quats.data_ptr<float>(),
                v_scales.data_ptr<float>(),
                viewmats_requires_grad ? v_viewmats.data_ptr<float>() : nullptr
            );
    }
    return std::make_tuple(v_means, v_quats, v_scales, v_viewmats);
}

} // namespace gsplat