#include "hip/hip_runtime.h"
#include <ATen/Dispatch.h>
#include <ATen/core/Tensor.h>
#include <ATen/cuda/Atomic.cuh>
#include <c10/cuda/CUDAStream.h>
#include <hip/hip_cooperative_groups.h>
#include <hipcub/hipcub.hpp>

#include "Common.h"
#include "Projection.h"
#include "Utils.cuh"

namespace gsplat {

namespace cg = cooperative_groups;

template <typename scalar_t>
__global__ void projection_ewa_3dgs_packed_fwd_kernel(
    const uint32_t C,
    const uint32_t N,
    const scalar_t *__restrict__ means,    // [N, 3]
    const scalar_t *__restrict__ covars,   // [N, 6] Optional
    const scalar_t *__restrict__ quats,    // [N, 4] Optional
    const scalar_t *__restrict__ scales,   // [N, 3] Optional
    const scalar_t *__restrict__ viewmats, // [C, 4, 4]
    const scalar_t *__restrict__ Ks,       // [C, 3, 3]
    const int32_t image_width,
    const int32_t image_height,
    const float eps2d,
    const float near_plane,
    const float far_plane,
    const float radius_clip,
    const int32_t
        *__restrict__ block_accum, // [C * blocks_per_row] packing helper
    const CameraModelType camera_model,
    // outputs
    int32_t *__restrict__ block_cnts,    // [C * blocks_per_row] packing helper
    int32_t *__restrict__ indptr,        // [C + 1]
    int64_t *__restrict__ camera_ids,    // [nnz]
    int64_t *__restrict__ gaussian_ids,  // [nnz]
    int32_t *__restrict__ radii,         // [nnz]
    scalar_t *__restrict__ means2d,      // [nnz, 2]
    scalar_t *__restrict__ depths,       // [nnz]
    scalar_t *__restrict__ conics,       // [nnz, 3]
    scalar_t *__restrict__ compensations // [nnz] optional
) {
    int32_t blocks_per_row = gridDim.x;

    int32_t row_idx = blockIdx.y; // cid
    int32_t block_col_idx = blockIdx.x;
    int32_t block_idx = row_idx * blocks_per_row + block_col_idx;

    int32_t col_idx = block_col_idx * blockDim.x + threadIdx.x; // gid

    bool valid = (row_idx < C) && (col_idx < N);

    // check if points are with camera near and far plane
    vec3 mean_c;
    mat3 R;
    if (valid) {
        // shift pointers to the current camera and gaussian
        means += col_idx * 3;
        viewmats += row_idx * 16;

        // glm is column-major but input is row-major
        R = mat3(
            viewmats[0],
            viewmats[4],
            viewmats[8], // 1st column
            viewmats[1],
            viewmats[5],
            viewmats[9], // 2nd column
            viewmats[2],
            viewmats[6],
            viewmats[10] // 3rd column
        );
        vec3 t = vec3(viewmats[3], viewmats[7], viewmats[11]);

        // transform Gaussian center to camera space
        posW2C(R, t, glm::make_vec3(means), mean_c);
        if (mean_c.z < near_plane || mean_c.z > far_plane) {
            valid = false;
        }
    }

    // check if the perspective projection is valid.
    mat2 covar2d;
    vec2 mean2d;
    mat2 covar2d_inv;
    float compensation;
    float det;
    if (valid) {
        // transform Gaussian covariance to camera space
        mat3 covar;
        if (covars != nullptr) {
            // if a precomputed covariance is provided
            covars += col_idx * 6;
            covar = mat3(
                covars[0],
                covars[1],
                covars[2], // 1st column
                covars[1],
                covars[3],
                covars[4], // 2nd column
                covars[2],
                covars[4],
                covars[5] // 3rd column
            );
        } else {
            // if not then compute it from quaternions and scales
            quats += col_idx * 4;
            scales += col_idx * 3;
            quat_scale_to_covar_preci(
                glm::make_vec4(quats), glm::make_vec3(scales), &covar, nullptr
            );
        }
        mat3 covar_c;
        covarW2C(R, covar, covar_c);

        Ks += row_idx * 9;
        switch (camera_model) {
        case CameraModelType::PINHOLE: // perspective projection
            persp_proj(
                mean_c,
                covar_c,
                Ks[0],
                Ks[4],
                Ks[2],
                Ks[5],
                image_width,
                image_height,
                covar2d,
                mean2d
            );
            break;
        case CameraModelType::ORTHO: // orthographic projection
            ortho_proj(
                mean_c,
                covar_c,
                Ks[0],
                Ks[4],
                Ks[2],
                Ks[5],
                image_width,
                image_height,
                covar2d,
                mean2d
            );
            break;
        case CameraModelType::FISHEYE: // fisheye projection
            fisheye_proj(
                mean_c,
                covar_c,
                Ks[0],
                Ks[4],
                Ks[2],
                Ks[5],
                image_width,
                image_height,
                covar2d,
                mean2d
            );
            break;
        }

        det = add_blur(eps2d, covar2d, compensation);
        if (det <= 0.f) {
            valid = false;
        } else {
            // compute the inverse of the 2d covariance
            covar2d_inv = glm::inverse(covar2d);
        }
    }

    // check if the points are in the image region
    float radius;
    if (valid) {
        // take 3 sigma as the radius (non differentiable)
        float b = 0.5f * (covar2d[0][0] + covar2d[1][1]);
        float v1 = b + sqrt(max(0.1f, b * b - det));
        float v2 = b - sqrt(max(0.1f, b * b - det));
        radius = ceil(3.f * sqrt(max(v1, v2)));

        if (radius <= radius_clip) {
            valid = false;
        }

        // mask out gaussians outside the image region
        if (mean2d.x + radius <= 0 || mean2d.x - radius >= image_width ||
            mean2d.y + radius <= 0 || mean2d.y - radius >= image_height) {
            valid = false;
        }
    }

    int32_t thread_data = static_cast<int32_t>(valid);
    if (block_cnts != nullptr) {
        // First pass: compute the block-wide sum
        int32_t aggregate;
        if (__syncthreads_or(thread_data)) {
            typedef hipcub::BlockReduce<int32_t, N_THREADS_PACKED> BlockReduce;
            __shared__ typename BlockReduce::TempStorage temp_storage;
            aggregate = BlockReduce(temp_storage).Sum(thread_data);
        } else {
            aggregate = 0;
        }
        if (threadIdx.x == 0) {
            block_cnts[block_idx] = aggregate;
        }
    } else {
        // Second pass: write out the indices of the non zero elements
        if (__syncthreads_or(thread_data)) {
            typedef hipcub::BlockScan<int32_t, N_THREADS_PACKED> BlockScan;
            __shared__ typename BlockScan::TempStorage temp_storage;
            BlockScan(temp_storage).ExclusiveSum(thread_data, thread_data);
        }
        if (valid) {
            if (block_idx > 0) {
                int32_t offset = block_accum[block_idx - 1];
                thread_data += offset;
            }
            // write to outputs
            camera_ids[thread_data] = row_idx;   // cid
            gaussian_ids[thread_data] = col_idx; // gid
            radii[thread_data] = (int32_t)radius;
            means2d[thread_data * 2] = mean2d.x;
            means2d[thread_data * 2 + 1] = mean2d.y;
            depths[thread_data] = mean_c.z;
            conics[thread_data * 3] = covar2d_inv[0][0];
            conics[thread_data * 3 + 1] = covar2d_inv[0][1];
            conics[thread_data * 3 + 2] = covar2d_inv[1][1];
            if (compensations != nullptr) {
                compensations[thread_data] = compensation;
            }
        }
        // lane 0 of the first block in each row writes the indptr
        if (threadIdx.x == 0 && block_col_idx == 0) {
            if (row_idx == 0) {
                indptr[0] = 0;
                indptr[C] = block_accum[C * blocks_per_row - 1];
            } else {
                indptr[row_idx] = block_accum[block_idx - 1];
            }
        }
    }
}

void launch_projection_ewa_3dgs_packed_fwd_kernel(
    // inputs
    const at::Tensor means,                // [N, 3]
    const at::optional<at::Tensor> covars, // [N, 6] optional
    const at::optional<at::Tensor> quats,  // [N, 4] optional
    const at::optional<at::Tensor> scales, // [N, 3] optional
    const at::Tensor viewmats,             // [C, 4, 4]
    const at::Tensor Ks,                   // [C, 3, 3]
    const uint32_t image_width,
    const uint32_t image_height,
    const float eps2d,
    const float near_plane,
    const float far_plane,
    const float radius_clip,
    const at::optional<at::Tensor>
        block_accum, // [C * blocks_per_row] packing helper
    const CameraModelType camera_model,
    // outputs
    at::optional<at::Tensor> block_cnts, // [C * blocks_per_row] packing helper
    at::optional<at::Tensor> indptr,     // [C + 1]
    at::optional<at::Tensor> camera_ids, // [nnz]
    at::optional<at::Tensor> gaussian_ids, // [nnz]
    at::optional<at::Tensor> radii,        // [nnz]
    at::optional<at::Tensor> means2d,      // [nnz, 2]
    at::optional<at::Tensor> depths,       // [nnz]
    at::optional<at::Tensor> conics,       // [nnz, 3]
    at::optional<at::Tensor> compensations // [nnz] optional
) {
    uint32_t N = means.size(0);    // number of gaussians
    uint32_t C = viewmats.size(0); // number of cameras

    uint32_t nrows = C;
    uint32_t ncols = N;
    uint32_t blocks_per_row = (ncols + N_THREADS_PACKED - 1) / N_THREADS_PACKED;

    dim3 threads(N_THREADS_PACKED);
    // limit on the number of blocks: [2**31 - 1, 65535, 65535]
    dim3 grid(blocks_per_row, nrows, 1);
    int64_t shmem_size = 0; // No shared memory used in this kernel

    if (N == 0 || C == 0) {
        // skip the kernel launch if there are no elements
        return;
    }

    AT_DISPATCH_FLOATING_TYPES(
        means.scalar_type(),
        "projection_ewa_3dgs_packed_fwd_kernel",
        [&]() {
            projection_ewa_3dgs_packed_fwd_kernel<scalar_t>
                <<<grid,
                   threads,
                   shmem_size,
                   at::cuda::getCurrentCUDAStream()>>>(
                    C,
                    N,
                    means.data_ptr<scalar_t>(),
                    covars.has_value() ? covars.value().data_ptr<scalar_t>()
                                       : nullptr,
                    quats.has_value() ? quats.value().data_ptr<scalar_t>()
                                      : nullptr,
                    scales.has_value() ? scales.value().data_ptr<scalar_t>()
                                       : nullptr,
                    viewmats.data_ptr<scalar_t>(),
                    Ks.data_ptr<scalar_t>(),
                    image_width,
                    image_height,
                    eps2d,
                    near_plane,
                    far_plane,
                    radius_clip,
                    block_accum.has_value()
                        ? block_accum.value().data_ptr<int32_t>()
                        : nullptr,
                    camera_model,
                    block_cnts.has_value()
                        ? block_cnts.value().data_ptr<int32_t>()
                        : nullptr,
                    indptr.has_value() ? indptr.value().data_ptr<int32_t>()
                                       : nullptr,
                    camera_ids.has_value()
                        ? camera_ids.value().data_ptr<int64_t>()
                        : nullptr,
                    gaussian_ids.has_value()
                        ? gaussian_ids.value().data_ptr<int64_t>()
                        : nullptr,
                    radii.has_value() ? radii.value().data_ptr<int32_t>()
                                      : nullptr,
                    means2d.has_value() ? means2d.value().data_ptr<scalar_t>()
                                        : nullptr,
                    depths.has_value() ? depths.value().data_ptr<scalar_t>()
                                       : nullptr,
                    conics.has_value() ? conics.value().data_ptr<scalar_t>()
                                       : nullptr,
                    compensations.has_value()
                        ? compensations.value().data_ptr<scalar_t>()
                        : nullptr
                );
        }
    );
}

template <typename scalar_t>
__global__ void projection_ewa_3dgs_packed_bwd_kernel(
    // fwd inputs
    const uint32_t C,
    const uint32_t N,
    const uint32_t nnz,
    const scalar_t *__restrict__ means,    // [N, 3]
    const scalar_t *__restrict__ covars,   // [N, 6] Optional
    const scalar_t *__restrict__ quats,    // [N, 4] Optional
    const scalar_t *__restrict__ scales,   // [N, 3] Optional
    const scalar_t *__restrict__ viewmats, // [C, 4, 4]
    const scalar_t *__restrict__ Ks,       // [C, 3, 3]
    const int32_t image_width,
    const int32_t image_height,
    const scalar_t eps2d,
    const CameraModelType camera_model,
    // fwd outputs
    const int64_t *__restrict__ camera_ids,     // [nnz]
    const int64_t *__restrict__ gaussian_ids,   // [nnz]
    const scalar_t *__restrict__ conics,        // [nnz, 3]
    const scalar_t *__restrict__ compensations, // [nnz] optional
    // grad outputs
    const scalar_t *__restrict__ v_means2d,       // [nnz, 2]
    const scalar_t *__restrict__ v_depths,        // [nnz]
    const scalar_t *__restrict__ v_conics,        // [nnz, 3]
    const scalar_t *__restrict__ v_compensations, // [nnz] optional
    const bool sparse_grad, // whether the outputs are in COO format [nnz, ...]
    // grad inputs
    scalar_t *__restrict__ v_means,   // [N, 3] or [nnz, 3]
    scalar_t *__restrict__ v_covars,  // [N, 6] or [nnz, 6] Optional
    scalar_t *__restrict__ v_quats,   // [N, 4] or [nnz, 4] Optional
    scalar_t *__restrict__ v_scales,  // [N, 3] or [nnz, 3] Optional
    scalar_t *__restrict__ v_viewmats // [C, 4, 4] Optional
) {
    // parallelize over nnz.
    uint32_t idx = cg::this_grid().thread_rank();
    if (idx >= nnz) {
        return;
    }
    const int64_t cid = camera_ids[idx];   // camera id
    const int64_t gid = gaussian_ids[idx]; // gaussian id

    // shift pointers to the current camera and gaussian
    means += gid * 3;
    viewmats += cid * 16;
    Ks += cid * 9;

    conics += idx * 3;

    v_means2d += idx * 2;
    v_depths += idx;
    v_conics += idx * 3;

    // vjp: compute the inverse of the 2d covariance
    mat2 covar2d_inv = mat2(conics[0], conics[1], conics[1], conics[2]);
    mat2 v_covar2d_inv =
        mat2(v_conics[0], v_conics[1] * .5f, v_conics[1] * .5f, v_conics[2]);
    mat2 v_covar2d(0.f);
    inverse_vjp(covar2d_inv, v_covar2d_inv, v_covar2d);

    if (v_compensations != nullptr) {
        // vjp: compensation term
        const float compensation = compensations[idx];
        const float v_compensation = v_compensations[idx];
        add_blur_vjp(
            eps2d, covar2d_inv, compensation, v_compensation, v_covar2d
        );
    }

    // transform Gaussian to camera space
    mat3 R = mat3(
        viewmats[0],
        viewmats[4],
        viewmats[8], // 1st column
        viewmats[1],
        viewmats[5],
        viewmats[9], // 2nd column
        viewmats[2],
        viewmats[6],
        viewmats[10] // 3rd column
    );
    vec3 t = vec3(viewmats[3], viewmats[7], viewmats[11]);
    mat3 covar;
    vec4 quat;
    vec3 scale;
    if (covars != nullptr) {
        // if a precomputed covariance is provided
        covars += gid * 6;
        covar = mat3(
            covars[0],
            covars[1],
            covars[2], // 1st column
            covars[1],
            covars[3],
            covars[4], // 2nd column
            covars[2],
            covars[4],
            covars[5] // 3rd column
        );
    } else {
        // if not then compute it from quaternions and scales
        quat = glm::make_vec4(quats + gid * 4);
        scale = glm::make_vec3(scales + gid * 3);
        quat_scale_to_covar_preci(quat, scale, &covar, nullptr);
    }
    vec3 mean_c;
    posW2C(R, t, glm::make_vec3(means), mean_c);
    mat3 covar_c;
    covarW2C(R, covar, covar_c);

    float fx = Ks[0], cx = Ks[2], fy = Ks[4], cy = Ks[5];
    mat3 v_covar_c(0.f);
    vec3 v_mean_c(0.f);
    switch (camera_model) {
    case CameraModelType::PINHOLE: // perspective projection
        persp_proj_vjp(
            mean_c,
            covar_c,
            fx,
            fy,
            cx,
            cy,
            image_width,
            image_height,
            v_covar2d,
            glm::make_vec2(v_means2d),
            v_mean_c,
            v_covar_c
        );
        break;
    case CameraModelType::ORTHO: // orthographic projection
        ortho_proj_vjp(
            mean_c,
            covar_c,
            fx,
            fy,
            cx,
            cy,
            image_width,
            image_height,
            v_covar2d,
            glm::make_vec2(v_means2d),
            v_mean_c,
            v_covar_c
        );
        break;
    case CameraModelType::FISHEYE: // fisheye projection
        fisheye_proj_vjp(
            mean_c,
            covar_c,
            fx,
            fy,
            cx,
            cy,
            image_width,
            image_height,
            v_covar2d,
            glm::make_vec2(v_means2d),
            v_mean_c,
            v_covar_c
        );
        break;
    }

    // add contribution from v_depths
    v_mean_c.z += v_depths[0];

    // vjp: transform Gaussian covariance to camera space
    vec3 v_mean(0.f);
    mat3 v_covar(0.f);
    mat3 v_R(0.f);
    vec3 v_t(0.f);
    posW2C_VJP(R, t, glm::make_vec3(means), v_mean_c, v_R, v_t, v_mean);
    covarW2C_VJP(R, covar, v_covar_c, v_R, v_covar);

    auto warp = cg::tiled_partition<32>(cg::this_thread_block());
    if (sparse_grad) {
        // write out results with sparse layout
        if (v_means != nullptr) {
            v_means += idx * 3;
#pragma unroll
            for (uint32_t i = 0; i < 3; i++) {
                v_means[i] = v_mean[i];
            }
        }
        if (v_covars != nullptr) {
            v_covars += idx * 6;
            v_covars[0] = v_covar[0][0];
            v_covars[1] = v_covar[0][1] + v_covar[1][0];
            v_covars[2] = v_covar[0][2] + v_covar[2][0];
            v_covars[3] = v_covar[1][1];
            v_covars[4] = v_covar[1][2] + v_covar[2][1];
            v_covars[5] = v_covar[2][2];
        } else {
            mat3 rotmat = quat_to_rotmat(quat);
            vec4 v_quat(0.f);
            vec3 v_scale(0.f);
            quat_scale_to_covar_vjp(
                quat, scale, rotmat, v_covar, v_quat, v_scale
            );
            v_quats += idx * 4;
            v_scales += idx * 3;
            v_quats[0] = v_quat[0];
            v_quats[1] = v_quat[1];
            v_quats[2] = v_quat[2];
            v_quats[3] = v_quat[3];
            v_scales[0] = v_scale[0];
            v_scales[1] = v_scale[1];
            v_scales[2] = v_scale[2];
        }
    } else {
        // write out results with dense layout
        // #if __CUDA_ARCH__ >= 700
        // write out results with warp-level reduction
        auto warp_group_g = cg::labeled_partition(warp, gid);
        if (v_means != nullptr) {
            warpSum(v_mean, warp_group_g);
            if (warp_group_g.thread_rank() == 0) {
                v_means += gid * 3;
#pragma unroll
                for (uint32_t i = 0; i < 3; i++) {
                    gpuAtomicAdd(v_means + i, v_mean[i]);
                }
            }
        }
        if (v_covars != nullptr) {
            // Directly output gradients w.r.t. the covariance
            warpSum(v_covar, warp_group_g);
            if (warp_group_g.thread_rank() == 0) {
                v_covars += gid * 6;
                gpuAtomicAdd(v_covars, v_covar[0][0]);
                gpuAtomicAdd(v_covars + 1, v_covar[0][1] + v_covar[1][0]);
                gpuAtomicAdd(v_covars + 2, v_covar[0][2] + v_covar[2][0]);
                gpuAtomicAdd(v_covars + 3, v_covar[1][1]);
                gpuAtomicAdd(v_covars + 4, v_covar[1][2] + v_covar[2][1]);
                gpuAtomicAdd(v_covars + 5, v_covar[2][2]);
            }
        } else {
            // Directly output gradients w.r.t. the quaternion and scale
            mat3 rotmat = quat_to_rotmat(quat);
            vec4 v_quat(0.f);
            vec3 v_scale(0.f);
            quat_scale_to_covar_vjp(
                quat, scale, rotmat, v_covar, v_quat, v_scale
            );
            warpSum(v_quat, warp_group_g);
            warpSum(v_scale, warp_group_g);
            if (warp_group_g.thread_rank() == 0) {
                v_quats += gid * 4;
                v_scales += gid * 3;
                gpuAtomicAdd(v_quats, v_quat[0]);
                gpuAtomicAdd(v_quats + 1, v_quat[1]);
                gpuAtomicAdd(v_quats + 2, v_quat[2]);
                gpuAtomicAdd(v_quats + 3, v_quat[3]);
                gpuAtomicAdd(v_scales, v_scale[0]);
                gpuAtomicAdd(v_scales + 1, v_scale[1]);
                gpuAtomicAdd(v_scales + 2, v_scale[2]);
            }
        }
    }
    // v_viewmats is always in dense layout
    if (v_viewmats != nullptr) {
        auto warp_group_c = cg::labeled_partition(warp, cid);
        warpSum(v_R, warp_group_c);
        warpSum(v_t, warp_group_c);
        if (warp_group_c.thread_rank() == 0) {
            v_viewmats += cid * 16;
#pragma unroll
            for (uint32_t i = 0; i < 3; i++) { // rows
#pragma unroll
                for (uint32_t j = 0; j < 3; j++) { // cols
                    gpuAtomicAdd(v_viewmats + i * 4 + j, v_R[j][i]);
                }
                gpuAtomicAdd(v_viewmats + i * 4 + 3, v_t[i]);
            }
        }
    }
}

void launch_projection_ewa_3dgs_packed_bwd_kernel(
    // fwd inputs
    const at::Tensor means,                // [N, 3]
    const at::optional<at::Tensor> covars, // [N, 6]
    const at::optional<at::Tensor> quats,  // [N, 4]
    const at::optional<at::Tensor> scales, // [N, 3]
    const at::Tensor viewmats,             // [C, 4, 4]
    const at::Tensor Ks,                   // [C, 3, 3]
    const uint32_t image_width,
    const uint32_t image_height,
    const float eps2d,
    const CameraModelType camera_model,
    // fwd outputs
    const at::Tensor camera_ids,                  // [nnz]
    const at::Tensor gaussian_ids,                // [nnz]
    const at::Tensor conics,                      // [nnz, 3]
    const at::optional<at::Tensor> compensations, // [nnz] optional
    // grad outputs
    const at::Tensor v_means2d,                     // [nnz, 2]
    const at::Tensor v_depths,                      // [nnz]
    const at::Tensor v_conics,                      // [nnz, 3]
    const at::optional<at::Tensor> v_compensations, // [nnz] optional
    const bool sparse_grad,
    // grad inputs
    at::Tensor v_means,                 // [N, 3] or [nnz, 3]
    at::optional<at::Tensor> v_covars,  // [N, 6] or [nnz, 6] Optional
    at::optional<at::Tensor> v_quats,   // [N, 4] or [nnz, 4] Optional
    at::optional<at::Tensor> v_scales,  // [N, 3] or [nnz, 3] Optional
    at::optional<at::Tensor> v_viewmats // [C, 4, 4] Optional
) {
    uint32_t N = means.size(0);    // number of gaussians
    uint32_t C = viewmats.size(0); // number of cameras
    uint32_t nnz = camera_ids.size(0);

    dim3 threads(256);
    dim3 grid((nnz + threads.x - 1) / threads.x);
    int64_t shmem_size = 0; // No shared memory used in this kernel

    if (nnz == 0) {
        // skip the kernel launch if there are no elements
        return;
    }

    AT_DISPATCH_FLOATING_TYPES(
        means.scalar_type(),
        "projection_ewa_3dgs_packed_bwd_kernel",
        [&]() {
            projection_ewa_3dgs_packed_bwd_kernel<scalar_t>
                <<<grid,
                   threads,
                   shmem_size,
                   at::cuda::getCurrentCUDAStream()>>>(
                    C,
                    N,
                    nnz,
                    means.data_ptr<scalar_t>(),
                    covars.has_value() ? covars.value().data_ptr<scalar_t>()
                                       : nullptr,
                    covars.has_value() ? nullptr
                                       : quats.value().data_ptr<scalar_t>(),
                    covars.has_value() ? nullptr
                                       : scales.value().data_ptr<scalar_t>(),
                    viewmats.data_ptr<scalar_t>(),
                    Ks.data_ptr<scalar_t>(),
                    image_width,
                    image_height,
                    eps2d,
                    camera_model,
                    camera_ids.data_ptr<int64_t>(),
                    gaussian_ids.data_ptr<int64_t>(),
                    conics.data_ptr<scalar_t>(),
                    compensations.has_value()
                        ? compensations.value().data_ptr<scalar_t>()
                        : nullptr,
                    v_means2d.data_ptr<scalar_t>(),
                    v_depths.data_ptr<scalar_t>(),
                    v_conics.data_ptr<scalar_t>(),
                    v_compensations.has_value()
                        ? v_compensations.value().data_ptr<scalar_t>()
                        : nullptr,
                    sparse_grad,
                    v_means.data_ptr<scalar_t>(),
                    covars.has_value() ? v_covars.value().data_ptr<scalar_t>()
                                       : nullptr,
                    covars.has_value() ? nullptr
                                       : v_quats.value().data_ptr<scalar_t>(),
                    covars.has_value() ? nullptr
                                       : v_scales.value().data_ptr<scalar_t>(),
                    v_viewmats.has_value()
                        ? v_viewmats.value().data_ptr<scalar_t>()
                        : nullptr
                );
        }
    );
}

} // namespace gsplat