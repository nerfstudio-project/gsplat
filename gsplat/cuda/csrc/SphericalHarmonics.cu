#include "hip/hip_runtime.h"
#include <ATen/Dispatch.h>
#include <ATen/core/Tensor.h>
#include <ATen/cuda/Atomic.cuh>
#include <c10/cuda/CUDAStream.h>
#include <hip/hip_cooperative_groups.h>

#include "Common.h"
#include "SphericalHarmonics.h"
#include "Utils.cuh"

namespace gsplat {

namespace cg = cooperative_groups;

// Evaluate spherical harmonics bases at unit direction for high orders using
// approach described by Efficient Spherical Harmonic Evaluation, Peter-Pike
// Sloan, JCGT 2013 See https://jcgt.org/published/0002/02/06/ for reference
// implementation

template <typename scalar_t>
__device__ void sh_coeffs_to_color_fast(
    const uint32_t degree,  // degree of SH to be evaluated
    const uint32_t c,       // color channel
    const vec3 &dir,        // [3]
    const scalar_t *coeffs, // [K, 3]
    // output
    scalar_t *colors // [3]
) {
    float result = 0.2820947917738781f * coeffs[c];
    if (degree >= 1) {
        // Normally rsqrt is faster than sqrt, but --use_fast_math will optimize
        // sqrt on single precision, so we use sqrt here.
        float inorm = rsqrtf(dir.x * dir.x + dir.y * dir.y + dir.z * dir.z);
        float x = dir.x * inorm;
        float y = dir.y * inorm;
        float z = dir.z * inorm;

        result +=
            0.48860251190292f * (-y * coeffs[1 * 3 + c] +
                                 z * coeffs[2 * 3 + c] - x * coeffs[3 * 3 + c]);
        if (degree >= 2) {
            float z2 = z * z;

            float fTmp0B = -1.092548430592079f * z;
            float fC1 = x * x - y * y;
            float fS1 = 2.f * x * y;
            float pSH6 = (0.9461746957575601f * z2 - 0.3153915652525201f);
            float pSH7 = fTmp0B * x;
            float pSH5 = fTmp0B * y;
            float pSH8 = 0.5462742152960395f * fC1;
            float pSH4 = 0.5462742152960395f * fS1;

            result += pSH4 * coeffs[4 * 3 + c] + pSH5 * coeffs[5 * 3 + c] +
                      pSH6 * coeffs[6 * 3 + c] + pSH7 * coeffs[7 * 3 + c] +
                      pSH8 * coeffs[8 * 3 + c];
            if (degree >= 3) {
                float fTmp0C = -2.285228997322329f * z2 + 0.4570457994644658f;
                float fTmp1B = 1.445305721320277f * z;
                float fC2 = x * fC1 - y * fS1;
                float fS2 = x * fS1 + y * fC1;
                float pSH12 =
                    z * (1.865881662950577f * z2 - 1.119528997770346f);
                float pSH13 = fTmp0C * x;
                float pSH11 = fTmp0C * y;
                float pSH14 = fTmp1B * fC1;
                float pSH10 = fTmp1B * fS1;
                float pSH15 = -0.5900435899266435f * fC2;
                float pSH9 = -0.5900435899266435f * fS2;

                result +=
                    pSH9 * coeffs[9 * 3 + c] + pSH10 * coeffs[10 * 3 + c] +
                    pSH11 * coeffs[11 * 3 + c] + pSH12 * coeffs[12 * 3 + c] +
                    pSH13 * coeffs[13 * 3 + c] + pSH14 * coeffs[14 * 3 + c] +
                    pSH15 * coeffs[15 * 3 + c];

                if (degree >= 4) {
                    float fTmp0D =
                        z * (-4.683325804901025f * z2 + 2.007139630671868f);
                    float fTmp1C = 3.31161143515146f * z2 - 0.47308734787878f;
                    float fTmp2B = -1.770130769779931f * z;
                    float fC3 = x * fC2 - y * fS2;
                    float fS3 = x * fS2 + y * fC2;
                    float pSH20 =
                        (1.984313483298443f * z * pSH12 -
                         1.006230589874905f * pSH6);
                    float pSH21 = fTmp0D * x;
                    float pSH19 = fTmp0D * y;
                    float pSH22 = fTmp1C * fC1;
                    float pSH18 = fTmp1C * fS1;
                    float pSH23 = fTmp2B * fC2;
                    float pSH17 = fTmp2B * fS2;
                    float pSH24 = 0.6258357354491763f * fC3;
                    float pSH16 = 0.6258357354491763f * fS3;

                    result += pSH16 * coeffs[16 * 3 + c] +
                              pSH17 * coeffs[17 * 3 + c] +
                              pSH18 * coeffs[18 * 3 + c] +
                              pSH19 * coeffs[19 * 3 + c] +
                              pSH20 * coeffs[20 * 3 + c] +
                              pSH21 * coeffs[21 * 3 + c] +
                              pSH22 * coeffs[22 * 3 + c] +
                              pSH23 * coeffs[23 * 3 + c] +
                              pSH24 * coeffs[24 * 3 + c];
                }
            }
        }
    }

    colors[c] = result;
}

template <typename scalar_t>
__device__ void sh_coeffs_to_color_fast_vjp(
    const uint32_t degree,    // degree of SH to be evaluated
    const uint32_t c,         // color channel
    const vec3 &dir,          // [3]
    const scalar_t *coeffs,   // [K, 3]
    const scalar_t *v_colors, // [3]
    // output
    scalar_t *v_coeffs, // [K, 3]
    vec3 *v_dir         // [3] optional
) {
    float v_colors_local = v_colors[c];

    v_coeffs[c] = 0.2820947917738781f * v_colors_local;
    if (degree < 1) {
        return;
    }
    float inorm = rsqrtf(dir.x * dir.x + dir.y * dir.y + dir.z * dir.z);
    float x = dir.x * inorm;
    float y = dir.y * inorm;
    float z = dir.z * inorm;
    float v_x = 0.f, v_y = 0.f, v_z = 0.f;

    v_coeffs[1 * 3 + c] = -0.48860251190292f * y * v_colors_local;
    v_coeffs[2 * 3 + c] = 0.48860251190292f * z * v_colors_local;
    v_coeffs[3 * 3 + c] = -0.48860251190292f * x * v_colors_local;

    if (v_dir != nullptr) {
        v_x += -0.48860251190292f * coeffs[3 * 3 + c] * v_colors_local;
        v_y += -0.48860251190292f * coeffs[1 * 3 + c] * v_colors_local;
        v_z += 0.48860251190292f * coeffs[2 * 3 + c] * v_colors_local;
    }
    if (degree < 2) {
        if (v_dir != nullptr) {
            vec3 dir_n = vec3(x, y, z);
            vec3 v_dir_n = vec3(v_x, v_y, v_z);
            vec3 v_d = (v_dir_n - glm::dot(v_dir_n, dir_n) * dir_n) * inorm;

            v_dir->x = v_d.x;
            v_dir->y = v_d.y;
            v_dir->z = v_d.z;
        }
        return;
    }

    float z2 = z * z;
    float fTmp0B = -1.092548430592079f * z;
    float fC1 = x * x - y * y;
    float fS1 = 2.f * x * y;
    float pSH6 = (0.9461746957575601f * z2 - 0.3153915652525201f);
    float pSH7 = fTmp0B * x;
    float pSH5 = fTmp0B * y;
    float pSH8 = 0.5462742152960395f * fC1;
    float pSH4 = 0.5462742152960395f * fS1;
    v_coeffs[4 * 3 + c] = pSH4 * v_colors_local;
    v_coeffs[5 * 3 + c] = pSH5 * v_colors_local;
    v_coeffs[6 * 3 + c] = pSH6 * v_colors_local;
    v_coeffs[7 * 3 + c] = pSH7 * v_colors_local;
    v_coeffs[8 * 3 + c] = pSH8 * v_colors_local;

    float fTmp0B_z, fC1_x, fC1_y, fS1_x, fS1_y, pSH6_z, pSH7_x, pSH7_z, pSH5_y,
        pSH5_z, pSH8_x, pSH8_y, pSH4_x, pSH4_y;
    if (v_dir != nullptr) {
        fTmp0B_z = -1.092548430592079f;
        fC1_x = 2.f * x;
        fC1_y = -2.f * y;
        fS1_x = 2.f * y;
        fS1_y = 2.f * x;
        pSH6_z = 2.f * 0.9461746957575601f * z;
        pSH7_x = fTmp0B;
        pSH7_z = fTmp0B_z * x;
        pSH5_y = fTmp0B;
        pSH5_z = fTmp0B_z * y;
        pSH8_x = 0.5462742152960395f * fC1_x;
        pSH8_y = 0.5462742152960395f * fC1_y;
        pSH4_x = 0.5462742152960395f * fS1_x;
        pSH4_y = 0.5462742152960395f * fS1_y;

        v_x += v_colors_local *
               (pSH4_x * coeffs[4 * 3 + c] + pSH8_x * coeffs[8 * 3 + c] +
                pSH7_x * coeffs[7 * 3 + c]);
        v_y += v_colors_local *
               (pSH4_y * coeffs[4 * 3 + c] + pSH8_y * coeffs[8 * 3 + c] +
                pSH5_y * coeffs[5 * 3 + c]);
        v_z += v_colors_local *
               (pSH6_z * coeffs[6 * 3 + c] + pSH7_z * coeffs[7 * 3 + c] +
                pSH5_z * coeffs[5 * 3 + c]);
    }

    if (degree < 3) {
        if (v_dir != nullptr) {
            vec3 dir_n = vec3(x, y, z);
            vec3 v_dir_n = vec3(v_x, v_y, v_z);
            vec3 v_d = (v_dir_n - glm::dot(v_dir_n, dir_n) * dir_n) * inorm;

            v_dir->x = v_d.x;
            v_dir->y = v_d.y;
            v_dir->z = v_d.z;
        }
        return;
    }

    float fTmp0C = -2.285228997322329f * z2 + 0.4570457994644658f;
    float fTmp1B = 1.445305721320277f * z;
    float fC2 = x * fC1 - y * fS1;
    float fS2 = x * fS1 + y * fC1;
    float pSH12 = z * (1.865881662950577f * z2 - 1.119528997770346f);
    float pSH13 = fTmp0C * x;
    float pSH11 = fTmp0C * y;
    float pSH14 = fTmp1B * fC1;
    float pSH10 = fTmp1B * fS1;
    float pSH15 = -0.5900435899266435f * fC2;
    float pSH9 = -0.5900435899266435f * fS2;
    v_coeffs[9 * 3 + c] = pSH9 * v_colors_local;
    v_coeffs[10 * 3 + c] = pSH10 * v_colors_local;
    v_coeffs[11 * 3 + c] = pSH11 * v_colors_local;
    v_coeffs[12 * 3 + c] = pSH12 * v_colors_local;
    v_coeffs[13 * 3 + c] = pSH13 * v_colors_local;
    v_coeffs[14 * 3 + c] = pSH14 * v_colors_local;
    v_coeffs[15 * 3 + c] = pSH15 * v_colors_local;

    float fTmp0C_z, fTmp1B_z, fC2_x, fC2_y, fS2_x, fS2_y, pSH12_z, pSH13_x,
        pSH13_z, pSH11_y, pSH11_z, pSH14_x, pSH14_y, pSH14_z, pSH10_x, pSH10_y,
        pSH10_z, pSH15_x, pSH15_y, pSH9_x, pSH9_y;
    if (v_dir != nullptr) {
        fTmp0C_z = -2.285228997322329f * 2.f * z;
        fTmp1B_z = 1.445305721320277f;
        fC2_x = fC1 + x * fC1_x - y * fS1_x;
        fC2_y = x * fC1_y - fS1 - y * fS1_y;
        fS2_x = fS1 + x * fS1_x + y * fC1_x;
        fS2_y = x * fS1_y + fC1 + y * fC1_y;
        pSH12_z = 3.f * 1.865881662950577f * z2 - 1.119528997770346f;
        pSH13_x = fTmp0C;
        pSH13_z = fTmp0C_z * x;
        pSH11_y = fTmp0C;
        pSH11_z = fTmp0C_z * y;
        pSH14_x = fTmp1B * fC1_x;
        pSH14_y = fTmp1B * fC1_y;
        pSH14_z = fTmp1B_z * fC1;
        pSH10_x = fTmp1B * fS1_x;
        pSH10_y = fTmp1B * fS1_y;
        pSH10_z = fTmp1B_z * fS1;
        pSH15_x = -0.5900435899266435f * fC2_x;
        pSH15_y = -0.5900435899266435f * fC2_y;
        pSH9_x = -0.5900435899266435f * fS2_x;
        pSH9_y = -0.5900435899266435f * fS2_y;

        v_x += v_colors_local *
               (pSH9_x * coeffs[9 * 3 + c] + pSH15_x * coeffs[15 * 3 + c] +
                pSH10_x * coeffs[10 * 3 + c] + pSH14_x * coeffs[14 * 3 + c] +
                pSH13_x * coeffs[13 * 3 + c]);

        v_y += v_colors_local *
               (pSH9_y * coeffs[9 * 3 + c] + pSH15_y * coeffs[15 * 3 + c] +
                pSH10_y * coeffs[10 * 3 + c] + pSH14_y * coeffs[14 * 3 + c] +
                pSH11_y * coeffs[11 * 3 + c]);

        v_z += v_colors_local *
               (pSH12_z * coeffs[12 * 3 + c] + pSH13_z * coeffs[13 * 3 + c] +
                pSH11_z * coeffs[11 * 3 + c] + pSH14_z * coeffs[14 * 3 + c] +
                pSH10_z * coeffs[10 * 3 + c]);
    }

    if (degree < 4) {
        if (v_dir != nullptr) {
            vec3 dir_n = vec3(x, y, z);
            vec3 v_dir_n = vec3(v_x, v_y, v_z);
            vec3 v_d = (v_dir_n - glm::dot(v_dir_n, dir_n) * dir_n) * inorm;

            v_dir->x = v_d.x;
            v_dir->y = v_d.y;
            v_dir->z = v_d.z;
        }
        return;
    }

    float fTmp0D = z * (-4.683325804901025f * z2 + 2.007139630671868f);
    float fTmp1C = 3.31161143515146f * z2 - 0.47308734787878f;
    float fTmp2B = -1.770130769779931f * z;
    float fC3 = x * fC2 - y * fS2;
    float fS3 = x * fS2 + y * fC2;
    float pSH20 = (1.984313483298443f * z * pSH12 + -1.006230589874905f * pSH6);
    float pSH21 = fTmp0D * x;
    float pSH19 = fTmp0D * y;
    float pSH22 = fTmp1C * fC1;
    float pSH18 = fTmp1C * fS1;
    float pSH23 = fTmp2B * fC2;
    float pSH17 = fTmp2B * fS2;
    float pSH24 = 0.6258357354491763f * fC3;
    float pSH16 = 0.6258357354491763f * fS3;
    v_coeffs[16 * 3 + c] = pSH16 * v_colors_local;
    v_coeffs[17 * 3 + c] = pSH17 * v_colors_local;
    v_coeffs[18 * 3 + c] = pSH18 * v_colors_local;
    v_coeffs[19 * 3 + c] = pSH19 * v_colors_local;
    v_coeffs[20 * 3 + c] = pSH20 * v_colors_local;
    v_coeffs[21 * 3 + c] = pSH21 * v_colors_local;
    v_coeffs[22 * 3 + c] = pSH22 * v_colors_local;
    v_coeffs[23 * 3 + c] = pSH23 * v_colors_local;
    v_coeffs[24 * 3 + c] = pSH24 * v_colors_local;

    float fTmp0D_z, fTmp1C_z, fTmp2B_z, fC3_x, fC3_y, fS3_x, fS3_y, pSH20_z,
        pSH21_x, pSH21_z, pSH19_y, pSH19_z, pSH22_x, pSH22_y, pSH22_z, pSH18_x,
        pSH18_y, pSH18_z, pSH23_x, pSH23_y, pSH23_z, pSH17_x, pSH17_y, pSH17_z,
        pSH24_x, pSH24_y, pSH16_x, pSH16_y;
    if (v_dir != nullptr) {
        fTmp0D_z = 3.f * -4.683325804901025f * z2 + 2.007139630671868f;
        fTmp1C_z = 2.f * 3.31161143515146f * z;
        fTmp2B_z = -1.770130769779931f;
        fC3_x = fC2 + x * fC2_x - y * fS2_x;
        fC3_y = x * fC2_y - fS2 - y * fS2_y;
        fS3_x = fS2 + y * fC2_x + x * fS2_x;
        fS3_y = x * fS2_y + fC2 + y * fC2_y;
        pSH20_z = 1.984313483298443f * (pSH12 + z * pSH12_z) +
                  -1.006230589874905f * pSH6_z;
        pSH21_x = fTmp0D;
        pSH21_z = fTmp0D_z * x;
        pSH19_y = fTmp0D;
        pSH19_z = fTmp0D_z * y;
        pSH22_x = fTmp1C * fC1_x;
        pSH22_y = fTmp1C * fC1_y;
        pSH22_z = fTmp1C_z * fC1;
        pSH18_x = fTmp1C * fS1_x;
        pSH18_y = fTmp1C * fS1_y;
        pSH18_z = fTmp1C_z * fS1;
        pSH23_x = fTmp2B * fC2_x;
        pSH23_y = fTmp2B * fC2_y;
        pSH23_z = fTmp2B_z * fC2;
        pSH17_x = fTmp2B * fS2_x;
        pSH17_y = fTmp2B * fS2_y;
        pSH17_z = fTmp2B_z * fS2;
        pSH24_x = 0.6258357354491763f * fC3_x;
        pSH24_y = 0.6258357354491763f * fC3_y;
        pSH16_x = 0.6258357354491763f * fS3_x;
        pSH16_y = 0.6258357354491763f * fS3_y;

        v_x += v_colors_local *
               (pSH16_x * coeffs[16 * 3 + c] + pSH24_x * coeffs[24 * 3 + c] +
                pSH17_x * coeffs[17 * 3 + c] + pSH23_x * coeffs[23 * 3 + c] +
                pSH18_x * coeffs[18 * 3 + c] + pSH22_x * coeffs[22 * 3 + c] +
                pSH21_x * coeffs[21 * 3 + c]);
        v_y += v_colors_local *
               (pSH16_y * coeffs[16 * 3 + c] + pSH24_y * coeffs[24 * 3 + c] +
                pSH17_y * coeffs[17 * 3 + c] + pSH23_y * coeffs[23 * 3 + c] +
                pSH18_y * coeffs[18 * 3 + c] + pSH22_y * coeffs[22 * 3 + c] +
                pSH19_y * coeffs[19 * 3 + c]);
        v_z += v_colors_local *
               (pSH20_z * coeffs[20 * 3 + c] + pSH21_z * coeffs[21 * 3 + c] +
                pSH19_z * coeffs[19 * 3 + c] + pSH22_z * coeffs[22 * 3 + c] +
                pSH18_z * coeffs[18 * 3 + c] + pSH23_z * coeffs[23 * 3 + c] +
                pSH17_z * coeffs[17 * 3 + c]);

        vec3 dir_n = vec3(x, y, z);
        vec3 v_dir_n = vec3(v_x, v_y, v_z);
        vec3 v_d = (v_dir_n - glm::dot(v_dir_n, dir_n) * dir_n) * inorm;

        v_dir->x = v_d.x;
        v_dir->y = v_d.y;
        v_dir->z = v_d.z;
    }
}

template <typename scalar_t>
__global__ void spherical_harmonics_fwd_kernel(
    const uint32_t N,
    const uint32_t K,
    const uint32_t degrees_to_use,
    const vec3 *__restrict__ dirs,       // [N, 3]
    const scalar_t *__restrict__ coeffs, // [N, K, 3]
    const bool *__restrict__ masks,      // [N]
    scalar_t *__restrict__ colors        // [N, 3]
) {
    // parallelize over N * 3
    uint32_t idx = cg::this_grid().thread_rank();
    if (idx >= N * 3) {
        return;
    }
    uint32_t elem_id = idx / 3;
    uint32_t c = idx % 3; // color channel
    if (masks != nullptr && !masks[elem_id]) {
        return;
    }
    sh_coeffs_to_color_fast(
        degrees_to_use,
        c,
        dirs[elem_id],
        coeffs + elem_id * K * 3,
        colors + elem_id * 3
    );
}

void launch_spherical_harmonics_fwd_kernel(
    // inputs
    const uint32_t degrees_to_use,
    const at::Tensor dirs,                // [..., 3]
    const at::Tensor coeffs,              // [..., K, 3]
    const at::optional<at::Tensor> masks, // [...]
    // outputs
    at::Tensor colors // [..., 2]
) {
    const uint32_t K = coeffs.size(-2);
    const uint32_t N = dirs.numel() / 3;

    // parallelize over N * 3
    int64_t n_elements = N * 3;
    dim3 threads(256);
    dim3 grid((n_elements + threads.x - 1) / threads.x);
    int64_t shmem_size = 0; // No shared memory used in this kernel

    if (n_elements == 0) {
        // skip the kernel launch if there are no elements
        return;
    }

    AT_DISPATCH_FLOATING_TYPES(
        dirs.scalar_type(),
        "spherical_harmonics_fwd_kernel",
        [&]() {
            spherical_harmonics_fwd_kernel<scalar_t>
                <<<grid,
                   threads,
                   shmem_size,
                   at::cuda::getCurrentCUDAStream()>>>(
                    N,
                    K,
                    degrees_to_use,
                    reinterpret_cast<vec3 *>(dirs.data_ptr<scalar_t>()),
                    coeffs.data_ptr<scalar_t>(),
                    masks.has_value() ? masks.value().data_ptr<bool>()
                                      : nullptr,
                    colors.data_ptr<scalar_t>()
                );
        }
    );
}

template <typename scalar_t>
__global__ void spherical_harmonics_bwd_kernel(
    const uint32_t N,
    const uint32_t K,
    const uint32_t degrees_to_use,
    const vec3 *__restrict__ dirs,         // [N, 3]
    const scalar_t *__restrict__ coeffs,   // [N, K, 3]
    const bool *__restrict__ masks,        // [N]
    const scalar_t *__restrict__ v_colors, // [N, 3
    scalar_t *__restrict__ v_coeffs,       // [N, K, 3]
    scalar_t *__restrict__ v_dirs          // [N, 3] optional
) {
    // parallelize over N * 3
    uint32_t idx = cg::this_grid().thread_rank();
    if (idx >= N * 3) {
        return;
    }
    uint32_t elem_id = idx / 3;
    uint32_t c = idx % 3; // color channel
    if (masks != nullptr && !masks[elem_id]) {
        return;
    }

    vec3 v_dir = {0.f, 0.f, 0.f};
    sh_coeffs_to_color_fast_vjp(
        degrees_to_use,
        c,
        dirs[elem_id],
        coeffs + elem_id * K * 3,
        v_colors + elem_id * 3,
        v_coeffs + elem_id * K * 3,
        v_dirs == nullptr ? nullptr : &v_dir
    );
    if (v_dirs != nullptr) {
        gpuAtomicAdd(v_dirs + elem_id * 3, v_dir.x);
        gpuAtomicAdd(v_dirs + elem_id * 3 + 1, v_dir.y);
        gpuAtomicAdd(v_dirs + elem_id * 3 + 2, v_dir.z);
    }
}

void launch_spherical_harmonics_bwd_kernel(
    // inputs
    const uint32_t degrees_to_use,
    const at::Tensor dirs,                // [..., 3]
    const at::Tensor coeffs,              // [..., K, 3]
    const at::optional<at::Tensor> masks, // [...]
    const at::Tensor v_colors,            // [..., 3]
    // outputs
    at::Tensor v_coeffs,            // [..., K, 3]
    at::optional<at::Tensor> v_dirs // [..., 3]
) {
    const uint32_t K = coeffs.size(-2);
    const uint32_t N = dirs.numel() / 3;

    // parallelize over N * 3
    int64_t n_elements = N * 3;
    dim3 threads(256);
    dim3 grid((n_elements + threads.x - 1) / threads.x);
    int64_t shmem_size = 0; // No shared memory used in this kernel

    if (n_elements == 0) {
        // skip the kernel launch if there are no elements
        return;
    }

    AT_DISPATCH_FLOATING_TYPES(
        dirs.scalar_type(),
        "spherical_harmonics_bwd_kernel",
        [&]() {
            spherical_harmonics_bwd_kernel<scalar_t>
                <<<grid,
                   threads,
                   shmem_size,
                   at::cuda::getCurrentCUDAStream()>>>(
                    N,
                    K,
                    degrees_to_use,
                    reinterpret_cast<vec3 *>(dirs.data_ptr<scalar_t>()),
                    coeffs.data_ptr<scalar_t>(),
                    masks.has_value() ? masks.value().data_ptr<bool>()
                                      : nullptr,
                    v_colors.data_ptr<scalar_t>(),
                    v_coeffs.data_ptr<scalar_t>(),
                    v_dirs.has_value() ? v_dirs.value().data_ptr<scalar_t>()
                                       : nullptr
                );
        }
    );
}

} // namespace gsplat
