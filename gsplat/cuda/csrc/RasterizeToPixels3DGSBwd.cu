#include "hip/hip_runtime.h"
#include <ATen/Dispatch.h>
#include <ATen/core/Tensor.h>
#include <ATen/cuda/Atomic.cuh>
#include <c10/cuda/CUDAStream.h>
#include <hip/hip_cooperative_groups.h>

#include "Common.h"
#include "Rasterization.h"
#include "Utils.cuh"

namespace gsplat {

namespace cg = cooperative_groups;

template <uint32_t CDIM, typename scalar_t>
__global__ void rasterize_to_pixels_3dgs_bwd_kernel(
    const uint32_t C,
    const uint32_t N,
    const uint32_t n_isects,
    const bool packed,
    // fwd inputs
    const vec2 *__restrict__ means2d,         // [C, N, 2] or [nnz, 2]
    const vec3 *__restrict__ conics,          // [C, N, 3] or [nnz, 3]
    const scalar_t *__restrict__ colors,      // [C, N, CDIM] or [nnz, CDIM]
    const scalar_t *__restrict__ opacities,   // [C, N] or [nnz]
    const scalar_t *__restrict__ backgrounds, // [C, CDIM] or [nnz, CDIM]
    const bool *__restrict__ masks,           // [C, tile_height, tile_width]
    const uint32_t image_width,
    const uint32_t image_height,
    const uint32_t tile_size,
    const uint32_t tile_width,
    const uint32_t tile_height,
    const int32_t *__restrict__ tile_offsets, // [C, tile_height, tile_width]
    const int32_t *__restrict__ flatten_ids,  // [n_isects]
    // fwd outputs
    const scalar_t
        *__restrict__ render_alphas,      // [C, image_height, image_width, 1]
    const int32_t *__restrict__ last_ids, // [C, image_height, image_width]
    // grad outputs
    const scalar_t *__restrict__ v_render_colors, // [C, image_height,
                                                  // image_width, CDIM]
    const scalar_t
        *__restrict__ v_render_alphas, // [C, image_height, image_width, 1]
    // grad inputs
    vec2 *__restrict__ v_means2d_abs,  // [C, N, 2] or [nnz, 2]
    vec2 *__restrict__ v_means2d,      // [C, N, 2] or [nnz, 2]
    vec3 *__restrict__ v_conics,       // [C, N, 3] or [nnz, 3]
    scalar_t *__restrict__ v_colors,   // [C, N, CDIM] or [nnz, CDIM]
    scalar_t *__restrict__ v_opacities // [C, N] or [nnz]
) {
    auto block = cg::this_thread_block();
    uint32_t camera_id = block.group_index().x;
    uint32_t tile_id =
        block.group_index().y * tile_width + block.group_index().z;
    uint32_t i = block.group_index().y * tile_size + block.thread_index().y;
    uint32_t j = block.group_index().z * tile_size + block.thread_index().x;

    tile_offsets += camera_id * tile_height * tile_width;
    render_alphas += camera_id * image_height * image_width;
    last_ids += camera_id * image_height * image_width;
    v_render_colors += camera_id * image_height * image_width * CDIM;
    v_render_alphas += camera_id * image_height * image_width;
    if (backgrounds != nullptr) {
        backgrounds += camera_id * CDIM;
    }
    if (masks != nullptr) {
        masks += camera_id * tile_height * tile_width;
    }

    // when the mask is provided, do nothing and return if
    // this tile is labeled as False
    if (masks != nullptr && !masks[tile_id]) {
        return;
    }

    const float px = (float)j + 0.5f;
    const float py = (float)i + 0.5f;
    // clamp this value to the last pixel
    const int32_t pix_id =
        min(i * image_width + j, image_width * image_height - 1);

    // keep not rasterizing threads around for reading data
    bool inside = (i < image_height && j < image_width);

    // have all threads in tile process the same gaussians in batches
    // first collect gaussians between range.x and range.y in batches
    // which gaussians to look through in this tile
    int32_t range_start = tile_offsets[tile_id];
    int32_t range_end =
        (camera_id == C - 1) && (tile_id == tile_width * tile_height - 1)
            ? n_isects
            : tile_offsets[tile_id + 1];
    const uint32_t block_size = block.size();
    const uint32_t num_batches =
        (range_end - range_start + block_size - 1) / block_size;

    extern __shared__ int s[];
    int32_t *id_batch = (int32_t *)s; // [block_size]
    vec3 *xy_opacity_batch =
        reinterpret_cast<vec3 *>(&id_batch[block_size]); // [block_size]
    vec3 *conic_batch =
        reinterpret_cast<vec3 *>(&xy_opacity_batch[block_size]); // [block_size]
    float *rgbs_batch =
        (float *)&conic_batch[block_size]; // [block_size * CDIM]

    // this is the T AFTER the last gaussian in this pixel
    float T_final = 1.0f - render_alphas[pix_id];
    float T = T_final;
    // the contribution from gaussians behind the current one
    float buffer[CDIM] = {0.f};
    // index of last gaussian to contribute to this pixel
    const int32_t bin_final = inside ? last_ids[pix_id] : 0;

    // df/d_out for this pixel
    float v_render_c[CDIM];
#pragma unroll
    for (uint32_t k = 0; k < CDIM; ++k) {
        v_render_c[k] = v_render_colors[pix_id * CDIM + k];
    }
    const float v_render_a = v_render_alphas[pix_id];

    // collect and process batches of gaussians
    // each thread loads one gaussian at a time before rasterizing
    const uint32_t tr = block.thread_rank();
    cg::thread_block_tile<32> warp = cg::tiled_partition<32>(block);
    const int32_t warp_bin_final =
        cg::reduce(warp, bin_final, cg::greater<int>());
    for (uint32_t b = 0; b < num_batches; ++b) {
        // resync all threads before writing next batch of shared mem
        block.sync();

        // each thread fetch 1 gaussian from back to front
        // 0 index will be furthest back in batch
        // index of gaussian to load
        // batch end is the index of the last gaussian in the batch
        // These values can be negative so must be int32 instead of uint32
        const int32_t batch_end = range_end - 1 - block_size * b;
        const int32_t batch_size = min(block_size, batch_end + 1 - range_start);
        const int32_t idx = batch_end - tr;
        if (idx >= range_start) {
            int32_t g = flatten_ids[idx]; // flatten index in [C * N] or [nnz]
            id_batch[tr] = g;
            const vec2 xy = means2d[g];
            const float opac = opacities[g];
            xy_opacity_batch[tr] = {xy.x, xy.y, opac};
            conic_batch[tr] = conics[g];
#pragma unroll
            for (uint32_t k = 0; k < CDIM; ++k) {
                rgbs_batch[tr * CDIM + k] = colors[g * CDIM + k];
            }
        }
        // wait for other threads to collect the gaussians in batch
        block.sync();
        // process gaussians in the current batch for this pixel
        // 0 index is the furthest back gaussian in the batch
        for (uint32_t t = max(0, batch_end - warp_bin_final); t < batch_size;
             ++t) {
            bool valid = inside;
            if (batch_end - t > bin_final) {
                valid = 0;
            }
            float alpha;
            float opac;
            vec2 delta;
            vec3 conic;
            float vis;

            if (valid) {
                conic = conic_batch[t];
                vec3 xy_opac = xy_opacity_batch[t];
                opac = xy_opac.z;
                delta = {xy_opac.x - px, xy_opac.y - py};
                float sigma = 0.5f * (conic.x * delta.x * delta.x +
                                      conic.z * delta.y * delta.y) +
                              conic.y * delta.x * delta.y;
                vis = __expf(-sigma);
                alpha = min(0.999f, opac * vis);
                if (sigma < 0.f || alpha < 1.f / 255.f) {
                    valid = false;
                }
            }

            // if all threads are inactive in this warp, skip this loop
            if (!warp.any(valid)) {
                continue;
            }
            float v_rgb_local[CDIM] = {0.f};
            vec3 v_conic_local = {0.f, 0.f, 0.f};
            vec2 v_xy_local = {0.f, 0.f};
            vec2 v_xy_abs_local = {0.f, 0.f};
            float v_opacity_local = 0.f;
            // initialize everything to 0, only set if the lane is valid
            if (valid) {
                // compute the current T for this gaussian
                float ra = 1.0f / (1.0f - alpha);
                T *= ra;
                // update v_rgb for this gaussian
                const float fac = alpha * T;
#pragma unroll
                for (uint32_t k = 0; k < CDIM; ++k) {
                    v_rgb_local[k] = fac * v_render_c[k];
                }
                // contribution from this pixel
                float v_alpha = 0.f;
#pragma unroll
                for (uint32_t k = 0; k < CDIM; ++k) {
                    v_alpha += (rgbs_batch[t * CDIM + k] * T - buffer[k] * ra) *
                               v_render_c[k];
                }

                v_alpha += T_final * ra * v_render_a;
                // contribution from background pixel
                if (backgrounds != nullptr) {
                    float accum = 0.f;
#pragma unroll
                    for (uint32_t k = 0; k < CDIM; ++k) {
                        accum += backgrounds[k] * v_render_c[k];
                    }
                    v_alpha += -T_final * ra * accum;
                }

                if (opac * vis <= 0.999f) {
                    const float v_sigma = -opac * vis * v_alpha;
                    v_conic_local = {
                        0.5f * v_sigma * delta.x * delta.x,
                        v_sigma * delta.x * delta.y,
                        0.5f * v_sigma * delta.y * delta.y
                    };
                    v_xy_local = {
                        v_sigma * (conic.x * delta.x + conic.y * delta.y),
                        v_sigma * (conic.y * delta.x + conic.z * delta.y)
                    };
                    if (v_means2d_abs != nullptr) {
                        v_xy_abs_local = {abs(v_xy_local.x), abs(v_xy_local.y)};
                    }
                    v_opacity_local = vis * v_alpha;
                }

#pragma unroll
                for (uint32_t k = 0; k < CDIM; ++k) {
                    buffer[k] += rgbs_batch[t * CDIM + k] * fac;
                }
            }
            warpSum<CDIM>(v_rgb_local, warp);
            warpSum(v_conic_local, warp);
            warpSum(v_xy_local, warp);
            if (v_means2d_abs != nullptr) {
                warpSum(v_xy_abs_local, warp);
            }
            warpSum(v_opacity_local, warp);
            if (warp.thread_rank() == 0) {
                int32_t g = id_batch[t]; // flatten index in [C * N] or [nnz]
                float *v_rgb_ptr = (float *)(v_colors) + CDIM * g;
#pragma unroll
                for (uint32_t k = 0; k < CDIM; ++k) {
                    gpuAtomicAdd(v_rgb_ptr + k, v_rgb_local[k]);
                }

                float *v_conic_ptr = (float *)(v_conics) + 3 * g;
                gpuAtomicAdd(v_conic_ptr, v_conic_local.x);
                gpuAtomicAdd(v_conic_ptr + 1, v_conic_local.y);
                gpuAtomicAdd(v_conic_ptr + 2, v_conic_local.z);

                float *v_xy_ptr = (float *)(v_means2d) + 2 * g;
                gpuAtomicAdd(v_xy_ptr, v_xy_local.x);
                gpuAtomicAdd(v_xy_ptr + 1, v_xy_local.y);

                if (v_means2d_abs != nullptr) {
                    float *v_xy_abs_ptr = (float *)(v_means2d_abs) + 2 * g;
                    gpuAtomicAdd(v_xy_abs_ptr, v_xy_abs_local.x);
                    gpuAtomicAdd(v_xy_abs_ptr + 1, v_xy_abs_local.y);
                }

                gpuAtomicAdd(v_opacities + g, v_opacity_local);
            }
        }
    }
}

template <uint32_t CDIM>
void launch_rasterize_to_pixels_3dgs_bwd_kernel(
    // Gaussian parameters
    const at::Tensor means2d,                   // [C, N, 2] or [nnz, 2]
    const at::Tensor conics,                    // [C, N, 3] or [nnz, 3]
    const at::Tensor colors,                    // [C, N, 3] or [nnz, 3]
    const at::Tensor opacities,                 // [C, N] or [nnz]
    const at::optional<at::Tensor> backgrounds, // [C, 3]
    const at::optional<at::Tensor> masks,       // [C, tile_height, tile_width]
    // image size
    const uint32_t image_width,
    const uint32_t image_height,
    const uint32_t tile_size,
    // intersections
    const at::Tensor tile_offsets, // [C, tile_height, tile_width]
    const at::Tensor flatten_ids,  // [n_isects]
    // forward outputs
    const at::Tensor render_alphas, // [C, image_height, image_width, 1]
    const at::Tensor last_ids,      // [C, image_height, image_width]
    // gradients of outputs
    const at::Tensor v_render_colors, // [C, image_height, image_width, 3]
    const at::Tensor v_render_alphas, // [C, image_height, image_width, 1]
    // outputs
    at::optional<at::Tensor> v_means2d_abs, // [C, N, 2] or [nnz, 2]
    at::Tensor v_means2d,                   // [C, N, 2] or [nnz, 2]
    at::Tensor v_conics,                    // [C, N, 3] or [nnz, 3]
    at::Tensor v_colors,                    // [C, N, 3] or [nnz, 3]
    at::Tensor v_opacities                  // [C, N] or [nnz]
) {
    bool packed = means2d.dim() == 2;

    uint32_t C = tile_offsets.size(0);         // number of cameras
    uint32_t N = packed ? 0 : means2d.size(1); // number of gaussians
    uint32_t tile_height = tile_offsets.size(1);
    uint32_t tile_width = tile_offsets.size(2);
    uint32_t n_isects = flatten_ids.size(0);

    // Each block covers a tile on the image. In total there are
    // C * tile_height * tile_width blocks.
    dim3 threads = {tile_size, tile_size, 1};
    dim3 grid = {C, tile_height, tile_width};

    int64_t shmem_size =
        tile_size * tile_size *
        (sizeof(int32_t) + sizeof(vec3) + sizeof(vec3) + sizeof(float) * CDIM);

    if (n_isects == 0) {
        // skip the kernel launch if there are no elements
        return;
    }

    // TODO: an optimization can be done by passing the actual number of
    // channels into the kernel functions and avoid necessary global memory
    // writes. This requires moving the channel padding from python to C side.
    if (hipFuncSetAttribute(reinterpret_cast<const void*>(
            rasterize_to_pixels_3dgs_bwd_kernel<CDIM), float>,
            hipFuncAttributeMaxDynamicSharedMemorySize,
            shmem_size
        ) != hipSuccess) {
        AT_ERROR(
            "Failed to set maximum shared memory size (requested ",
            shmem_size,
            " bytes), try lowering tile_size."
        );
    }

    rasterize_to_pixels_3dgs_bwd_kernel<CDIM, float>
        <<<grid, threads, shmem_size, at::cuda::getCurrentCUDAStream()>>>(
            C,
            N,
            n_isects,
            packed,
            reinterpret_cast<vec2 *>(means2d.data_ptr<float>()),
            reinterpret_cast<vec3 *>(conics.data_ptr<float>()),
            colors.data_ptr<float>(),
            opacities.data_ptr<float>(),
            backgrounds.has_value() ? backgrounds.value().data_ptr<float>()
                                    : nullptr,
            masks.has_value() ? masks.value().data_ptr<bool>() : nullptr,
            image_width,
            image_height,
            tile_size,
            tile_width,
            tile_height,
            tile_offsets.data_ptr<int32_t>(),
            flatten_ids.data_ptr<int32_t>(),
            render_alphas.data_ptr<float>(),
            last_ids.data_ptr<int32_t>(),
            v_render_colors.data_ptr<float>(),
            v_render_alphas.data_ptr<float>(),
            v_means2d_abs.has_value()
                ? reinterpret_cast<vec2 *>(
                      v_means2d_abs.value().data_ptr<float>()
                  )
                : nullptr,
            reinterpret_cast<vec2 *>(v_means2d.data_ptr<float>()),
            reinterpret_cast<vec3 *>(v_conics.data_ptr<float>()),
            v_colors.data_ptr<float>(),
            v_opacities.data_ptr<float>()
        );
}

// Explicit Instantiation: this should match how it is being called in .cpp
// file.
// TODO: this is slow to compile, can we do something about it?
#define __INS__(CDIM)                                                          \
    template void launch_rasterize_to_pixels_3dgs_bwd_kernel<CDIM>(            \
        const at::Tensor means2d,                                              \
        const at::Tensor conics,                                               \
        const at::Tensor colors,                                               \
        const at::Tensor opacities,                                            \
        const at::optional<at::Tensor> backgrounds,                            \
        const at::optional<at::Tensor> masks,                                  \
        uint32_t image_width,                                                  \
        uint32_t image_height,                                                 \
        uint32_t tile_size,                                                    \
        const at::Tensor tile_offsets,                                         \
        const at::Tensor flatten_ids,                                          \
        const at::Tensor render_alphas,                                        \
        const at::Tensor last_ids,                                             \
        const at::Tensor v_render_colors,                                      \
        const at::Tensor v_render_alphas,                                      \
        at::optional<at::Tensor> v_means2d_abs,                                \
        at::Tensor v_means2d,                                                  \
        at::Tensor v_conics,                                                   \
        at::Tensor v_colors,                                                   \
        at::Tensor v_opacities                                                 \
    );

__INS__(1)
__INS__(2)
__INS__(3)
__INS__(4)
__INS__(5)
__INS__(8)
__INS__(9)
__INS__(16)
__INS__(17)
__INS__(32)
__INS__(33)
__INS__(64)
__INS__(65)
__INS__(128)
__INS__(129)
__INS__(256)
__INS__(257)
__INS__(512)
__INS__(513)
#undef __INS__

} // namespace gsplat
