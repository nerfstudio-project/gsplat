#include "hip/hip_runtime.h"
#include "forward.cuh"
#include "helpers.cuh"
#include <algorithm>
#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/reduce.h>
#include <hipcub/hipcub.hpp>
#include <cub/device/device_radix_sort.cuh>
#include <iostream>

namespace cg = cooperative_groups;

// kernel function for projecting each gaussian on device
// each thread processes one gaussian
__global__ void project_gaussians_forward_kernel(
    const int num_points,
    const float3 *means3d,
    const float3 *scales,
    const float glob_scale,
    const float4 *quats,
    const float *viewmat,
    const float *projmat,
    const float4 intrins,
    const dim3 img_size,
    const dim3 tile_bounds,
    const float clip_thresh,
    float *covs3d,
    float2 *xys,
    float *depths,
    int *radii,
    float3 *conics,
    int32_t *num_tiles_hit
) {
    unsigned idx = cg::this_grid().thread_rank(); // idx of thread within grid
    if (idx >= num_points) {
        return;
    }
    radii[idx] = 0;
    num_tiles_hit[idx] = 0;

    float3 p_world = means3d[idx];
    // printf("p_world %d %.2f %.2f %.2f\n", idx, p_world.x, p_world.y,
    // p_world.z);
    float3 p_view;
    if (clip_near_plane(p_world, viewmat, p_view, clip_thresh)) {
        // printf("%d is out of frustum z %.2f, returning\n", idx, p_view.z);
        return;
    }
    // printf("p_view %d %.2f %.2f %.2f\n", idx, p_view.x, p_view.y, p_view.z);

    // compute the projected covariance
    float3 scale = scales[idx];
    float4 quat = quats[idx];
    // printf("%d scale %.2f %.2f %.2f\n", idx, scale.x, scale.y, scale.z);
    // printf("%d quat %.2f %.2f %.2f %.2f\n", idx, quat.w, quat.x, quat.y,
    // quat.z);
    float *cur_cov3d = &(covs3d[6 * idx]);
    scale_rot_to_cov3d(scale, glob_scale, quat, cur_cov3d);

    // project to 2d with ewa approximation
    float fx = intrins.x;
    float fy = intrins.y;
    float cx = intrins.z;
    float cy = intrins.w;
    float tan_fovx = 0.5 * img_size.x / fx;
    float tan_fovy = 0.5 * img_size.y / fy;
    float3 cov2d = project_cov3d_ewa(
        p_world, cur_cov3d, viewmat, fx, fy, tan_fovx, tan_fovy
    );
    // printf("cov2d %d, %.2f %.2f %.2f\n", idx, cov2d.x, cov2d.y, cov2d.z);

    float3 conic;
    float radius;
    bool ok = compute_cov2d_bounds(cov2d, conic, radius);
    if (!ok)
        return; // zero determinant
    // printf("conic %d %.2f %.2f %.2f\n", idx, conic.x, conic.y, conic.z);
    conics[idx] = conic;

    // compute the projected mean
    float2 center = project_pix(projmat, p_world, img_size, {cx, cy});
    uint2 tile_min, tile_max;
    get_tile_bbox(center, radius, tile_bounds, tile_min, tile_max);
    int32_t tile_area = (tile_max.x - tile_min.x) * (tile_max.y - tile_min.y);
    if (tile_area <= 0) {
        // printf("%d point bbox outside of bounds\n", idx);
        return;
    }

    num_tiles_hit[idx] = tile_area;
    depths[idx] = p_view.z;
    radii[idx] = (int)radius;
    xys[idx] = center;
    // printf(
    //     "point %d x %.2f y %.2f z %.2f, radius %d, # tiles %d, tile_min %d
    //     %d, tile_max %d %d\n", idx, center.x, center.y, depths[idx],
    //     radii[idx], tile_area, tile_min.x, tile_min.y, tile_max.x, tile_max.y
    // );
}

// host function to launch the projection in parallel on device
void project_gaussians_forward_impl(
    const int num_points,
    const float3 *means3d,
    const float3 *scales,
    const float glob_scale,
    const float4 *quats,
    const float *viewmat,
    const float *projmat,
    const float4 intrins,
    const dim3 img_size,
    const dim3 tile_bounds,
    const float clip_thresh,
    float *covs3d,
    float2 *xys,
    float *depths,
    int *radii,
    float3 *conics,
    int *num_tiles_hit
) {
    project_gaussians_forward_kernel<<<
        (num_points + N_THREADS - 1) / N_THREADS,
        N_THREADS>>>(
        num_points,
        means3d,
        scales,
        glob_scale,
        quats,
        viewmat,
        projmat,
        intrins,
        img_size,
        tile_bounds,
        clip_thresh,
        covs3d,
        xys,
        depths,
        radii,
        conics,
        num_tiles_hit
    );
}

// kernel to map each intersection from tile ID and depth to a gaussian
// writes output to isect_ids and gaussian_ids
__global__ void map_gaussian_to_intersects(
    const int num_points,
    const float2 *xys,
    const float *depths,
    const int *radii,
    const int32_t *cum_tiles_hit,
    const dim3 tile_bounds,
    int64_t *isect_ids,
    int32_t *gaussian_ids
) {
    unsigned idx = cg::this_grid().thread_rank();
    if (idx >= num_points)
        return;
    if (radii[idx] <= 0)
        return;
    // get the tile bbox for gaussian
    uint2 tile_min, tile_max;
    float2 center = xys[idx];
    get_tile_bbox(center, radii[idx], tile_bounds, tile_min, tile_max);
    // printf("point %d, %d radius, min %d %d, max %d %d\n", idx, radii[idx],
    // tile_min.x, tile_min.y, tile_max.x, tile_max.y);

    // update the intersection info for all tiles this gaussian hits
    int32_t cur_idx = (idx == 0) ? 0 : cum_tiles_hit[idx - 1];
    // printf("point %d starting at %d\n", idx, cur_idx);
    int64_t depth_id = (int64_t) * (int32_t *)&(depths[idx]);
    for (int i = tile_min.y; i < tile_max.y; ++i) {
        for (int j = tile_min.x; j < tile_max.x; ++j) {
            // isect_id is tile ID and depth as int32
            int64_t tile_id = i * tile_bounds.x + j; // tile within image
            isect_ids[cur_idx] = (tile_id << 32) | depth_id; // tile | depth id
            gaussian_ids[cur_idx] = idx;                     // 3D gaussian id
            ++cur_idx; // handles gaussians that hit more than one tile
        }
    }
    // printf("point %d ending at %d\n", idx, cur_idx);
}

// kernel to map sorted intersection IDs to tile bins
// expect that intersection IDs are sorted by increasing tile ID
// i.e. intersections of a tile are in contiguous chunks
__global__ void get_tile_bin_edges(
    const int num_intersects, const int64_t *isect_ids_sorted, int2 *tile_bins
) {
    unsigned idx = cg::this_grid().thread_rank();
    if (idx >= num_intersects)
        return;
    // save the indices where the tile_id changes
    int32_t cur_tile_idx = (int32_t)(isect_ids_sorted[idx] >> 32);
    if (idx == 0) {
        tile_bins[cur_tile_idx].x = 0;
        return;
    }
    if (idx == num_intersects - 1) {
        tile_bins[cur_tile_idx].y = num_intersects;
        return;
    }
    int32_t prev_tile_idx = (int32_t)(isect_ids_sorted[idx - 1] >> 32);
    if (prev_tile_idx != cur_tile_idx) {
        tile_bins[prev_tile_idx].y = idx;
        tile_bins[cur_tile_idx].x = idx;
        return;
    }
}

// launch on-device prefix sum to get the cumulative number of tiles for
// gaussians
void compute_cumulative_intersects(
    const int num_points,
    const int32_t *num_tiles_hit,
    int32_t &num_intersects,
    int32_t *cum_tiles_hit
) {
    // ref:
    // https://nvlabs.github.io/cub/structcub_1_1_device_scan.html#a9416ac1ea26f9fde669d83ddc883795a
    // allocate sum workspace
    void *sum_ws = nullptr;
    size_t sum_ws_bytes;
    hipcub::DeviceScan::InclusiveSum(
        sum_ws, sum_ws_bytes, num_tiles_hit, cum_tiles_hit, num_points
    );
    hipMalloc(&sum_ws, sum_ws_bytes);
    hipcub::DeviceScan::InclusiveSum(
        sum_ws, sum_ws_bytes, num_tiles_hit, cum_tiles_hit, num_points
    );
    hipMemcpy(
        &num_intersects,
        &(cum_tiles_hit[num_points - 1]),
        sizeof(int32_t),
        hipMemcpyDeviceToHost
    );
    hipFree(sum_ws);
}

// figure out which gaussians, sorted by depth, to render for which tile of the
// output image output gaussian IDs for each tile, sorted by depth, as
// continguous array output start and end indices in this list of gaussians for
// each tile
void bin_and_sort_gaussians(
    const int num_points,
    const int num_intersects,
    const float2 *xys,
    const float *depths,
    const int *radii,
    const int32_t *cum_tiles_hit,
    const dim3 tile_bounds,
    int64_t *isect_ids_unsorted,
    int32_t *gaussian_ids_unsorted,
    int64_t *isect_ids_sorted,
    int32_t *gaussian_ids_sorted,
    int2 *tile_bins
) {
    // for each intersection map the tile ID and depth to a gaussian ID
    // allocate intermediate results
    // int32_t *gaussian_ids_unsorted;
    // int64_t *isect_ids_unsorted; // *isect_ids_sorted;
    // hipMalloc((void**) &gaussian_ids_unsorted, num_intersects *
    // sizeof(int32_t)); hipMalloc((void**) &isect_ids_unsorted,
    // num_intersects * sizeof(int64_t)); hipMalloc((void**)
    // &isect_ids_sorted, num_intersects * sizeof(int64_t));
    map_gaussian_to_intersects<<<
        (num_points + N_THREADS - 1) / N_THREADS,
        N_THREADS>>>(
        num_points,
        xys,
        depths,
        radii,
        cum_tiles_hit,
        tile_bounds,
        isect_ids_unsorted,
        gaussian_ids_unsorted
    );

    // sort intersections by ascending tile ID and depth with RadixSort
    int32_t max_tile_id = (int32_t)(tile_bounds.x * tile_bounds.y);
    int msb = 32 - __builtin_clz(max_tile_id) + 1;
    // allocate workspace memory
    void *sort_ws = nullptr;
    size_t sort_ws_bytes;
    hipcub::DeviceRadixSort::SortPairs(
        sort_ws,
        sort_ws_bytes,
        isect_ids_unsorted,
        isect_ids_sorted,
        gaussian_ids_unsorted,
        gaussian_ids_sorted,
        num_intersects,
        0,
        32 + msb
    );
    hipMalloc(&sort_ws, sort_ws_bytes);
    hipcub::DeviceRadixSort::SortPairs(
        sort_ws,
        sort_ws_bytes,
        isect_ids_unsorted,
        isect_ids_sorted,
        gaussian_ids_unsorted,
        gaussian_ids_sorted,
        num_intersects,
        0,
        32 + msb
    );
    hipFree(sort_ws);

    // get the start and end indices for the gaussians in each tile
    // printf("launching tile binning %d %d\n",
    // (num_intersects + N_THREADS - 1) / N_THREADS,
    // N_THREADS);
    get_tile_bin_edges<<<
        (num_intersects + N_THREADS - 1) / N_THREADS,
        N_THREADS>>>(num_intersects, isect_ids_sorted, tile_bins);

    // free intermediate work spaces
    // hipFree(isect_ids_unsorted);
    // hipFree(isect_ids_sorted);
    // hipFree(gaussian_ids_unsorted);
}

// kernel function for rasterizing each tile
// each thread treats a single pixel
// each thread group uses the same gaussian data in a tile
__global__ void nd_rasterize_forward_kernel(
    const dim3 tile_bounds,
    const dim3 img_size,
    const unsigned channels,
    const int32_t *gaussian_ids_sorted,
    const int2 *tile_bins,
    const float2 *xys,
    const float3 *conics,
    const float *colors,
    const float *opacities,
    float *final_Ts,
    int *final_index,
    float *out_img,
    const float *background
) {
    // current naive implementation where tile data loading is redundant
    // TODO tile data should be shared between tile threads
    int32_t tile_id = blockIdx.y * tile_bounds.x + blockIdx.x;
    unsigned i = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned j = blockIdx.x * blockDim.x + threadIdx.x;
    float px = (float)j;
    float py = (float)i;
    int32_t pix_id = i * img_size.x + j;

    // return if out of bounds
    if (i >= img_size.y || j >= img_size.x) {
        return;
    }

    // which gaussians to look through in this tile
    int2 range = tile_bins[tile_id];
    float T = 1.f;

    // iterate over all gaussians and apply rendering EWA equation (e.q. 2 from
    // paper)
    int idx;
    for (idx = range.x; idx < range.y; ++idx) {
        const int32_t g = gaussian_ids_sorted[idx];
        const float3 conic = conics[g];
        const float2 center = xys[g];
        const float2 delta = {center.x - px, center.y - py};

        // Mahalanobis distance (here referred to as sigma) measures how many
        // standard deviations away distance delta is. sigma = -0.5(d.T * conic
        // * d)
        const float sigma =
            0.5f * (conic.x * delta.x * delta.x + conic.z * delta.y * delta.y) +
            conic.y * delta.x * delta.y;
        if (sigma < 0.f) {
            continue;
        }
        const float opac = opacities[g];

        const float alpha = min(0.999f, opac * exp(-sigma));

        // break out conditions
        if (alpha < 1.f / 255.f) {
            continue;
        }
        const float next_T = T * (1.f - alpha);
        if (next_T <= 1e-4f) {
            // we want to render the last gaussian that contributes and note
            // that here idx > range.x so we don't underflow
            idx -= 1;
            break;
        }
        const float vis = alpha * T;
        for (int c = 0; c < channels; ++c) {
            out_img[channels * pix_id + c] += colors[channels * g + c] * vis;
        }
        T = next_T;
    }
    final_Ts[pix_id] = T; // transmittance at last gaussian in this pixel
    final_index[pix_id] =
        (idx == range.y)
            ? idx - 1
            : idx; // index of in bin of last gaussian in this pixel
    for (int c = 0; c < channels; ++c) {
        out_img[channels * pix_id + c] += T * background[c];
    }
}

// host function to launch parallel rasterization of sorted gaussians on device
void nd_rasterize_forward_impl(
    const dim3 tile_bounds,
    const dim3 block,
    const dim3 img_size,
    const unsigned channels,
    const int32_t *gaussian_ids_sorted,
    const int2 *tile_bins,
    const float2 *xys,
    const float3 *conics,
    const float *colors,
    const float *opacities,
    float *final_Ts,
    int *final_index,
    float *out_img,
    const float *background
) {
    nd_rasterize_forward_kernel<<<tile_bounds, block>>>(
        tile_bounds,
        img_size,
        channels,
        gaussian_ids_sorted,
        tile_bins,
        xys,
        conics,
        colors,
        opacities,
        final_Ts,
        final_index,
        out_img,
        background
    );
}

__global__ void rasterize_forward_kernel(
    const dim3 tile_bounds,
    const dim3 img_size,
    const int32_t *gaussian_ids_sorted,
    const int2 *tile_bins,
    const float2 *xys,
    const float3 *conics,
    const float3 *colors,
    const float *opacities,
    float *final_Ts,
    int *final_index,
    float3 *out_img,
    const float3 &background
) {
    // each thread draws one pixel, but also timeshares caching gaussians in a
    // shared tile

    auto block = cg::this_thread_block();
    int32_t tile_id =
        block.group_index().y * tile_bounds.x + block.group_index().x;
    unsigned i =
        block.group_index().y * block.group_dim().y + block.thread_index().y;
    unsigned j =
        block.group_index().x * block.group_dim().x + block.thread_index().x;

    float px = (float)j;
    float py = (float)i;
    int32_t pix_id = i * img_size.x + j;

    // return if out of bounds
    // keep not rasterizing threads around for reading data
    bool inside = (i < img_size.y && j < img_size.x);
    bool done = !inside;

    // have all threads in tile process the same gaussians in batches
    // first collect gaussians between range.x and range.y in batches
    // which gaussians to look through in this tile
    int2 range = tile_bins[tile_id];
    int num_batches = (range.y - range.x + BLOCK_SIZE - 1) / BLOCK_SIZE;

    __shared__ int32_t id_batch[BLOCK_SIZE];
    __shared__ float2 xy_batch[BLOCK_SIZE];
    __shared__ float3 conic_batch[BLOCK_SIZE];
    __shared__ float opacity_batch[BLOCK_SIZE];

    // current visibility left to render
    float T = 1.f;
    // index of most recent gaussian to write to this thread's pixel
    int cur_idx = 0;

    // collect and process batches of gaussians
    // each thread loads one gaussian at a time before rasterizing its
    // designated pixel
    int tr = block.thread_rank();
    float3 pix_out = {0.f, 0.f, 0.f};
    for (int b = 0; b < num_batches; ++b) {
        // resync all threads before beginning next batch
        // end early if entire tile is done
        int num_done = __syncthreads_count(done);
        if (num_done >= BLOCK_SIZE) {
            break;
        }

        // each thread fetch 1 gaussian from front to back
        // index of gaussian to load
        int batch_start = range.x + BLOCK_SIZE * b;
        int idx = batch_start + tr;
        if (idx < range.y) {
            int32_t g_id = gaussian_ids_sorted[idx];
            id_batch[tr] = g_id;
            xy_batch[tr] = xys[g_id];
            conic_batch[tr] = conics[g_id];
            opacity_batch[tr] = opacities[g_id];
        }

        // wait for other threads to collect the gaussians in batch
        block.sync();

        // process gaussians in the current batch for this pixel
        int batch_size = min(BLOCK_SIZE, range.y - batch_start);
        for (int t = 0; (t < batch_size) && !done; ++t) {
            const float3 conic = conic_batch[t];
            const float2 center = xy_batch[t];
            const float opac = opacity_batch[t];
            const float2 delta = {center.x - px, center.y - py};
            const float sigma = 0.5f * (conic.x * delta.x * delta.x +
                                        conic.z * delta.y * delta.y) +
                                conic.y * delta.x * delta.y;
            const float alpha = min(0.999f, opac * exp(-sigma));
            if (sigma < 0.f || alpha < 1.f / 255.f) {
                continue;
            }

            const float next_T = T * (1.f - alpha);
            if (next_T <= 1e-4f) { // this pixel is done
                // we want to render the last gaussian that contributes and note
                // that here idx > range.x so we don't underflow
                done = true;
                break;
            }

            int32_t g = id_batch[t];
            const float vis = alpha * T;
            const float3 c = colors[g];
            pix_out.x = pix_out.x + c.x * vis;
            pix_out.y = pix_out.y + c.y * vis;
            pix_out.z = pix_out.z + c.z * vis;
            T = next_T;
            cur_idx = batch_start + t;
        }
    }

    if (inside) {
        // add background
        final_Ts[pix_id] = T; // transmittance at last gaussian in this pixel
        final_index[pix_id] =
            cur_idx; // index of in bin of last gaussian in this pixel
        float3 final_color;
        final_color.x = pix_out.x + T * background.x;
        final_color.y = pix_out.y + T * background.y;
        final_color.z = pix_out.z + T * background.z;
        out_img[pix_id] = final_color;
    }
}

// host function to launch parallel rasterization of sorted gaussians on device
void rasterize_forward_impl(
    const dim3 tile_bounds,
    const dim3 block,
    const dim3 img_size,
    const int32_t *gaussian_ids_sorted,
    const int2 *tile_bins,
    const float2 *xys,
    const float3 *conics,
    const float3 *colors,
    const float *opacities,
    float *final_Ts,
    int *final_index,
    float3 *out_img,
    const float3 &background
) {
    rasterize_forward_kernel<<<tile_bounds, block>>>(
        tile_bounds,
        img_size,
        gaussian_ids_sorted,
        tile_bins,
        xys,
        conics,
        colors,
        opacities,
        final_Ts,
        final_index,
        out_img,
        background
    );
}

// device helper to approximate projected 2d cov from 3d mean and cov
__device__ float3 project_cov3d_ewa(
    const float3 &mean3d,
    const float *cov3d,
    const float *viewmat,
    const float fx,
    const float fy,
    const float tan_fovx,
    const float tan_fovy
) {
    // clip the
    // we expect row major matrices as input, glm uses column major
    // upper 3x3 submatrix
    glm::mat3 W = glm::mat3(
        viewmat[0],
        viewmat[4],
        viewmat[8],
        viewmat[1],
        viewmat[5],
        viewmat[9],
        viewmat[2],
        viewmat[6],
        viewmat[10]
    );
    glm::vec3 p = glm::vec3(viewmat[3], viewmat[7], viewmat[11]);
    glm::vec3 t = W * glm::vec3(mean3d.x, mean3d.y, mean3d.z) + p;

    // clip so that the covariance
    float lim_x = 1.3f * tan_fovx;
    float lim_y = 1.3f * tan_fovy;
    t.x = t.z * std::min(lim_x, std::max(-lim_x, t.x / t.z));
    t.y = t.z * std::min(lim_y, std::max(-lim_y, t.y / t.z));

    float rz = 1.f / t.z;
    float rz2 = rz * rz;

    // column major
    // we only care about the top 2x2 submatrix
    glm::mat3 J = glm::mat3(
        fx * rz,
        0.f,
        0.f,
        0.f,
        fy * rz,
        0.f,
        -fx * t.x * rz2,
        -fy * t.y * rz2,
        0.f
    );
    glm::mat3 T = J * W;

    glm::mat3 V = glm::mat3(
        cov3d[0],
        cov3d[1],
        cov3d[2],
        cov3d[1],
        cov3d[3],
        cov3d[4],
        cov3d[2],
        cov3d[4],
        cov3d[5]
    );

    glm::mat3 cov = T * V * glm::transpose(T);

    // add a little blur along axes and save upper triangular elements
    return (float3
    ){float(cov[0][0]) + 0.3f, float(cov[0][1]), float(cov[1][1]) + 0.3f};
}

// device helper to get 3D covariance from scale and quat parameters
__device__ void scale_rot_to_cov3d(
    const float3 scale, const float glob_scale, const float4 quat, float *cov3d
) {
    // printf("quat %.2f %.2f %.2f %.2f\n", quat.x, quat.y, quat.z, quat.w);
    glm::mat3 R = quat_to_rotmat(quat);
    // printf("R %.2f %.2f %.2f\n", R[0][0], R[1][1], R[2][2]);
    glm::mat3 S = scale_to_mat(scale, glob_scale);
    // printf("S %.2f %.2f %.2f\n", S[0][0], S[1][1], S[2][2]);

    glm::mat3 M = R * S;
    glm::mat3 tmp = M * glm::transpose(M);
    // printf("tmp %.2f %.2f %.2f\n", tmp[0][0], tmp[1][1], tmp[2][2]);

    // save upper right because symmetric
    cov3d[0] = tmp[0][0];
    cov3d[1] = tmp[0][1];
    cov3d[2] = tmp[0][2];
    cov3d[3] = tmp[1][1];
    cov3d[4] = tmp[1][2];
    cov3d[5] = tmp[2][2];
}
